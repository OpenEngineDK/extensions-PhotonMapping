#include "hip/hip_runtime.h"
// Brute force ray tracer for CUDA
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/BruteTracer.h>

#include <Display/IRenderCanvas.h>
#include <Display/IViewingVolume.h>
#include <Utils/CUDA/GeometryList.h>
#include <Utils/CUDA/SharedMemory.h>
#include <Utils/CUDA/Utils.h>
#include <Utils/CUDA/IntersectionTests.h>
#include <Utils/CUDA/LoggerExtensions.h>

namespace OpenEngine {
    using namespace Display;
    using namespace Resources;
    using namespace Resources::CUDA;
    namespace Utils {
        namespace CUDA {

#include <Utils/CUDA/Kernels/ColorKernels.h>

            __constant__ int d_rays;

            BruteTracer::BruteTracer(GeometryList* geom)
                : geom(geom) {

                cutCreateTimer(&timerID);

                float3 lightPosition = make_float3(0.0f, 4.0f, 0.0f);
                hipMemcpyToSymbol(HIP_SYMBOL(d_lightPosition), &lightPosition, sizeof(float3));
                float3 lightColor = make_float3(1.0f, 0.92f, 0.8f);
                float3 ambient = lightColor * 0.3f;
                hipMemcpyToSymbol(HIP_SYMBOL(d_lightAmbient), &ambient, sizeof(float3));
                float3 diffuse = lightColor * 0.7f;
                hipMemcpyToSymbol(HIP_SYMBOL(d_lightDiffuse), &diffuse, sizeof(float3));
                float3 specular = lightColor * 0.3f;
                hipMemcpyToSymbol(HIP_SYMBOL(d_lightSpecular), &specular, sizeof(float3));
                CHECK_FOR_CUDA_ERROR();
            }

            BruteTracer::~BruteTracer() {}

#define MAX_PRIMS 4096
#define MAX_THREADS 128
#define MIN_BLOCKS 2
            
            template <bool useWoop>
            inline __host__ __device__ 
            uchar4 Exhaustive(int id, float4* origins, float4* directions,
                              float4 *v0s, float4 *v1s, float4 *v2s,
                              float4 *n0s, float4 *n1s, float4 *n2s,
                              uchar4 *c0s,
                              int prims){
                
                prims = prims < MAX_PRIMS ? prims : MAX_PRIMS;
                
                float3 origin = make_float3(FetchGlobalData(origins, id));
                float3 dir = make_float3(FetchGlobalData(directions, id));
                
                float3 tHit;
                
                float4 color = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
                
                do {
                    tHit.x = fInfinity;
                        
                    int primHit = -1;
                    for (int prim = 0; prim < prims; ++prim){

                        if (useWoop){
                            IRayTracer::Woop(v0s, v1s, v2s, prim,
                                             origin, dir, primHit, tHit);
                        }else{
                            IRayTracer::MoellerTrumbore(v0s, v1s, v2s, prim,
                                                        origin, dir, primHit, tHit);
                        }
                    }
                        
                    if (tHit.x < fInfinity){
                        float4 newColor = Lighting(tHit, origin, dir, 
                                                   n0s, n1s, n2s,
                                                   c0s, primHit);
                            
                        color = BlendColor(color, newColor);
                    }
                } while(tHit.x < fInfinity && color.w < 0.97f);
                    
                return make_uchar4(color.x * 255, color.y * 255, color.z * 255, color.w * 255);
            }            

            template <bool useWoop>
            __global__ void 
            __launch_bounds__(MAX_THREADS, MIN_BLOCKS) 
                ExhaustiveKernel(float4* origins, float4* directions,
                                 float4 *v0s, float4 *v1s, float4 *v2s,
                                 float4 *n0s, float4 *n1s, float4 *n2s,
                                 uchar4 *c0s,
                                 uchar4 *canvas,
                                 int prims,
                                 int screenWidth){

                int id = blockDim.x * blockIdx.x + threadIdx.x;
                
                if (id < d_rays){
                    
                    id = IRayTracer::PacketIndex(id, screenWidth);

                    uchar4 color = Exhaustive<useWoop>
                        (id, origins, directions, 
                         v0s, v1s, v2s, n0s, n1s, n2s, c0s, prims);
                    
                    DumpGlobalData(color, canvas, id);
                }                
            }

            void BruteTracer::Trace(Display::IRenderCanvas* canvas, uchar4* canvasData){
                CreateInitialRays(canvas);

                int height = canvas->GetHeight();
                int width = canvas->GetWidth();
                
                int rays = height * width;

                hipMemcpyToSymbol(HIP_SYMBOL(d_rays), &rays, sizeof(int));

                if (visualizeRays){
                    RenderRays(canvasData, rays);
                    return;
                }

                START_TIMER(timerID);
                if (intersectionAlgorithm == WOOP){
                    float4 *woop0, *woop1, *woop2;
                    geom->GetWoopValues(&woop0, &woop1, &woop2);

                    KernelConf conf = KernelConf1D(rays, MAX_THREADS);
                    ExhaustiveKernel<true><<<conf.blocks, conf.threads>>>
                        (origin->GetDeviceData(), direction->GetDeviceData(),
                         woop0, woop1, woop2,
                         geom->GetNormal0Data(), geom->GetNormal1Data(), geom->GetNormal2Data(),
                         geom->GetColor0Data(),
                         canvasData,
                         geom->GetSize(),
                         width);
                    if (printTiming) PRINT_TIMER(timerID, "Brute tracing using Woop");

                }else{
                    KernelConf conf = KernelConf1D(rays, 64);
                    if (printTiming) START_TIMER(timerID);
                    //logger.info << "BruteTracing<<<" << blocks << ", " << threads << ", " << smemSize << ">>>" << logger.end;
                    ExhaustiveKernel<false><<<conf.blocks, conf.threads>>>
                        (origin->GetDeviceData(), direction->GetDeviceData(),
                         geom->GetP0Data(), geom->GetP1Data(), geom->GetP2Data(), 
                         geom->GetNormal0Data(), geom->GetNormal1Data(), geom->GetNormal2Data(),
                         geom->GetColor0Data(),
                         canvasData,
                         geom->GetSize(),
                         width);
                    if (printTiming) PRINT_TIMER(timerID, "Brute tracing using Möller-Trumbore");
                }
                hipDeviceSynchronize();
                cutStopTimer(timerID);
                renderTime = cutGetTimerValue(timerID);
                CHECK_FOR_CUDA_ERROR();
            }

            void BruteTracer::HostTrace(int x, int y, TriangleNode* nodes){
                int id = x + y * screenWidth;
                
                uchar4 color;
                if (this->intersectionAlgorithm == WOOP){                
                    float4 *woop0, *woop1, *woop2;
                    geom->GetWoopValues(&woop0, &woop1, &woop2);

                    color = Exhaustive<true>
                        (id, origin->GetDeviceData(), direction->GetDeviceData(),
                         woop0, woop1, woop2,
                         geom->GetNormal0Data(), geom->GetNormal1Data(), geom->GetNormal2Data(),
                         geom->GetColor0Data(),
                         geom->GetSize());

                }else{
                    color = Exhaustive<false>
                        (id, origin->GetDeviceData(), direction->GetDeviceData(),
                         geom->GetP0Data(), geom->GetP1Data(), geom->GetP2Data(), 
                         geom->GetNormal0Data(), geom->GetNormal1Data(), geom->GetNormal2Data(),
                         geom->GetColor0Data(),
                         geom->GetSize());
                }

                logger.info << "Final color: " << make_int4(color.x, color.y, color.z, color.w) << logger.end;
            }            

        }
    }
}
