#include "hip/hip_runtime.h"
// Triangle map class for CUDA
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/TriangleMap.h>
#include <Utils/CUDA/Utils.h>
#include <Utils/CUDA/Convert.h>

#include <Utils/CUDA/Kernels/PhotonMapDeviceVars.h>
#include <Utils/CUDA/Kernels/TriangleUpper.h>
#include <Utils/CUDA/Kernels/TriangleUpperSegment.h>
#include <Utils/CUDA/Kernels/ReduceSegments.h>
#include <Utils/CUDA/Kernels/TriangleUpperChildren.h>
#include <Utils/CUDA/Kernels/TriangleKernels.h>

namespace OpenEngine {
    namespace Utils {
        namespace CUDA {

            using namespace Kernels;

            void TriangleMap::CreateUpperNodes(){
                int activeIndex = 0, activeRange = 1;
                int childrenCreated;
                
                upperNodeLeafs = upperLeafPrimitives = 0;
                
                hipMemcpyToSymbol(HIP_SYMBOL(d_emptySpaceThreshold), &emptySpaceThreshold, sizeof(float));

                // Setup root node!
                int2 i = make_int2(0, triangles);
                hipMemcpy(nodes->GetPrimitiveInfoData(), &i, sizeof(int2), hipMemcpyHostToDevice);
                int parent = 0;
                hipMemcpy(nodes->GetParentData(), &parent, sizeof(int), hipMemcpyHostToDevice);
                nodes->size = 1;

                // Setup bounding box info
                unsigned int blocks, threads;
                Calc1DKernelDimensions(triangles, blocks, threads);
                AddIndexToAabb<<<blocks, threads>>>(geom->GetAabbMinData(), geom->GetSize(), aabbMin->GetDeviceData());
                // @OPT Just switch the arrays.
                hipMemcpy(aabbMax->GetDeviceData(), geom->GetAabbMaxData(), 
                           triangles * sizeof(float4), hipMemcpyDeviceToDevice);
                CHECK_FOR_CUDA_ERROR();                

                START_TIMER(timerID);
                while (activeRange > 0){
                    ProcessUpperNodes(activeIndex, activeRange, 
                                      childrenCreated);

                    //for (int i = 0; i < activeRange; ++i)
                    //logger.info << nodes->ToString(i + activeIndex) << logger.end;

                    activeIndex = nodes->size - childrenCreated;
                    activeRange = childrenCreated;
                }
                PRINT_TIMER(timerID, "triangle upper map");

                triangles = resultMin->GetSize();

                Calc1DKernelDimensions(resultMin->GetSize(), blocks, threads, 128);
                START_TIMER(timerID);                
                AdjustBoundingBox<<<blocks, threads>>>(resultMin->GetDeviceData(), 
                                                       resultMax->GetDeviceData(),
                                                       geom->GetP0Data(),
                                                       geom->GetP1Data(),
                                                       geom->GetP2Data(),
                                                       geom->GetAabbMinData(),
                                                       geom->GetAabbMaxData(),
                                                       geom->GetSurfaceAreaData(),
                                                       resultMin->GetSize());
                PRINT_TIMER(timerID, "Adjusting bounding box");
                CHECK_FOR_CUDA_ERROR();

                /*
                float4 min[resultMin->GetSize()];
                hipMemcpy(min, resultMin->GetDeviceData(), sizeof(float4) * resultMin->GetSize(), hipMemcpyDeviceToHost);
                float4 max[resultMax->GetSize()];
                hipMemcpy(max, resultMax->GetDeviceData(), sizeof(float4) * resultMax->GetSize(), hipMemcpyDeviceToHost);
                int cnt = 0;
                for (int i = 0; i < resultMax->GetSize(); ++i)
                    if (max[i].w == 0.0f) cnt++;

                logger.info << "empty bb's " << cnt << logger.end;

                float3 aabbMin = make_float3(3, 0, -0.00001f);
                float3 aabbMax = make_float3(8, 1, 4);
                bool hit = TightTriangleBB(make_float3(0, 6, 0), make_float3(8, 4, 0), make_float3(6, 0, 0),
                                           aabbMin, aabbMax, true);

                if (hit)
                    logger.info << "min: " << Convert::ToString(aabbMin) << ", max: " << Convert::ToString(aabbMax) << logger.end;
                else
                    logger.info << "missed it: min: " << Convert::ToString(aabbMin) << ", max: " << Convert::ToString(aabbMax) << logger.end;
                */
            }

            void TriangleMap::ProcessUpperNodes(int activeIndex, int activeRange, 
                                                int &childrenCreated){

                //logger.info << "=== Process " << activeRange << " Upper Nodes Starting at " << activeIndex << " === with " << triangles << " triangles" << logger.end;

                // Copy bookkeeping to symbols
                hipMemcpyToSymbol(HIP_SYMBOL(d_activeNodeIndex), &activeIndex, sizeof(int));
                hipMemcpyToSymbol(HIP_SYMBOL(d_activeNodeRange), &activeRange, sizeof(int));
                hipMemcpyToSymbol(HIP_SYMBOL(d_triangles), &triangles, sizeof(int));
                CHECK_FOR_CUDA_ERROR();

                Segment(activeIndex, activeRange);
                CHECK_FOR_CUDA_ERROR();

                // Calculate aabb
                ReduceAabb(activeIndex, activeRange);
                CHECK_FOR_CUDA_ERROR();

                // Calculate children placement
                CreateChildren(activeIndex, activeRange, childrenCreated);
            }

            void TriangleMap::Segment(int activeIndex, int activeRange){
                nodeSegments->Extend(activeRange+1);

                unsigned int blocks, threads;
                Calc1DKernelDimensions(activeRange, blocks, threads);
                NodeSegments<<<blocks, threads>>>(nodes->GetPrimitiveInfoData() + activeIndex,
                                                  nodeSegments->GetDeviceData());

                CHECK_FOR_CUDA_ERROR();
                cudppScan(scanHandle, nodeSegments->GetDeviceData(), nodeSegments->GetDeviceData(), activeRange+1);

                int amountOfSegments;
                hipMemcpy(&amountOfSegments, nodeSegments->GetDeviceData() + activeRange, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpyToSymbol(HIP_SYMBOL(d_segments), nodeSegments->GetDeviceData() + activeRange, sizeof(int), 0, hipMemcpyDeviceToDevice);
                CHECK_FOR_CUDA_ERROR();

                segments.Extend(amountOfSegments);
                segments.size = amountOfSegments;

                hipMemset(segments.GetOwnerData(), 0, amountOfSegments * sizeof(int));
                MarkOwnerStart<<<blocks, threads>>>(segments.GetOwnerData(),
                                                    nodeSegments->GetDeviceData());
                CHECK_FOR_CUDA_ERROR();

                cudppScan(scanInclHandle, segments.GetOwnerData(), segments.GetOwnerData(), amountOfSegments);

                Calc1DKernelDimensions(amountOfSegments, blocks, threads);
                CalcSegmentPrimitives<<<blocks, threads>>>(segments.GetOwnerData(),
                                                           nodeSegments->GetDeviceData(),
                                                           nodes->GetPrimitiveInfoData(),
                                                           segments.GetPrimitiveInfoData());
                CHECK_FOR_CUDA_ERROR();
            }

            void TriangleMap::ReduceAabb(int activeIndex, int activeRange){
                
                // Reduce aabb pr segment
                unsigned int blocks = segments.size;
                unsigned int threads = Segments::SEGMENT_SIZE;
                unsigned int memSize = 2 * sizeof(float4) * segments.SEGMENT_SIZE;

                //START_TIMER(timerID);
                logger.info << "ReduceSegments<<<" << blocks << ", " << threads << ", " << memSize << ">>>" << logger.end;
                ReduceSegmentsShared<<<blocks, threads, memSize>>>(segments.GetPrimitiveInfoData(),
                                                                   aabbMin->GetDeviceData(), aabbMax->GetDeviceData(),
                                                                   segments.GetAabbMinData(), segments.GetAabbMaxData());
                //PRINT_TIMER(timerID, "ReduceSegments");
                // @TODO has provoked an "unspecified launch failure"
                CHECK_FOR_CUDA_ERROR();

#if CPU_VERIFY
                int2 info[segments.size];
                hipMemcpy(info, segments.GetPrimitiveInfoData(), 
                           segments.size * sizeof(int2), hipMemcpyDeviceToHost);

                float4 segMin[segments.size];
                hipMemcpy(segMin, segments.GetAabbMinData(), 
                           segments.size * sizeof(float4), hipMemcpyDeviceToHost);
                float4 segMax[segments.size];
                hipMemcpy(segMax, segments.GetAabbMaxData(), 
                           segments.size * sizeof(float4), hipMemcpyDeviceToHost);

                for (int i = 0; i < segments.size; ++i){
                    int index = info[i].x;
                    int range = info[i].y;

                    float4 cpuMin[range];
                    hipMemcpy(cpuMin, aabbMin->GetDeviceData() + index, 
                               range * sizeof(float4), hipMemcpyDeviceToHost);
                    float4 cpuMax[range];
                    hipMemcpy(cpuMax, aabbMax->GetDeviceData() + index, 
                               range * sizeof(float4), hipMemcpyDeviceToHost);

                    for (int j = 1; j < range; ++j){
                        cpuMin[0] = min(cpuMin[0], cpuMin[j]);
                        cpuMax[0] = max(cpuMax[0], cpuMax[j]);
                    }
                    
                    if (cpuMin[0].x != segMin[i].x || cpuMin[0].y != segMin[i].y || cpuMin[0].z != segMin[i].z)
                        throw Core::Exception("aabbMin error at segment " + Utils::Convert::ToString(i) +
                                              ": CPU min " + Utils::CUDA::Convert::ToString(cpuMin[0])
                                              + ", GPU min " + Utils::CUDA::Convert::ToString(segMin[i]));

                    if (cpuMax[0].x != segMax[i].x || cpuMax[0].y != segMax[i].y || cpuMax[0].z != segMax[i].z)
                        throw Core::Exception("aabbMax error at segment " + Utils::Convert::ToString(i) +
                                              ": CPU max " + Utils::CUDA::Convert::ToString(cpuMax[0])
                                              + ", GPU max " + Utils::CUDA::Convert::ToString(segMax[i]));
                    
                }

                // Do the final reduce
                int segOwner[segments.size];
                hipMemcpy(segOwner, segments.GetOwnerData(), 
                           segments.size * sizeof(int), hipMemcpyDeviceToHost);
                
                float4 cpuMin[activeRange];
                float4 cpuMax[activeRange];

                int owner0 = segOwner[0];
                float4 localMin = segMin[0];
                float4 localMax = segMax[0];
                for (int i = 1; i < segments.size; ++i){
                    int owner1 = segOwner[i];
                    if (owner0 != owner1){
                        cpuMin[owner0 - activeIndex] = localMin;
                        cpuMax[owner0 - activeIndex] = localMax;
                        owner0 = segOwner[i];
                        localMin = segMin[i];
                        localMax = segMax[i];
                    }else{
                        localMin = min(localMin, segMin[i]);
                        localMax = max(localMax, segMax[i]);
                    }
                }
                cpuMin[owner0 - activeIndex] = localMin;
                cpuMax[owner0 - activeIndex] = localMax;

#endif

                //threads = min(blocks, activeCudaDevice.maxThreadsDim[0]);
                threads = min((segments.size / 32) * 32 + 32, activeCudaDevice.maxThreadsDim[0]);
                //START_TIMER(timerID);
                logger.info << "SegmentedReduce0<<<1, " << threads << ">>>" << logger.end;
                SegmentedReduce0<<<1, threads>>>(segments.GetAabbMinData(),
                                                 segments.GetAabbMaxData(),
                                                 segments.GetOwnerData(),
                                                 nodes->GetAabbMinData(),
                                                 nodes->GetAabbMaxData());
                //PRINT_TIMER(timerID, "Segmented reduce");
                CHECK_FOR_CUDA_ERROR();

#if CPU_VERIFY
                float4 gpuMin[activeRange];
                hipMemcpy(gpuMin, nodes->GetAabbMinData() + activeIndex,
                           activeRange * sizeof(float4), hipMemcpyDeviceToHost);
                float4 gpuMax[activeRange];
                hipMemcpy(gpuMax, nodes->GetAabbMaxData() + activeIndex,
                           activeRange * sizeof(float4), hipMemcpyDeviceToHost);
                for (int i = 0; i < activeRange; ++i){
                    if (cpuMin[i].x != gpuMin[i].x || cpuMin[i].y != gpuMin[i].y || cpuMin[i].z != gpuMin[i].z)
                        throw Core::Exception("aabbMin error at node " + Utils::Convert::ToString(i + activeIndex) +
                                              ": CPU min " + Utils::CUDA::Convert::ToString(cpuMin[i])
                                              + ", GPU min " + Utils::CUDA::Convert::ToString(gpuMin[i]));

                    if (cpuMax[i].x != gpuMax[i].x || cpuMax[i].y != gpuMax[i].y || cpuMax[i].z != gpuMax[i].z)
                        throw Core::Exception("aabbMax error at node " + Utils::Convert::ToString(i + activeIndex) +
                                              ": CPU max " + Utils::CUDA::Convert::ToString(cpuMax[i])
                                              + ", GPU max " + Utils::CUDA::Convert::ToString(gpuMax[i]));
                }
#endif

                // Calc splitting planes.
                Calc1DKernelDimensions(activeRange, blocks, threads);
                CalcUpperNodeSplitInfo<<<blocks, threads>>>(nodes->GetAabbMinData() + activeIndex,
                                                            nodes->GetAabbMaxData() + activeIndex,
                                                            nodes->GetSplitPositionData() + activeIndex,
                                                            nodes->GetInfoData() + activeIndex);
                CHECK_FOR_CUDA_ERROR();
            }

            void TriangleMap::CreateChildren(int activeIndex, int activeRange,
                                             int &childrenCreated){
                unsigned int blocks = NextPow2(segments.size), threads = Segments::SEGMENT_SIZE;

                splitSide->Extend(triangles * 2, false);
                splitAddr->Extend(triangles * 2 + 1, false);
                leafSide->Extend(triangles * 2, false);
                leafAddr->Extend(triangles * 2 + 1, false);
                childSize->Extend(activeRange, false);
                nodes->Extend(nodes->size + activeRange * 2);

                //START_TIMER(timerID);
                SetSplitSide<<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                  segments.GetOwnerData(),
                                                  nodes->GetInfoData(),
                                                  nodes->GetSplitPositionData(),
                                                  aabbMin->GetDeviceData(),
                                                  aabbMax->GetDeviceData(),
                                                  splitSide->GetDeviceData());
                //PRINT_TIMER(timerID, "Set split Side");
                CHECK_FOR_CUDA_ERROR();

                cudppScan(scanHandle, splitAddr->GetDeviceData(), splitSide->GetDeviceData(), triangles * 2 + 1);
                //logger.info << "splitAddr " << Convert::ToString(splitAddr->GetDeviceData(), 20) << logger.end;

#ifdef CPU_VERIFY
                CheckSplits();
#endif

                int newTriangles;
                hipMemcpy(&newTriangles, splitAddr->GetDeviceData() + triangles * 2, sizeof(int), hipMemcpyDeviceToHost);
                //logger.info << "new triangles " << newTriangles << logger.end;
                CHECK_FOR_CUDA_ERROR();
                
                if (newTriangles < triangles)
                    throw Exception("New triangles amount " + Utils::Convert::ToString(newTriangles) + " was below old. WTF");
                
                bool createdLeafs = false;
                hipMemcpyToSymbol(HIP_SYMBOL(d_createdLeafs), &createdLeafs, sizeof(bool));

                unsigned int hatte, traade;
                Calc1DKernelDimensions(activeRange, hatte, traade);
                CalcNodeChildSize<<<hatte, traade>>>(nodes->GetPrimitiveInfoData() + activeIndex,
                                                   splitAddr->GetDeviceData(),
                                                   childSize->GetDeviceData());
                CHECK_FOR_CUDA_ERROR();
                hipMemcpyFromSymbol(&createdLeafs, HIP_SYMBOL(d_createdLeafs), sizeof(bool));

                /*
                EmptySpaceSplits<<<hatte, traade>>>(nodes->GetAabbMinData(),
                                                    nodes->GetAabbMaxData(),
                                                    nodes->GetInfoData(),
                                                    nodes->GetSplitPositionData(),
                                                    nodes->GetParentData(),
                                                    emptySpaceSplits->GetDeviceData());
                CHECK_FOR_CUDA_ERROR();
                cudppScan(scanHandle, emptySpaceAddrs->GetDeviceData(), emptySpaceSplits->GetDeviceData(), activeRange+1);

                logger.info << "Empty space splits: " << Convert::ToString(emptySpaceSplits->GetDeviceData(), activeRange) << logger.end;
                */

                if (createdLeafs){
                    //logger.info << "Created leafs" << logger.end;

                    SetPrimitiveLeafSide<<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                              segments.GetOwnerData(),
                                                              childSize->GetDeviceData(),
                                                              splitSide->GetDeviceData(),
                                                              leafSide->GetDeviceData());
                    CHECK_FOR_CUDA_ERROR();
                    
                    cudppScan(scanHandle, leafAddr->GetDeviceData(), leafSide->GetDeviceData(), triangles * 2 + 1);

                    /*
                    logger.info << "SetPrimitiveLeafSide<<<" << blocks << ", " << Segments::SEGMENT_SIZE << ">>>" << logger.end;
                    logger.info << "Segments primitive info: " << Utils::CUDA::Convert::ToString(segments.GetPrimitiveInfoData(), segments.size) << logger.end;
                    logger.info << "Segments owner: " << Utils::CUDA::Convert::ToString(segments.GetOwnerData(), segments.size) << logger.end;
                    logger.info << "Child sizes: " << Utils::CUDA::Convert::ToString(childSize->GetDeviceData(), activeRange) << logger.end;
                    logger.info << "Split side: " << Utils::CUDA::Convert::ToString(splitSide->GetDeviceData()+triangles-100, 200) << logger.end;
                    logger.info << "===" << logger.end;
                    logger.info << "Leaf side: " << Utils::CUDA::Convert::ToString(leafSide->GetDeviceData()+triangles-100, 200) << logger.end;
                    logger.info << "Leaf addr: " << Utils::CUDA::Convert::ToString(leafAddr->GetDeviceData()+triangles-100, 200) << logger.end;
                    */

                    int leafTriangles;
                    hipMemcpy(&leafTriangles, leafAddr->GetDeviceData() + triangles * 2, sizeof(int), hipMemcpyDeviceToHost);
                    //logger.info << "leaf triangles: " << leafTriangles << logger.end;
                    
                    newTriangles -= leafTriangles;

                    tempAabbMin->Extend(newTriangles);
                    tempAabbMax->Extend(newTriangles);
                    resultMax->Extend(upperLeafPrimitives + leafTriangles);
                    resultMin->Extend(upperLeafPrimitives + leafTriangles);

                    SplitTriangles<<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                        segments.GetOwnerData(),
                                                        nodes->GetInfoData(),
                                                        nodes->GetSplitPositionData(),
                                                        splitSide->GetDeviceData(),
                                                        splitAddr->GetDeviceData(),
                                                        leafSide->GetDeviceData(),
                                                        leafAddr->GetDeviceData(),
                                                        aabbMin->GetDeviceData(),
                                                        aabbMax->GetDeviceData(),
                                                        tempAabbMin->GetDeviceData(),
                                                        tempAabbMax->GetDeviceData(),
                                                        resultMin->GetDeviceData() + upperLeafPrimitives,
                                                        resultMax->GetDeviceData() + upperLeafPrimitives);
                    CHECK_FOR_CUDA_ERROR();
                    std::swap(aabbMin, tempAabbMin);
                    std::swap(aabbMax, tempAabbMax);
                    
                    // @TODO handle leaf nodes, probably add indices
                    // to an array for future processing.

                    MarkNodeLeafs<<<hatte, traade>>>(childSize->GetDeviceData(),
                                                     leafSide->GetDeviceData());
                    CHECK_FOR_CUDA_ERROR();
                    cudppScan(scanHandle, splitSide->GetDeviceData(), leafSide->GetDeviceData(), activeRange * 2 + 1);
                    CHECK_FOR_CUDA_ERROR();
                    
                    int leafNodes;
                    hipMemcpy(&leafNodes, splitSide->GetDeviceData() + activeRange * 2, sizeof(int), hipMemcpyDeviceToHost);
                    hipMemcpyToSymbol(HIP_SYMBOL(d_leafNodes), splitSide->GetDeviceData() + activeRange * 2, sizeof(int), 0, hipMemcpyDeviceToDevice);
                    //logger.info << "leaf nodes: " << leafNodes << logger.end;

                    /*                                        
                    logger.info << "CreateChildren<<<" << hatte << ", " << traade << ">>>" << logger.end;
                    logger.info << "primitive info " << Convert::ToString(nodes->GetPrimitiveInfoData() + activeIndex, activeRange) << logger.end;
                    logger.info << "child size " << Convert::ToString(childSize->GetDeviceData(), activeRange) << logger.end;
                    logger.info << "Node leaf addrs " << Convert::ToString(splitSide->GetDeviceData(), activeRange * 2 + 1) << logger.end;
                    */

                    Kernels::CreateChildren
                        <<<hatte, traade>>>(nodes->GetPrimitiveInfoData(),
                                            childSize->GetDeviceData(),
                                            splitAddr->GetDeviceData(),
                                            leafAddr->GetDeviceData(),
                                            splitSide->GetDeviceData(),
                                            nodes->GetLeftData(),
                                            nodes->GetRightData(),
                                            nodes->GetParentData(),
                                            upperLeafPrimitives);
                    CHECK_FOR_CUDA_ERROR();

                    upperLeafPrimitives += leafTriangles;
                    triangles = newTriangles;
                    nodes->size += activeRange * 2;
                    childrenCreated = activeRange * 2 - leafNodes;

                    upperNodeLeafList->Extend(upperNodeLeafs + leafNodes);
                    Calc1DKernelDimensions(leafNodes, blocks, threads);
                    int leafIndex = nodes->size - activeRange * 2;
                    //logger.info << "leaf index " << leafIndex << logger.end;
                    MarkLeafNodes
                        <<<blocks, threads>>>(upperNodeLeafList->GetDeviceData() + upperNodeLeafs, 
                                              nodes->GetInfoData() + leafIndex,
                                              leafIndex, leafNodes);
                    upperNodeLeafs += leafNodes;
                    
                    //logger.info << "UpperNode Leafs: " << upperNodeLeafs << logger.end;
                    
                }else{
                    logger.info << "No leafs created. Split resulted in " << newTriangles << " triangles."  << logger.end;

                    tempAabbMin->Extend(newTriangles);
                    tempAabbMax->Extend(newTriangles);

                    Kernels::CreateChildren
                        <<<hatte, traade>>>(nodes->GetPrimitiveInfoData(),
                                            childSize->GetDeviceData(),
                                            splitAddr->GetDeviceData(),
                                            nodes->GetLeftData(),
                                            nodes->GetRightData(),
                                            nodes->GetParentData());
                    CHECK_FOR_CUDA_ERROR();

                    //logger.info << "Left " << Utils::CUDA::Convert::ToString(nodes->GetLeftData() + activeIndex, activeRange) << logger.end;
                    //logger.info << "Right " << Utils::CUDA::Convert::ToString(nodes->GetRightData() + activeIndex, activeRange) << logger.end;
                    //logger.info << "Children primitive info: " << Utils::CUDA::Convert::ToString(nodes->GetPrimitiveInfoData() + activeIndex + activeRange, activeRange * 2) << logger.end;

                    SplitTriangles<<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                        segments.GetOwnerData(),
                                                        nodes->GetInfoData(),
                                                        nodes->GetSplitPositionData(),
                                                        splitSide->GetDeviceData(),
                                                        splitAddr->GetDeviceData(),
                                                        aabbMin->GetDeviceData(),
                                                        aabbMax->GetDeviceData(),
                                                        tempAabbMin->GetDeviceData(),
                                                        tempAabbMax->GetDeviceData());
                    CHECK_FOR_CUDA_ERROR();

                    std::swap(aabbMin, tempAabbMin);
                    std::swap(aabbMax, tempAabbMax);
                    
                    nodes->size += activeRange * 2;
                    childrenCreated = activeRange * 2;
                    triangles = newTriangles;
                }

#if CPU_VERIFY
                // Check that all primitive bounding boxes are tight or inside the primitive
                
                float4 primAabbMin[triangles];
                hipMemcpy(primAabbMin, aabbMin->GetDeviceData(), triangles * sizeof(float4), hipMemcpyDeviceToHost);
                float4 primAabbMax[triangles];
                hipMemcpy(primAabbMax, aabbMax->GetDeviceData(), triangles * sizeof(float4), hipMemcpyDeviceToHost);
                CHECK_FOR_CUDA_ERROR();
                for (int i = 0; i < triangles; ++i){
                    int index = primAabbMin[i].w;
                    float4 p0, p1, p2;
                    hipMemcpy(&p0, geom->p0->GetDeviceData() + index, sizeof(float4), hipMemcpyDeviceToHost);
                    hipMemcpy(&p1, geom->p1->GetDeviceData() + index, sizeof(float4), hipMemcpyDeviceToHost);
                    hipMemcpy(&p2, geom->p2->GetDeviceData() + index, sizeof(float4), hipMemcpyDeviceToHost);

                    float4 aabbMin = min(p0, min(p1, p2));
                    float4 aabbMax = max(p0, max(p1, p2));

                    if (primAabbMin[i].x < aabbMin.x || 
                        primAabbMin[i].y < aabbMin.y || 
                        primAabbMin[i].z < aabbMin.z ||
                        aabbMax.x < primAabbMax[i].x ||
                        aabbMax.y < primAabbMax[i].y ||
                        aabbMax.z < primAabbMax[i].z)
                        throw Exception("Element " + Utils::Convert::ToString(i) + 
                                        " with cornors " + Convert::ToString(p0) +
                                        ", " + Convert::ToString(p1) + " and " + Convert::ToString(p2) +
                                        " is not strictly contained in aabb " + Convert::ToString(primAabbMin[i]) +
                                        " -> " + Convert::ToString(primAabbMax[i]));
                }
                CHECK_FOR_CUDA_ERROR();

                // Check that the nodes aabb cover all their respective primitives.
                for (int i = activeIndex; i < activeIndex + activeRange; ++i){
                    float4 parentAabbMin, parentAabbMax;
                    hipMemcpy(&parentAabbMin, nodes->GetAabbMinData() + i, sizeof(float4), hipMemcpyDeviceToHost);
                    hipMemcpy(&parentAabbMax, nodes->GetAabbMaxData() + i, sizeof(float4), hipMemcpyDeviceToHost);

                    CheckUpperNode(i, parentAabbMin, parentAabbMax, activeRange);
                }
#endif

            }

            void TriangleMap::CheckUpperNode(int index, float4 calcedAabbMin, float4 calcedAabbMax, int activeRange){
                //logger.info << "Checking node " << index << logger.end;
                char axis;
                hipMemcpy(&axis, nodes->GetInfoData() + index, sizeof(char), hipMemcpyDeviceToHost);
                
                if (axis == KDNode::LEAF){
                    CheckUpperLeaf(index, calcedAabbMin, calcedAabbMax);                    
                }else{
                    float splitPos;
                    hipMemcpy(&splitPos, nodes->GetSplitPositionData() + index, sizeof(float), hipMemcpyDeviceToHost);
                    
                    int leftIndex;
                    hipMemcpy(&leftIndex, nodes->GetLeftData() + index, sizeof(int), hipMemcpyDeviceToHost);

                    int leftParent;
                    hipMemcpy(&leftParent, nodes->GetParentData() + leftIndex, sizeof(int), hipMemcpyDeviceToHost);
                        
                    if (leftParent != index)
                        throw Exception("Node " + Utils::Convert::ToString(leftIndex) +
                                        "'s parent " + Utils::Convert::ToString(leftParent) +
                                        " does not match actual parent " + Utils::Convert::ToString(index));

                    float4 leftAabbMin = calcedAabbMin;
                    float4 leftAabbMax = make_float4(axis == KDNode::X ? splitPos : calcedAabbMax.x,
                                                     axis == KDNode::Y ? splitPos : calcedAabbMax.y,
                                                     axis == KDNode::Z ? splitPos : calcedAabbMax.z,
                                                     calcedAabbMax.w);

                    if (leftIndex < nodes->size - 2 * activeRange)
                        CheckUpperNode(leftIndex, leftAabbMin, leftAabbMax, activeRange);
                    else
                        CheckUpperLeaf(leftIndex, leftAabbMin, leftAabbMax);

                    int rightIndex;
                    hipMemcpy(&rightIndex, nodes->GetRightData() + index, sizeof(int), hipMemcpyDeviceToHost);
                        
                    int rightParent;
                    hipMemcpy(&rightParent, nodes->GetParentData() + rightIndex, sizeof(int), hipMemcpyDeviceToHost);

                    if (rightParent != index)
                        throw Exception("Node " + Utils::Convert::ToString(rightIndex) +
                                        "'s parent " + Utils::Convert::ToString(rightParent) +
                                        " does not match actual parent " + Utils::Convert::ToString(index));
                        
                    float4 rightAabbMin = make_float4(axis == KDNode::X ? splitPos : calcedAabbMin.x,
                                                      axis == KDNode::Y ? splitPos : calcedAabbMin.y,
                                                      axis == KDNode::Z ? splitPos : calcedAabbMin.z,
                                                      calcedAabbMin.w);
                    float4 rightAabbMax = calcedAabbMax;

                    if (rightIndex < nodes->size - 2 * activeRange)
                        CheckUpperNode(rightIndex, rightAabbMin, rightAabbMax, activeRange);
                    else
                        CheckUpperLeaf(rightIndex, rightAabbMin, rightAabbMax);
                }                
            }

            void TriangleMap::CheckUpperLeaf(int index, float4 calcedAabbMin, float4 calcedAabbMax){
                //logger.info << "Node " << index << " is a leaf" << logger.end;
                int2 primInfo;
                hipMemcpy(&primInfo, nodes->GetPrimitiveInfoData() + index, sizeof(int2), hipMemcpyDeviceToHost);
                CHECK_FOR_CUDA_ERROR();
                
                bool isLeaf = primInfo.y < TriangleNode::MAX_LOWER_SIZE;
                for (int j = primInfo.x; j < primInfo.x + primInfo.y; ++j){
                    float4 primMin, primMax;
                    if (isLeaf){
                        hipMemcpy(&primMin, resultMin->GetDeviceData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                        hipMemcpy(&primMax, resultMax->GetDeviceData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                    }else{
                        hipMemcpy(&primMin, aabbMin->GetDeviceData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                        hipMemcpy(&primMax, aabbMax->GetDeviceData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                    }
                    CHECK_FOR_CUDA_ERROR();
                            
                    if (!aabbContains(calcedAabbMin, calcedAabbMax, primMin))
                        throw Core::Exception("primitive  " + Utils::Convert::ToString(j) + 
                                              "'s min " + Convert::ToString(primMin) +
                                              " not included in node " + Utils::Convert::ToString(index) +
                                              "'s aabb " + Convert::ToString(calcedAabbMin) +
                                              " -> " + Convert::ToString(calcedAabbMax));

                    if (!aabbContains(calcedAabbMin, calcedAabbMax, primMax))
                        throw Core::Exception("primitive  " + Utils::Convert::ToString(j) + 
                                              "'s max " + Convert::ToString(primMax) +
                                              " not included in node " + Utils::Convert::ToString(index) +
                                              "'s aabb " + Convert::ToString(calcedAabbMin) +
                                              " -> " + Convert::ToString(calcedAabbMax));
                }
            }

            void TriangleMap::CheckSplits(){
                int sides[triangles * 2];
                hipMemcpy(sides, splitSide->GetDeviceData(), triangles * 2 * sizeof(int), hipMemcpyDeviceToHost);
                CHECK_FOR_CUDA_ERROR();

                int addrs[triangles * 2];
                hipMemcpy(addrs, splitAddr->GetDeviceData(), (triangles * 2 + 1) * sizeof(int), hipMemcpyDeviceToHost);
                CHECK_FOR_CUDA_ERROR();

                //int prims = 0;
                for (int i = 0; i < triangles; ++i){
                    // Check that a bounding box is at least assigned to one side.
                    //prims += sides[i] + sides[triangles + i];
                    if (sides[i] + sides[triangles + i] == 0)
                        throw Exception("Bounding box " + Utils::Convert::ToString(i) +
                                        "was neither left nor right.");
                }

                int prims = 0;
                for (int i = 1; i < triangles * 2 + 1; ++i){
                    prims += sides[i-1];
                    if (prims != addrs[i])
                        throw Exception("Stuff went wrong at bounding box " + Utils::Convert::ToString(i));
                }

                //logger.info << "New prims " << prims << logger.end;
            }

        }
    }
}
