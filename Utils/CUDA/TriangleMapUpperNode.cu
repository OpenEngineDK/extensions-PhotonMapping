#include "hip/hip_runtime.h"
// Triangle map class for CUDA
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/TriangleMap.h>
#include <Utils/CUDA/Utils.h>
#include <Utils/CUDA/Convert.h>

#include <Utils/CUDA/Kernels/PhotonMapDeviceVars.h>
#include <Utils/CUDA/Kernels/TriangleUpper.h>
#include <Utils/CUDA/Kernels/TriangleUpperSegment.h>
#include <Utils/CUDA/Kernels/ReduceSegments.h>
#include <Utils/CUDA/Kernels/TriangleUpperChildren.h>

namespace OpenEngine {
    namespace Utils {
        namespace CUDA {

            using namespace Kernels;


            void TriangleMap::CreateUpperNodes(){
                int activeIndex = 0, activeRange = 1;
                int childrenCreated;
                
                upperNodeLeafs = upperLeafPrimitives = 0;

                // Setup root node!
                int2 i = make_int2(0, triangles);
                hipMemcpy(upperNodes->GetPrimitiveInfoData(), &i, sizeof(int2), hipMemcpyHostToDevice);
                upperNodes->size = 1;

                // Setup bounding box info
                unsigned int blocks, threads;
                Calc1DKernelDimensions(triangles, blocks, threads);
                AddIndexToAabb<<<blocks, threads>>>(geom->GetAabbMinData(), geom->GetSize(), aabbMin->GetDeviceData());
                // @OPT Just switch the arrays.
                hipMemcpy(aabbMax->GetDeviceData(), geom->GetAabbMaxData(), 
                           triangles * sizeof(float4), hipMemcpyDeviceToDevice);
                CHECK_FOR_CUDA_ERROR();                

                while (activeRange > 0){
                    ProcessUpperNodes(activeIndex, activeRange, 
                                      childrenCreated);

                    activeIndex = upperNodes->size - childrenCreated;
                    activeRange = childrenCreated;
                }

            }

            void TriangleMap::ProcessUpperNodes(int activeIndex, int activeRange, 
                                                int &childrenCreated){

                logger.info << "=== Process " << activeRange << " Upper Nodes Starting at " << activeIndex << " === with " << triangles << " triangles" << logger.end;

                // Copy bookkeeping to symbols
                hipMemcpyToSymbol(HIP_SYMBOL(d_activeNodeIndex), &activeIndex, sizeof(int));
                hipMemcpyToSymbol(HIP_SYMBOL(d_activeNodeRange), &activeRange, sizeof(int));
                hipMemcpyToSymbol(HIP_SYMBOL(d_triangles), &triangles, sizeof(int));
                CHECK_FOR_CUDA_ERROR();

                Segment(activeIndex, activeRange);
                CHECK_FOR_CUDA_ERROR();

                // Calculate aabb
                ReduceAabb(activeIndex, activeRange);
                CHECK_FOR_CUDA_ERROR();

                // Calculate children placement
                CreateChildren(activeIndex, activeRange, childrenCreated);
            }

            void TriangleMap::Segment(int activeIndex, int activeRange){
                if (nodeSegments->GetSize() < (unsigned int)activeRange+1)
                    nodeSegments->Resize(activeRange+1);

                unsigned int blocks, threads;
                Calc1DKernelDimensions(activeRange, blocks, threads);
                NodeSegments<<<blocks, threads>>>(upperNodes->GetPrimitiveInfoData() + activeIndex,
                                                  nodeSegments->GetDeviceData());

                CHECK_FOR_CUDA_ERROR();
                cudppScan(scanHandle, nodeSegments->GetDeviceData(), nodeSegments->GetDeviceData(), activeRange+1);

                int amountOfSegments;
                hipMemcpy(&amountOfSegments, nodeSegments->GetDeviceData() + activeRange, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpyToSymbol(HIP_SYMBOL(d_segments), nodeSegments->GetDeviceData() + activeRange, sizeof(int), 0, hipMemcpyDeviceToDevice);
                CHECK_FOR_CUDA_ERROR();

                if (segments.maxSize < amountOfSegments)
                    segments.Resize(amountOfSegments);
                segments.size = amountOfSegments;

                hipMemset(segments.GetOwnerData(), 0, amountOfSegments * sizeof(int));
                MarkOwnerStart<<<blocks, threads>>>(segments.GetOwnerData(),
                                                    nodeSegments->GetDeviceData());
                CHECK_FOR_CUDA_ERROR();

                cudppScan(scanInclHandle, segments.GetOwnerData(), segments.GetOwnerData(), amountOfSegments);

                Calc1DKernelDimensions(amountOfSegments, blocks, threads);
                CalcSegmentPrimitives<<<blocks, threads>>>(segments.GetOwnerData(),
                                                           nodeSegments->GetDeviceData(),
                                                           upperNodes->GetPrimitiveInfoData(),
                                                           segments.GetPrimitiveInfoData());
                CHECK_FOR_CUDA_ERROR();
            }

            void TriangleMap::ReduceAabb(int activeIndex, int activeRange){
                
                // Reduce aabb pr segment
                unsigned int blocks = NextPow2(segments.size);
                unsigned int threads = Segments::SEGMENT_SIZE;
                unsigned int memSize = 2 * sizeof(float4) * segments.SEGMENT_SIZE;

                //START_TIMER(timerID);
                ReduceSegments<<<blocks, threads, memSize>>>(segments.GetPrimitiveInfoData(),
                                                             aabbMin->GetDeviceData(), aabbMax->GetDeviceData(),
                                                             segments.GetAabbMinData(), segments.GetAabbMaxData());
                //PRINT_TIMER(timerID, "ReduceSegments");
                CHECK_FOR_CUDA_ERROR();

#if CPU_VERIFY
                int2 info[segments.size];
                hipMemcpy(info, segments.GetPrimitiveInfoData(), 
                           segments.size * sizeof(int2), hipMemcpyDeviceToHost);

                float4 segMin[segments.size];
                hipMemcpy(segMin, segments.GetAabbMinData(), 
                           segments.size * sizeof(float4), hipMemcpyDeviceToHost);
                float4 segMax[segments.size];
                hipMemcpy(segMax, segments.GetAabbMaxData(), 
                           segments.size * sizeof(float4), hipMemcpyDeviceToHost);

                for (int i = 0; i < segments.size; ++i){
                    int index = info[i].x;
                    int range = info[i].y;

                    float4 cpuMin[range];
                    hipMemcpy(cpuMin, aabbMin->GetDeviceData() + index, 
                               range * sizeof(float4), hipMemcpyDeviceToHost);
                    float4 cpuMax[range];
                    hipMemcpy(cpuMax, aabbMax->GetDeviceData() + index, 
                               range * sizeof(float4), hipMemcpyDeviceToHost);

                    for (int j = 1; j < range; ++j){
                        cpuMin[0] = min(cpuMin[0], cpuMin[j]);
                        cpuMax[0] = max(cpuMax[0], cpuMax[j]);
                    }
                    
                    if (cpuMin[0].x != segMin[i].x || cpuMin[0].y != segMin[i].y || cpuMin[0].z != segMin[i].z)
                        throw Core::Exception("aabbMin error at segment " + Utils::Convert::ToString(i) +
                                              ": CPU min " + Utils::CUDA::Convert::ToString(cpuMin[0])
                                              + ", GPU min " + Utils::CUDA::Convert::ToString(segMin[i]));

                    if (cpuMax[0].x != segMax[i].x || cpuMax[0].y != segMax[i].y || cpuMax[0].z != segMax[i].z)
                        throw Core::Exception("aabbMax error at segment " + Utils::Convert::ToString(i) +
                                              ": CPU max " + Utils::CUDA::Convert::ToString(cpuMax[0])
                                              + ", GPU max " + Utils::CUDA::Convert::ToString(segMax[i]));
                    
                }

                // Do the final reduce
                int segOwner[segments.size];
                hipMemcpy(segOwner, segments.GetOwnerData(), 
                           segments.size * sizeof(int), hipMemcpyDeviceToHost);
                
                float4 cpuMin[activeRange];
                float4 cpuMax[activeRange];

                int owner0 = segOwner[0];
                float4 localMin = segMin[0];
                float4 localMax = segMax[0];
                for (int i = 1; i < segments.size; ++i){
                    int owner1 = segOwner[i];
                    if (owner0 != owner1){
                        cpuMin[owner0 - activeIndex] = localMin;
                        cpuMax[owner0 - activeIndex] = localMax;
                        owner0 = segOwner[i];
                        localMin = segMin[i];
                        localMax = segMax[i];
                    }else{
                        localMin = min(localMin, segMin[i]);
                        localMax = max(localMax, segMax[i]);
                    }
                }
                cpuMin[owner0 - activeIndex] = localMin;
                cpuMax[owner0 - activeIndex] = localMax;

#endif

                //threads = min(blocks, activeCudaDevice.maxThreadsDim[0]);
                threads = min((segments.size / 32) * 32 + 32, activeCudaDevice.maxThreadsDim[0]);
                //START_TIMER(timerID);
                SegmentedReduce0<<<1, threads>>>(segments.GetAabbMinData(),
                                                 segments.GetAabbMaxData(),
                                                 segments.GetOwnerData(),
                                                 upperNodes->GetAabbMinData(),
                                                 upperNodes->GetAabbMaxData());
                //PRINT_TIMER(timerID, "Segmented reduce");
                CHECK_FOR_CUDA_ERROR();

                /*                
                if (activeIndex == 92){
                    for (int i = 0; i < segments.size; ++i)
                        logger.info << "Segment " << i << "'s owner is " << segOwner[i] << " with max " << Convert::ToString(segMax[i]) << logger.end;                        
                }
                */

#if CPU_VERIFY
                float4 gpuMin[activeRange];
                hipMemcpy(gpuMin, upperNodes->GetAabbMinData() + activeIndex,
                           activeRange * sizeof(float4), hipMemcpyDeviceToHost);
                float4 gpuMax[activeRange];
                hipMemcpy(gpuMax, upperNodes->GetAabbMaxData() + activeIndex,
                           activeRange * sizeof(float4), hipMemcpyDeviceToHost);
                for (int i = 0; i < activeRange; ++i){
                    if (cpuMin[i].x != gpuMin[i].x || cpuMin[i].y != gpuMin[i].y || cpuMin[i].z != gpuMin[i].z)
                        throw Core::Exception("aabbMin error at node " + Utils::Convert::ToString(i + activeIndex) +
                                              ": CPU min " + Utils::CUDA::Convert::ToString(cpuMin[i])
                                              + ", GPU min " + Utils::CUDA::Convert::ToString(gpuMin[i]));

                    if (cpuMax[i].x != gpuMax[i].x || cpuMax[i].y != gpuMax[i].y || cpuMax[i].z != gpuMax[i].z)
                        throw Core::Exception("aabbMax error at node " + Utils::Convert::ToString(i + activeIndex) +
                                              ": CPU max " + Utils::CUDA::Convert::ToString(cpuMax[i])
                                              + ", GPU max " + Utils::CUDA::Convert::ToString(gpuMax[i]));
                }
#endif

                // Calc splitting planes.
                Calc1DKernelDimensions(activeRange, blocks, threads);
                CalcUpperNodeSplitInfo<<<blocks, threads>>>(upperNodes->GetAabbMinData() + activeIndex,
                                                            upperNodes->GetAabbMaxData() + activeIndex,
                                                            upperNodes->GetSplitPositionData() + activeIndex,
                                                            upperNodes->GetInfoData() + activeIndex);
                CHECK_FOR_CUDA_ERROR();
            }

            void TriangleMap::CreateChildren(int activeIndex, int activeRange,
                                             int &childrenCreated){
                unsigned int blocks = NextPow2(segments.size), threads = Segments::SEGMENT_SIZE;

                /*
                if (activeIndex > 0){
                    // Do empty space splitting and update activeIndex and activeRange
                    throw Exception("Empty space splitting not implemented.");
                }
                */

                if (splitSide->GetSize() < (unsigned int)triangles * 2) splitSide->Resize(triangles * 2, false);
                if (splitAddr->GetSize() < (unsigned int)triangles * 2 + 1) splitAddr->Resize(triangles * 2 + 1, false);
                if (leafSide->GetSize() < (unsigned int)triangles * 2) leafSide->Resize(triangles * 2, false);
                if (leafAddr->GetSize() < (unsigned int)triangles * 2 + 1) leafAddr->Resize(triangles * 2 + 1, false);
                if (childSize->GetSize() < (unsigned int)activeRange) childSize->Resize(activeRange, false);
                if (upperNodes->maxSize < upperNodes->size + activeRange * 2) upperNodes->Resize(upperNodes->size + activeRange * 2);

                //START_TIMER(timerID);
                SetSplitSide<<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                  segments.GetOwnerData(),
                                                  upperNodes->GetInfoData(),
                                                  upperNodes->GetSplitPositionData(),
                                                  aabbMin->GetDeviceData(),
                                                  aabbMax->GetDeviceData(),
                                                  splitSide->GetDeviceData());
                //PRINT_TIMER(timerID, "Set split Side");
                CHECK_FOR_CUDA_ERROR();

                cudppScan(scanHandle, splitAddr->GetDeviceData(), splitSide->GetDeviceData(), triangles * 2 + 1);

                int newTriangles;
                hipMemcpy(&newTriangles, splitAddr->GetDeviceData() + triangles * 2, sizeof(int), hipMemcpyDeviceToHost);
                //logger.info << "new triangles " << newTriangles << logger.end;
                
                bool createdLeafs = false;
                hipMemcpyToSymbol(HIP_SYMBOL(d_createdLeafs), &createdLeafs, sizeof(bool));

                unsigned int hatte, traade;
                Calc1DKernelDimensions(activeRange, hatte, traade);
                CalcNodeChildSize<<<hatte, traade>>>(upperNodes->GetPrimitiveInfoData() + activeIndex,
                                                   splitAddr->GetDeviceData(),
                                                   childSize->GetDeviceData());
                CHECK_FOR_CUDA_ERROR();
                hipMemcpyFromSymbol(&createdLeafs, HIP_SYMBOL(d_createdLeafs), sizeof(bool));

                if (createdLeafs){
                    //logger.info << "Created leafs" << logger.end;

                    SetPrimitiveLeafSide<<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                              segments.GetOwnerData(),
                                                              childSize->GetDeviceData(),
                                                              splitSide->GetDeviceData(),
                                                              leafSide->GetDeviceData());
                    CHECK_FOR_CUDA_ERROR();
                    
                    cudppScan(scanHandle, leafAddr->GetDeviceData(), leafSide->GetDeviceData(), triangles * 2 + 1);

                    /*
                    logger.info << "SetPrimitiveLeafSide<<<" << blocks << ", " << Segments::SEGMENT_SIZE << ">>>" << logger.end;
                    logger.info << "Segments primitive info: " << Utils::CUDA::Convert::ToString(segments.GetPrimitiveInfoData(), segments.size) << logger.end;
                    logger.info << "Segments owner: " << Utils::CUDA::Convert::ToString(segments.GetOwnerData(), segments.size) << logger.end;
                    logger.info << "Child sizes: " << Utils::CUDA::Convert::ToString(childSize->GetDeviceData(), activeRange) << logger.end;
                    logger.info << "Split side: " << Utils::CUDA::Convert::ToString(splitSide->GetDeviceData()+triangles-100, 200) << logger.end;
                    logger.info << "===" << logger.end;
                    logger.info << "Leaf side: " << Utils::CUDA::Convert::ToString(leafSide->GetDeviceData()+triangles-100, 200) << logger.end;
                    logger.info << "Leaf addr: " << Utils::CUDA::Convert::ToString(leafAddr->GetDeviceData()+triangles-100, 200) << logger.end;
                    */

                    int leafTriangles;
                    hipMemcpy(&leafTriangles, leafAddr->GetDeviceData() + triangles * 2, sizeof(int), hipMemcpyDeviceToHost);
                    //logger.info << "leaf triangles: " << leafTriangles << logger.end;
                    
                    newTriangles -= leafTriangles;

                    if (tempAabbMin->GetSize() < (unsigned int) newTriangles) tempAabbMin->Resize(newTriangles);
                    if (tempAabbMax->GetSize() < (unsigned int) newTriangles) tempAabbMax->Resize(newTriangles);
                    if (geom->GetAabbMin()->GetSize() < (unsigned int) upperLeafPrimitives + leafTriangles)
                        geom->GetAabbMin()->Resize(upperLeafPrimitives + leafTriangles);
                    if (geom->GetAabbMax()->GetSize() < (unsigned int) upperLeafPrimitives + leafTriangles)
                        geom->GetAabbMax()->Resize(upperLeafPrimitives + leafTriangles);

                    SplitTriangles<<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                        segments.GetOwnerData(),
                                                        upperNodes->GetInfoData(),
                                                        upperNodes->GetSplitPositionData(),
                                                        splitSide->GetDeviceData(),
                                                        splitAddr->GetDeviceData(),
                                                        leafSide->GetDeviceData(),
                                                        leafAddr->GetDeviceData(),
                                                        aabbMin->GetDeviceData(),
                                                        aabbMax->GetDeviceData(),
                                                        tempAabbMin->GetDeviceData(),
                                                        tempAabbMax->GetDeviceData(),
                                                        geom->GetAabbMinData() + upperLeafPrimitives,
                                                        geom->GetAabbMaxData() + upperLeafPrimitives);
                    CHECK_FOR_CUDA_ERROR();
                    std::swap(aabbMin, tempAabbMin);
                    std::swap(aabbMax, tempAabbMax);
                    
                    // @TODO handle leaf nodes, probably add indices
                    // to an array for future processing.

                    MarkNodeLeafs<<<hatte, traade>>>(childSize->GetDeviceData(),
                                                     leafSide->GetDeviceData());
                    CHECK_FOR_CUDA_ERROR();
                    cudppScan(scanHandle, splitSide->GetDeviceData(), leafSide->GetDeviceData(), activeRange * 2 + 1);
                    CHECK_FOR_CUDA_ERROR();
                    
                    int leafNodes;
                    hipMemcpy(&leafNodes, splitSide->GetDeviceData() + activeRange * 2, sizeof(int), hipMemcpyDeviceToHost);
                    //logger.info << "leaf nodes: " << leafNodes << logger.end;

                    /*                                        
                    logger.info << "CreateChildren<<<" << hatte << ", " << traade << ">>>" << logger.end;
                    logger.info << "primitive info " << Convert::ToString(upperNodes->GetPrimitiveInfoData() + activeIndex, activeRange) << logger.end;
                    logger.info << "child size " << Convert::ToString(childSize->GetDeviceData(), activeRange) << logger.end;
                    logger.info << "Node leaf addrs " << Convert::ToString(splitSide->GetDeviceData(), activeRange * 2 + 1) << logger.end;
                    */

                    Kernels::CreateChildren
                        <<<hatte, traade>>>(upperNodes->GetPrimitiveInfoData() + activeIndex,
                                            childSize->GetDeviceData(),
                                            splitAddr->GetDeviceData(),
                                            leafAddr->GetDeviceData(),
                                            splitSide->GetDeviceData(),
                                            upperNodes->GetLeftData() + activeIndex,
                                            upperNodes->GetRightData() + activeIndex,
                                            upperLeafPrimitives);
                    CHECK_FOR_CUDA_ERROR();

                    upperLeafPrimitives += leafTriangles;
                    triangles = newTriangles;
                    upperNodes->size += activeRange * 2;
                    childrenCreated = activeRange * 2 - leafNodes;

                    upperNodeLeafList->Extend(upperNodeLeafs + leafNodes);
                    Calc1DKernelDimensions(leafNodes, blocks, threads);
                    int leafIndex = upperNodes->size - activeRange * 2;
                    //logger.info << "leaf index " << leafIndex << logger.end;
                    MarkLeafNodes
                        <<<blocks, threads>>>(upperNodeLeafList->GetDeviceData() + upperNodeLeafs, 
                                              upperNodes->GetInfoData() + leafIndex,
                                              leafIndex, leafNodes);
                    upperNodeLeafs += leafNodes;
                    
                    //logger.info << "UpperNode Leafs: " << upperNodeLeafs << logger.end;
                    
                }else{
                    //logger.info << "No leafs created" << logger.end;

                    if (tempAabbMin->GetSize() < (unsigned int) newTriangles) tempAabbMin->Resize(newTriangles);
                    if (tempAabbMax->GetSize() < (unsigned int) newTriangles) tempAabbMax->Resize(newTriangles);

                    Kernels::CreateChildren
                        <<<hatte, traade>>>(upperNodes->GetPrimitiveInfoData() + activeIndex,
                                            childSize->GetDeviceData(),
                                            splitAddr->GetDeviceData(),
                                            upperNodes->GetLeftData() + activeIndex,
                                            upperNodes->GetRightData() + activeIndex);
                    CHECK_FOR_CUDA_ERROR();

                    //logger.info << "Left " << Utils::CUDA::Convert::ToString(upperNodes->GetLeftData() + activeIndex, activeRange) << logger.end;
                    //logger.info << "Right " << Utils::CUDA::Convert::ToString(upperNodes->GetRightData() + activeIndex, activeRange) << logger.end;
                    //logger.info << "Children primitive info: " << Utils::CUDA::Convert::ToString(upperNodes->GetPrimitiveInfoData() + activeIndex + activeRange, activeRange * 2) << logger.end;

                    SplitTriangles<<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                        segments.GetOwnerData(),
                                                        upperNodes->GetInfoData(),
                                                        upperNodes->GetSplitPositionData(),
                                                        splitSide->GetDeviceData(),
                                                        splitAddr->GetDeviceData(),
                                                        aabbMin->GetDeviceData(),
                                                        aabbMax->GetDeviceData(),
                                                        tempAabbMin->GetDeviceData(),
                                                        tempAabbMax->GetDeviceData());
                    CHECK_FOR_CUDA_ERROR();

                    std::swap(aabbMin, tempAabbMin);
                    std::swap(aabbMax, tempAabbMax);
                    
                    upperNodes->size += activeRange * 2;
                    childrenCreated = activeRange * 2;
                    triangles = newTriangles;
                }

#if CPU_VERIFY
                // Check that all primitive bounding boxes are tight or inside the primitive
                
                float4 primAabbMin[triangles];
                hipMemcpy(primAabbMin, aabbMin->GetDeviceData(), triangles * sizeof(float4), hipMemcpyDeviceToHost);
                float4 primAabbMax[triangles];
                hipMemcpy(primAabbMax, aabbMax->GetDeviceData(), triangles * sizeof(float4), hipMemcpyDeviceToHost);
                CHECK_FOR_CUDA_ERROR();
                for (int i = 0; i < triangles; ++i){
                    int index = primAabbMin[i].w;
                    float4 p0, p1, p2;
                    hipMemcpy(&p0, geom->p0->GetDeviceData() + index, sizeof(float4), hipMemcpyDeviceToHost);
                    hipMemcpy(&p1, geom->p1->GetDeviceData() + index, sizeof(float4), hipMemcpyDeviceToHost);
                    hipMemcpy(&p2, geom->p2->GetDeviceData() + index, sizeof(float4), hipMemcpyDeviceToHost);

                    float4 aabbMin = min(p0, min(p1, p2));
                    float4 aabbMax = max(p0, max(p1, p2));

                    if (primAabbMin[i].x < aabbMin.x || 
                        primAabbMin[i].y < aabbMin.y || 
                        primAabbMin[i].z < aabbMin.z ||
                        aabbMax.x < primAabbMax[i].x ||
                        aabbMax.y < primAabbMax[i].y ||
                        aabbMax.z < primAabbMax[i].z)
                        throw Exception("Element " + Utils::Convert::ToString(i) + 
                                        " with cornors " + Convert::ToString(p0) +
                                        ", " + Convert::ToString(p1) + " and " + Convert::ToString(p2) +
                                        " is not strictly contained in aabb " + Convert::ToString(primAabbMin[i]) +
                                        " -> " + Convert::ToString(primAabbMax[i]));
                }
                CHECK_FOR_CUDA_ERROR();

                // Check that the nodes aabb cover all their respective primitives.
                for (int i = activeIndex; i < activeIndex + activeRange; ++i){
                    char axis;
                    hipMemcpy(&axis, upperNodes->GetInfoData() + i, sizeof(char), hipMemcpyDeviceToHost);

                    float splitPos;
                    hipMemcpy(&splitPos, upperNodes->GetSplitPositionData() + i, sizeof(float), hipMemcpyDeviceToHost);
                        
                    float4 parentAabbMin, parentAabbMax;
                    hipMemcpy(&parentAabbMin, upperNodes->GetAabbMinData() + i, sizeof(float4), hipMemcpyDeviceToHost);
                    hipMemcpy(&parentAabbMax, upperNodes->GetAabbMaxData() + i, sizeof(float4), hipMemcpyDeviceToHost);
                        
                    int leftIndex;
                    hipMemcpy(&leftIndex, upperNodes->GetLeftData() + i, sizeof(int), hipMemcpyDeviceToHost);
                        
                    int2 leftPrimInfo;
                    hipMemcpy(&leftPrimInfo, upperNodes->GetPrimitiveInfoData() + leftIndex, sizeof(int2), hipMemcpyDeviceToHost);

                    float4 leftAabbMin = parentAabbMin;
                    float4 leftAabbMax = make_float4(axis == KDNode::X ? splitPos : parentAabbMax.x,
                                                     axis == KDNode::Y ? splitPos : parentAabbMax.y,
                                                     axis == KDNode::Z ? splitPos : parentAabbMax.z,
                                                     parentAabbMax.w);

                    bool leftIsLeaf = leftPrimInfo.y < TriangleLowerNode::MAX_SIZE;
                    for (int j = leftPrimInfo.x; j < leftPrimInfo.x + leftPrimInfo.y; ++j){
                        float4 primMin, primMax;
                        if (leftIsLeaf){
                            hipMemcpy(&primMin, geom->GetAabbMinData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                            hipMemcpy(&primMax, geom->GetAabbMaxData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                        }else{
                            hipMemcpy(&primMin, aabbMin->GetDeviceData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                            hipMemcpy(&primMax, aabbMax->GetDeviceData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                        }
                            
                        if (!aabbContains(leftAabbMin, leftAabbMax, primMin))
                            throw Core::Exception("primitive  " + Utils::Convert::ToString(j) + 
                                                  "'s min " + Utils::CUDA::Convert::ToString(primMin) +
                                                  " not included in node " + Utils::Convert::ToString(leftIndex) + 
                                                  "'s aabb " + Utils::CUDA::Convert::ToString(leftAabbMin) +
                                                  " -> " + Utils::CUDA::Convert::ToString(leftAabbMax));

                        if (!aabbContains(leftAabbMin, leftAabbMax, primMax))
                            throw Core::Exception("primitive  " + Utils::Convert::ToString(j) + 
                                                  "'s max " + Utils::CUDA::Convert::ToString(primMax) +
                                                  " not included in left aabb " + Utils::CUDA::Convert::ToString(leftAabbMin)
                                                  + " -> " + Utils::CUDA::Convert::ToString(leftAabbMax));
                    }

                    int rightIndex;
                    hipMemcpy(&rightIndex, upperNodes->GetRightData() + i, sizeof(int), hipMemcpyDeviceToHost);
                    CHECK_FOR_CUDA_ERROR();
                        
                    int2 rightPrimInfo;
                    hipMemcpy(&rightPrimInfo, upperNodes->GetPrimitiveInfoData() + rightIndex, sizeof(int2), hipMemcpyDeviceToHost);
                    CHECK_FOR_CUDA_ERROR();
                        
                    float4 rightAabbMin = make_float4(axis == KDNode::X ? splitPos : parentAabbMin.x,
                                                      axis == KDNode::Y ? splitPos : parentAabbMin.y,
                                                      axis == KDNode::Z ? splitPos : parentAabbMin.z,
                                                      parentAabbMin.w);
                    float4 rightAabbMax = parentAabbMax;

                    bool rightIsLeaf = rightPrimInfo.y < TriangleLowerNode::MAX_SIZE;
                    for (int j = rightPrimInfo.x; j < rightPrimInfo.x + rightPrimInfo.y; ++j){
                        float4 primMin, primMax;
                        if (rightIsLeaf){
                            hipMemcpy(&primMin, geom->GetAabbMinData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                            hipMemcpy(&primMax, geom->GetAabbMaxData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                        }else{
                            hipMemcpy(&primMin, aabbMin->GetDeviceData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                            hipMemcpy(&primMax, aabbMax->GetDeviceData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                        }
                        CHECK_FOR_CUDA_ERROR();
                            
                        if (!aabbContains(rightAabbMin, rightAabbMax, primMin))
                            throw Core::Exception("primitive  " + Utils::Convert::ToString(j) + 
                                                  "'s min " + Utils::CUDA::Convert::ToString(primMin) +
                                                  " not included in right aabb " + Utils::CUDA::Convert::ToString(rightAabbMin)
                                                  + " -> " + Utils::CUDA::Convert::ToString(rightAabbMax));

                        if (!aabbContains(rightAabbMin, rightAabbMax, primMax))
                            throw Core::Exception("primitive  " + Utils::Convert::ToString(j) + 
                                                  "'s max " + Utils::CUDA::Convert::ToString(primMax) +
                                                  " not included in right aabb " + Utils::CUDA::Convert::ToString(rightAabbMin)
                                                  + " -> " + Utils::CUDA::Convert::ToString(rightAabbMax));
                    }
                }
#endif

            }

        }
    }
}
