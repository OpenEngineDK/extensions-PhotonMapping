#include "hip/hip_runtime.h"
// Triangle map balanced creator interface for CUDA
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/TriangleMapBalancedCreator.h>

#include <Scene/TriangleNode.h>
#include <Utils/CUDA/Convert.h>
#include <Utils/CUDA/TriangleMap.h>
#include <Utils/CUDA/Utils.h>
#include <Utils/CUDA/IntersectionTests.h>
#include <Logging/Logger.h>

#include <Utils/CUDA/Kernels/PhotonMapDeviceVars.h>

namespace OpenEngine {    
    using namespace Resources::CUDA;
    using namespace Scene;
    namespace Utils {
        namespace CUDA {

            using namespace Kernels;

            namespace TriangleMapBalancedKernels {
#include <Utils/CUDA/Kernels/LowerTriangleMap.h>
            }
            using namespace TriangleMapBalancedKernels;

            TriangleMapBalancedCreator::TriangleMapBalancedCreator()
                : ITriangleMapCreator(), removeFalsePrimitives(true) {

                cutCreateTimer(&timerID);

                logger.info << "Create balanced lower tree creator" << logger.end;

                splitTriangleSet = new CUDADataBlock<1, KDNode::bitmap4>(1);
                childSets = new CUDADataBlock<1, KDNode::bitmap2>(1);
                splitSide = new CUDADataBlock<1, int>(1);
                splitAddr = new CUDADataBlock<1, int>(1);

                scanConfig.algorithm = CUDPP_SCAN;
                scanConfig.op = CUDPP_ADD;
                scanConfig.datatype = CUDPP_INT;
                scanConfig.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
                scanSize = 262144;
                
                CUDPPResult res = cudppPlan(&scanHandle, scanConfig, scanSize, 1, 0);
                if (CUDPP_SUCCESS != res)
                    throw Core::Exception("Error creating CUDPP scanPlan for Triangle Map SAH Creator");
            }

            TriangleMapBalancedCreator::~TriangleMapBalancedCreator(){
                if (splitTriangleSet) delete splitTriangleSet;
                if (childSets) delete childSets;
                if (splitSide) delete splitSide;
                if (splitAddr) delete splitAddr;
            }
            
            void TriangleMapBalancedCreator::Create(TriangleMap* map, 
                                                    CUDADataBlock<1, int>* upperLeafIDs){

                primMin = map->primMin;
                primMax = map->primMax;

                int activeIndex = map->nodes->GetSize(); int activeRange = upperLeafIDs->GetSize();
                int childrenCreated;

                int triangles = map->primMin->GetSize();
                hipMemcpyToSymbol(HIP_SYMBOL(d_triangles), &triangles, sizeof(int));

                //START_TIMER(timerID); 
                PreprocessLowerNodes(activeIndex, activeRange, map, upperLeafIDs);
                //PRINT_TIMER(timerID, "Preprocess lower nodes");

                START_TIMER(timerID); 
                ProcessLowerNodes(activeIndex, activeRange,
                                  map, upperLeafIDs, childrenCreated);
                
                activeIndex = map->nodes->GetSize() - childrenCreated;
                activeRange = childrenCreated;

                while (activeRange > 0){
                    ProcessLowerNodes(activeIndex, activeRange,
                                      map, NULL, childrenCreated);

                    activeIndex = map->nodes->GetSize() - childrenCreated;
                    activeRange = childrenCreated;
                }
                PRINT_TIMER(timerID, "Process lower nodes into balanced subtrees");
            }
            
            void TriangleMapBalancedCreator::PreprocessLowerNodes(int activeIndex, int activeRange, 
                                                                  TriangleMap* map, CUDADataBlock<1, int>* upperLeafIDs){
                int triangles = primMin->GetSize();
                logger.info << "=== Preprocess " << activeRange << " Lower Nodes Starting at " << activeIndex << " === with " << triangles << " indices" << logger.end;
                
                TriangleNode* nodes = map->nodes;

                splitTriangleSet->Extend(triangles * 3);
                
                unsigned int blocks, threads, smemSize;
                Calc1DKernelDimensions(activeRange, blocks, threads);
                PreprocessLeafNodes<<<blocks, threads>>>(upperLeafIDs->GetDeviceData(),
                                                         nodes->GetPrimitiveBitmapData(),
                                                         activeRange);
                CHECK_FOR_CUDA_ERROR();
                
                unsigned int smemPrThread = sizeof(float3) + sizeof(float3);
                Calc1DKernelDimensionsWithSmem(activeRange * TriangleNode::MAX_LOWER_SIZE, smemPrThread, 
                                               blocks, threads, smemSize, 256);
                CreateSplittingPlanes<<<blocks, threads, smemSize>>>
                    (upperLeafIDs->GetDeviceData(),
                     nodes->GetPrimitiveIndexData(),
                     nodes->GetPrimitiveBitmapData(),
                     primMin->GetDeviceData(), primMax->GetDeviceData(),
                     splitTriangleSet->GetDeviceData(), 
                     activeRange);
                CHECK_FOR_CUDA_ERROR();
            }
            
            void TriangleMapBalancedCreator::ProcessLowerNodes(int activeIndex, int activeRange, 
                                                               TriangleMap* map, CUDADataBlock<1, int>* upperLeafIDs,
                                                               int &childrenCreated){
                if (upperLeafIDs)
                    logger.info << "=== Process " << activeRange << " Lower Nodes from Indices ===" << logger.end;
                else
                    logger.info << "=== Process " << activeRange << " Lower Nodes Starting at " << activeIndex << " ===" << logger.end;

                TriangleNode* nodes = map->nodes;

                hipMemcpyToSymbol(HIP_SYMBOL(d_activeNodeIndex), &activeIndex, sizeof(int));
                hipMemcpyToSymbol(HIP_SYMBOL(d_activeNodeRange), &activeRange, sizeof(int));

                childSets->Extend(activeRange);
                splitSide->Extend(activeRange+1);
                splitAddr->Extend(activeRange+1);

                unsigned int blocks, threads, smemSize;
                unsigned int smemPrThread = TriangleNode::MAX_LOWER_SIZE * sizeof(float);
                Calc1DKernelDimensionsWithSmem(activeRange, smemPrThread, 
                                               blocks, threads, smemSize, 128);

                if (upperLeafIDs)
                    CalcSplit<true><<<blocks, threads>>>(upperLeafIDs->GetDeviceData(), 
                                                         nodes->GetInfoData(),
                                                         nodes->GetSplitPositionData(),
                                                         nodes->GetPrimitiveIndexData(),
                                                         nodes->GetPrimitiveBitmapData(),
                                                         primMin->GetDeviceData(),
                                                         primMax->GetDeviceData(),
                                                         splitTriangleSet->GetDeviceData(),
                                                         childSets->GetDeviceData(),
                                                         splitSide->GetDeviceData());
                else
                    CalcSplit<false><<<blocks, threads>>>(NULL, nodes->GetInfoData(),
                                                          nodes->GetSplitPositionData(),
                                                          nodes->GetPrimitiveIndexData(),
                                                          nodes->GetPrimitiveBitmapData(),
                                                          primMin->GetDeviceData(),
                                                          primMax->GetDeviceData(),
                                                          splitTriangleSet->GetDeviceData(),
                                                          childSets->GetDeviceData(),
                                                          splitSide->GetDeviceData());
                CHECK_FOR_CUDA_ERROR();

                cudppScan(scanHandle, splitAddr->GetDeviceData(), splitSide->GetDeviceData(), activeRange+1);
                CHECK_FOR_CUDA_ERROR();

                int splits;
                hipMemcpy(&splits, splitAddr->GetDeviceData() + activeRange, sizeof(int), hipMemcpyDeviceToHost);

                nodes->Extend(nodes->GetSize() + 2 * splits);

                Calc1DKernelDimensions(activeRange, blocks, threads);
                if (upperLeafIDs)
                    CreateChildren<true><<<blocks, threads>>>(upperLeafIDs->GetDeviceData(), 
                                                              splitSide->GetDeviceData(),
                                                              splitAddr->GetDeviceData(),
                                                              childSets->GetDeviceData(),
                                                              nodes->GetPrimitiveIndexData(),
                                                              nodes->GetPrimitiveBitmapData(),
                                                              nodes->GetChildrenData(),
                                                              splits);
                else
                    CreateChildren<false><<<blocks, threads>>>(NULL, splitSide->GetDeviceData(),
                                                               splitAddr->GetDeviceData(),
                                                               childSets->GetDeviceData(),
                                                               nodes->GetPrimitiveIndexData(),
                                                               nodes->GetPrimitiveAmountData(),
                                                               nodes->GetChildrenData(),
                                                               splits);
                CHECK_FOR_CUDA_ERROR();

                childrenCreated = splits * 2;

                if (removeFalsePrimitives && childrenCreated > 0){
                    
                    // @TODO propagate downwards or upwards? Test
                    // which is fastest (for non trivial splits
                    // sherlock
                    if (upperLeafIDs){
                        PropagateAabbToChildren<true><<<blocks, threads>>>(upperLeafIDs->GetDeviceData(), 
                                                                           nodes->GetInfoData(), nodes->GetSplitPositionData(),
                                                                           nodes->GetAabbMinData(), nodes->GetAabbMaxData(), 
                                                                           nodes->GetChildrenData());
                    }else
                        PropagateAabbToChildren<false><<<blocks, threads>>>(NULL, nodes->GetInfoData(), nodes->GetSplitPositionData(),
                                                                            nodes->GetAabbMinData(), nodes->GetAabbMaxData(), 
                                                                            nodes->GetChildrenData());
                    CHECK_FOR_CUDA_ERROR();

                    KernelConf conf = KernelConf1D(childrenCreated, 128, 39);
                    TrimChildBitmaps<<<conf.blocks, conf.threads>>>
                        (nodes->GetPrimitiveIndexData() + activeIndex + activeRange,
                         nodes->GetPrimitiveBitmapData() + activeIndex + activeRange,
                         nodes->GetAabbMinData() + activeIndex + activeRange,
                         nodes->GetAabbMaxData() + activeIndex + activeRange,
                         map->GetPrimitiveIndices()->GetDeviceData(),
                         map->GetGeometry()->GetP0Data(), map->GetGeometry()->GetP1Data(), map->GetGeometry()->GetP2Data(),
                         childrenCreated);
                    CHECK_FOR_CUDA_ERROR();
                }
            }

        }
    }
}
