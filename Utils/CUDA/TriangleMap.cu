#include "hip/hip_runtime.h"
// Triangle map class for CUDA
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/TriangleMap.h>
#include <Scene/ISceneNode.h>
#include <Scene/TriangleNode.h>
#include <Utils/CUDA/TriangleMapSAHCreator.h>
#include <Utils/CUDA/Convert.h>

using namespace OpenEngine::Scene;

namespace OpenEngine {
    namespace Utils {
        namespace CUDA {

            TriangleMap::TriangleMap(ISceneNode* scene) 
                : scene(scene), triangles(1), emptySpaceThreshold(0.25f) {

                // Initialized timer
                cutCreateTimer(&timerID);

                geom = new GeometryList(1);
                nodes = new TriangleNode(1);

                scanConfig.algorithm = CUDPP_SCAN;
                scanConfig.op = CUDPP_ADD;
                scanConfig.datatype = CUDPP_INT;
                scanConfig.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
                scanSize = triangles+1;
                scanSize = 262144;
                
                CUDPPResult res = cudppPlan(&scanHandle, scanConfig, scanSize, 1, 0);
                if (CUDPP_SUCCESS != res)
                    throw Core::Exception("Error creating CUDPP scanPlan");

                scanInclConfig.algorithm = CUDPP_SCAN;
                scanInclConfig.op = CUDPP_ADD;
                scanInclConfig.datatype = CUDPP_INT;
                scanInclConfig.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
                scanInclSize = triangles;
                scanInclSize = 262144;

                res = cudppPlan(&scanInclHandle, scanInclConfig, scanInclSize, 1, 0);
                if (CUDPP_SUCCESS != res)
                    throw Core::Exception("Error creating CUDPP inclusive scanPlan");
                
                aabbMin = new CUDADataBlock<1, float4>(1);
                aabbMax = new CUDADataBlock<1, float4>(1);
                tempAabbMin = new CUDADataBlock<1, float4>(1);
                tempAabbMax = new CUDADataBlock<1, float4>(1);
                primMin = new CUDADataBlock<1, float4>(1);
                primMax = new CUDADataBlock<1, float4>(1);
                primIndices = new CUDADataBlock<1, int>(1);

                segments = Segments(1);
                nodeSegments = new CUDADataBlock<1, int>(1);

                splitSide = new CUDADataBlock<1, int>(1);
                splitAddr = new CUDADataBlock<1, int>(1);
                leafSide = new CUDADataBlock<1, int>(1);
                leafAddr = new CUDADataBlock<1, int>(1);
                emptySpaceSplits = new CUDADataBlock<1, int>(1);
                emptySpaceAddrs = new CUDADataBlock<1, int>(1);
                childSize = new CUDADataBlock<1, int2>(1);

                leafIDs = new CUDADataBlock<1, int>(1);

                splitTriangleSet =  new CUDADataBlock<1, int4>(1);

                childAreas = new CUDADataBlock<1, float2>(1);
                childSets = childSize;

                lowerCreator = new TriangleMapSAHCreator();
            }

            void TriangleMap::Create(){

                Setup();
                
                CreateUpperNodes();

                lowerCreator->Create(this, leafIDs);
            }

            void TriangleMap::Setup(){
                geom->CollectGeometry(scene);
                triangles = geom->GetSize();

                logger.info << "Triangles " << triangles << logger.end;
                
                /*
                aabbMin->Extend(triangles);
                aabbMax->Extend(triangles);
                tempAabbMin->Extend(triangles);
                tempAabbMax->Extend(triangles);
                primMin->Extend(triangles);
                primMax->Extend(triangles);

                int approxSize = (2 * triangles / TriangleNode::MAX_LOWER_SIZE) - 1;
                nodes->Extend(approxSize);

                segments.Extend(triangles / Segments::SEGMENT_SIZE);
                */
            }
            
        }
    }
}
