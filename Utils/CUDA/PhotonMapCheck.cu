#include "hip/hip_runtime.h"
// Photon map class for CUDA
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/PhotonMap.h>
#include <Utils/CUDA/Point.h>
#include <Utils/CUDA/Utils.h>

#include <Core/Exceptions.h>

using namespace OpenEngine::Core;

#include <Logging/Logger.h>

namespace OpenEngine {
    namespace Utils {
        namespace CUDA {
            
            void PhotonMap::VerifyMap(){
                char info;
                hipMemcpy(&info, upperNodes.info, sizeof(char), hipMemcpyDeviceToHost);

                float splitPos;
                hipMemcpy(&splitPos, upperNodes.splitPos, sizeof(float), hipMemcpyDeviceToHost);

                point aabbMin, aabbMax;
                hipMemcpy(&aabbMin, upperNodes.aabbMin, sizeof(point), hipMemcpyDeviceToHost);
                hipMemcpy(&aabbMax, upperNodes.aabbMax, sizeof(point), hipMemcpyDeviceToHost);

                point aabbMinAdjusted = aabbMin, aabbMaxAdjusted = aabbMax;
                switch(info){
                case KDNode::X:
                    aabbMinAdjusted.x = aabbMaxAdjusted.x = splitPos;
                    break;
                case KDNode::Y:
                    aabbMinAdjusted.y = aabbMaxAdjusted.y = splitPos;
                    break;
                case KDNode::Z:
                    aabbMinAdjusted.z = aabbMaxAdjusted.z = splitPos;
                    break;
                }

                int left, right;
                hipMemcpy(&left, upperNodes.left, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&right, upperNodes.right, sizeof(int), hipMemcpyDeviceToHost);

                int leftSize = VerifyUpperNode(left, info, splitPos, aabbMin, aabbMaxAdjusted);
                int rightSize = VerifyUpperNode(right, info, splitPos, aabbMinAdjusted, aabbMax);

                int2 photonInfo;
                hipMemcpy(&photonInfo, upperNodes.photonInfo, sizeof(int2), hipMemcpyDeviceToHost);

                if (leftSize + rightSize != photonInfo.y)
                    throw Exception("Root nodes size " + 
                                    Utils::Convert::ToString(photonInfo.y) + 
                                    " isn't the sum of left size " + 
                                    Utils::Convert::ToString(leftSize) +
                                    " and right size " + Utils::Convert::ToString(rightSize));
            }

            int PhotonMap::VerifyUpperNode(int index, char parentInfo, float parentSplitPos,
                                           point parentAABBMin, point parentAABBMax){
                char info;
                hipMemcpy(&info, upperNodes.info+index, sizeof(char), hipMemcpyDeviceToHost);
                
                int2 photonInfo;
                hipMemcpy(&photonInfo, upperNodes.photonInfo+index, sizeof(int2), hipMemcpyDeviceToHost);
                    
                if (info == KDNode::LEAF){
                    // Base case
                    
                    point positions[photonInfo.y];
                    hipMemcpy(positions, photons.pos + photonInfo.x, 
                               photonInfo.y * sizeof(point), hipMemcpyDeviceToHost);
                    point aabbMin, aabbMax;
                    aabbMin = aabbMax = positions[0];
                    for (int i = 1; i < photonInfo.y; ++i){
                        aabbMin = pointMin(aabbMin, positions[i]);
                        aabbMax = pointMax(aabbMax, positions[i]);
                    }

                    if (!aabbContains(parentAABBMin, parentAABBMax, aabbMin))
                        throw Exception("Leaf node " + Utils::Convert::ToString(index) +
                                        " aabb minimum cornor " + Utils::CUDA::Convert::ToString(aabbMin) +
                                        " is not contained in parents aabb " + Utils::CUDA::Convert::ToString(parentAABBMin) + 
                                        " -> " + Utils::CUDA::Convert::ToString(parentAABBMax) + ".");

                    if (!aabbContains(parentAABBMin, parentAABBMax, aabbMax))
                        throw Exception("Leaf node " + Utils::Convert::ToString(index) +
                                        " aabb maximum cornor " + Utils::CUDA::Convert::ToString(aabbMax) +
                                        " is not contained in parents aabb " + Utils::CUDA::Convert::ToString(parentAABBMin) + 
                                        " -> " + Utils::CUDA::Convert::ToString(parentAABBMax) + ".");
                    
                }else{
                    // Check parent info
                    float splitPos;
                    hipMemcpy(&splitPos, upperNodes.splitPos+index, sizeof(float), hipMemcpyDeviceToHost);
                    
                    point aabbMin, aabbMax;
                    hipMemcpy(&aabbMin, upperNodes.aabbMin+index, sizeof(point), hipMemcpyDeviceToHost);
                    hipMemcpy(&aabbMax, upperNodes.aabbMax+index, sizeof(point), hipMemcpyDeviceToHost);

                    if (!aabbContains(parentAABBMin, parentAABBMax, aabbMin))
                        throw Exception("Node " + Utils::Convert::ToString(index) +
                                        " aabb minimum cornor " + Utils::CUDA::Convert::ToString(aabbMin) +
                                        " is not contained in parents aabb " + Utils::CUDA::Convert::ToString(parentAABBMin) + 
                                        " -> " + Utils::CUDA::Convert::ToString(parentAABBMax) + ".");

                    if (!aabbContains(parentAABBMin, parentAABBMax, aabbMax))
                        throw Exception("Node " + Utils::Convert::ToString(index) +
                                        " aabb maximum cornor " + Utils::CUDA::Convert::ToString(aabbMax) +
                                        " is not contained in parents aabb " + Utils::CUDA::Convert::ToString(parentAABBMin) + 
                                        " -> " + Utils::CUDA::Convert::ToString(parentAABBMax) + ".");


                    point aabbMinAdjusted = aabbMin, aabbMaxAdjusted = aabbMax;
                    switch(info){
                    case KDNode::X:
                        aabbMinAdjusted.x = aabbMaxAdjusted.x = splitPos;
                        break;
                    case KDNode::Y:
                        aabbMinAdjusted.y = aabbMaxAdjusted.y = splitPos;
                        break;
                    case KDNode::Z:
                        aabbMinAdjusted.z = aabbMaxAdjusted.z = splitPos;
                        break;
                    }
                    
                    int left, right;
                    hipMemcpy(&left, upperNodes.left+index, sizeof(int), hipMemcpyDeviceToHost);
                    hipMemcpy(&right, upperNodes.right+index, sizeof(int), hipMemcpyDeviceToHost);

                    int leftSize = VerifyUpperNode(left, info, splitPos, aabbMin, aabbMaxAdjusted);
                    int rightSize = VerifyUpperNode(right, info, splitPos, aabbMinAdjusted, aabbMax);

                    if (leftSize + rightSize != photonInfo.y)
                        throw Exception("The " + Utils::Convert::ToString(index) + 
                                        "'th node's size " + 
                                        Utils::Convert::ToString(photonInfo.y) + 
                                        " isn't the sum of left size " + 
                                        Utils::Convert::ToString(leftSize) +
                                        " and right size " + Utils::Convert::ToString(rightSize));
                }

                return photonInfo.y;
            }

        }
    }
}
