#include "hip/hip_runtime.h"
// Geometry List.
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/GeometryList.h>

#include <Geometry/Mesh.h>
#include <Geometry/GeometrySet.h>
#include <Math/CUDA/Matrix.h>
#include <Scene/ISceneNode.h>
#include <Scene/MeshNode.h>
#include <Scene/CUDAMeshNode.h>
#include <Scene/RenderStateNode.h>
#include <Scene/TransformationNode.h>
#include <Utils/CUDA/Utils.h>
#include <Utils/CUDA/Convert.h>

#include <sstream>

using namespace OpenEngine::Geometry;
using namespace OpenEngine::Math;
using namespace OpenEngine::Math::CUDA;
using namespace OpenEngine::Scene;
using namespace OpenEngine::Resources::CUDA;

namespace OpenEngine {
    namespace Utils {
        namespace CUDA {

            GeometryList::GeometryList()
                : maxSize(0), size(0) {}

            GeometryList::GeometryList(int size)
                : maxSize(size), size(0) {

                cutCreateTimer(&timerID);

                p0 = new CUDADataBlock<1, float4>(maxSize);
                p1 = new CUDADataBlock<1, float4>(maxSize);
                p2 = new CUDADataBlock<1, float4>(maxSize);

                n0 = new CUDADataBlock<1, float4>(maxSize);
                n1 = new CUDADataBlock<1, float4>(maxSize);
                n2 = new CUDADataBlock<1, float4>(maxSize);

                c0 = new CUDADataBlock<1, uchar4>(maxSize);
                c1 = new CUDADataBlock<1, uchar4>(maxSize);
                c2 = new CUDADataBlock<1, uchar4>(maxSize);
            }

            void GeometryList::Resize(int i){
                p0->Resize(i); p1->Resize(i); p2->Resize(i);
                n0->Resize(i); n1->Resize(i); n2->Resize(i);
                c0->Resize(i); c1->Resize(i); c2->Resize(i);

                maxSize = i;
                size = min(size, i);
            }

            void GeometryList::Extend(int i){
                if (maxSize < i)
                    Resize(i);
            }

            std::string GeometryList::ToString(unsigned int i) const {
                std::ostringstream out;

                out <<  "Triangle #" << i << "\n";

                float4 h_p0, h_p1, h_p2;
                hipMemcpy(&h_p0, p0->GetDeviceData() + i, sizeof(float4), hipMemcpyDeviceToHost);
                hipMemcpy(&h_p1, p1->GetDeviceData() + i, sizeof(float4), hipMemcpyDeviceToHost);
                hipMemcpy(&h_p2, p2->GetDeviceData() + i, sizeof(float4), hipMemcpyDeviceToHost);
                out << "Points: " << Utils::CUDA::Convert::ToString(h_p0) << ", " << Utils::CUDA::Convert::ToString(h_p1) << " & " << Utils::CUDA::Convert::ToString(h_p2) << "\n";

                float4 h_n0, h_n1, h_n2;
                hipMemcpy(&h_n0, n0->GetDeviceData() + i, sizeof(float4), hipMemcpyDeviceToHost);
                hipMemcpy(&h_n1, n1->GetDeviceData() + i, sizeof(float4), hipMemcpyDeviceToHost);
                hipMemcpy(&h_n2, n2->GetDeviceData() + i, sizeof(float4), hipMemcpyDeviceToHost);
                out << "Normals: " << Utils::CUDA::Convert::ToString(h_n0) << ", " << Utils::CUDA::Convert::ToString(h_n1) << " & " << Utils::CUDA::Convert::ToString(h_n2) << "\n";

                uchar4 h_c0, h_c1, h_c2;
                hipMemcpy(&h_c0, c0->GetDeviceData() + i, sizeof(uchar4), hipMemcpyDeviceToHost);
                hipMemcpy(&h_c1, c1->GetDeviceData() + i, sizeof(uchar4), hipMemcpyDeviceToHost);
                hipMemcpy(&h_c2, c2->GetDeviceData() + i, sizeof(uchar4), hipMemcpyDeviceToHost);
                out << "Colors: " << Utils::CUDA::Convert::ToString(h_c0) << ", " << Utils::CUDA::Convert::ToString(h_c1) << " & " << Utils::CUDA::Convert::ToString(h_c2) << "\n";
                
                return out.str();
            }

            __global__ void AddMeshKernel(unsigned int *indices,
                                          float3 *verticesIn,
                                          float3 *normalsIn,
                                          float4 *colorsIn,
                                          const Matrix44f modelMat, const Matrix33f normalMat,
                                          float4 *p0, float4 *p1, float4 *p2,
                                          float4 *n0, float4 *n1, float4 *n2,
                                          uchar4 *c0, uchar4 *c1, uchar4 *c2,
                                          int size){
                
                const int id = blockDim.x * blockIdx.x + threadIdx.x;

                if (id < size){
                    const int i = __mul24(id, 3);
                    const unsigned int i0 = indices[i];
                    const unsigned int i1 = indices[i+1];
                    const unsigned int i2 = indices[i+2];
                    const float3 v0 = verticesIn[i0];
                    const float3 v1 = verticesIn[i1];
                    const float3 v2 = verticesIn[i2];
                    
                    p0[id] = modelMat * make_float4(v0, 1.0f);
                    p1[id] = modelMat * make_float4(v1, 1.0f);
                    p2[id] = modelMat * make_float4(v2, 1.0f);
                    
                    n0[id] = make_float4(normalMat * normalsIn[i0], 0);
                    n1[id] = make_float4(normalMat * normalsIn[i1], 0);
                    n2[id] = make_float4(normalMat * normalsIn[i2], 0);

                    c0[id] = make_uchar4(colorsIn[i0].x * 255.0f, colorsIn[i0].y * 255.0f, colorsIn[i0].z * 255.0f, colorsIn[i0].w * 255.0f);
                    c1[id] = make_uchar4(colorsIn[i1].x * 255.0f, colorsIn[i1].y * 255.0f, colorsIn[i1].z * 255.0f, colorsIn[i1].w * 255.0f);
                    c2[id] = make_uchar4(colorsIn[i2].x * 255.0f, colorsIn[i2].y * 255.0f, colorsIn[i2].z * 255.0f, colorsIn[i2].w * 255.0f);
                }
            }

            __global__ void AddMeshKernel(unsigned int *indices,
                                          float4 *verticesIn,
                                          float4 *normalsIn,
                                          uchar4 *colorsIn,
                                          const Matrix44f modelMat, const Matrix33f normalMat,
                                          float4 *p0, float4 *p1, float4 *p2,
                                          float4 *n0, float4 *n1, float4 *n2,
                                          uchar4 *c0, uchar4 *c1, uchar4 *c2,
                                          int size){
                
                const int id = blockDim.x * blockIdx.x + threadIdx.x;

                if (id < size){
                    const int i = __mul24(id, 3);
                    const unsigned int i0 = indices[i];
                    const unsigned int i1 = indices[i+1];
                    const unsigned int i2 = indices[i+2];
                    const float4 v0 = verticesIn[i0];
                    const float4 v1 = verticesIn[i1];
                    const float4 v2 = verticesIn[i2];
                    
                    p0[id] = modelMat * v0;
                    p1[id] = modelMat * v1;
                    p2[id] = modelMat * v2;
                    
                    n0[id] = make_float4(normalMat * make_float3(normalsIn[i0]), 0);
                    n1[id] = make_float4(normalMat * make_float3(normalsIn[i1]), 0);
                    n2[id] = make_float4(normalMat * make_float3(normalsIn[i2]), 0);

                    c0[id] = colorsIn[i0];
                    c1[id] = colorsIn[i1];
                    c2[id] = colorsIn[i2];
                }
            }

            void GeometryList::AddMesh(MeshPtr mesh, Matrix<4,4,float> modelMat){

                GeometrySetPtr geom = mesh->GetGeometrySet();
                if (geom->GetDataBlock("vertex") && geom->GetDataBlock("vertex")->GetID() != 0){
                    // Geometry has been loaded to the graphics card
                    // and we can copy it from there.
                    IndicesPtr indices = mesh->GetIndices();
                    IDataBlockPtr vertices = geom->GetDataBlock("vertex");
                    IDataBlockPtr normals = geom->GetDataBlock("normal");
                    IDataBlockPtr colors = geom->GetDataBlock("color");

                    START_TIMER(timerID);
                    unsigned int triangles = indices->GetSize() / 3;
                    Extend(size + triangles);
                    
                    hipGraphicsResource *iResource, *vResource, *nResource, *cResource;
                    hipGraphicsGLRegisterBuffer(&iResource, indices->GetID(), cudaGraphicsMapFlagsReadOnly);
                    hipGraphicsMapResources(1, &iResource, 0);
                    CHECK_FOR_CUDA_ERROR();
                    hipGraphicsGLRegisterBuffer(&vResource, vertices->GetID(), cudaGraphicsMapFlagsReadOnly);
                    hipGraphicsMapResources(1, &vResource, 0);
                    CHECK_FOR_CUDA_ERROR();
                    hipGraphicsGLRegisterBuffer(&nResource, normals->GetID(), cudaGraphicsMapFlagsReadOnly);
                    hipGraphicsMapResources(1, &nResource, 0);
                    CHECK_FOR_CUDA_ERROR();
                    hipGraphicsGLRegisterBuffer(&cResource, colors->GetID(), cudaGraphicsMapFlagsReadOnly);
                    hipGraphicsMapResources(1, &cResource, 0);
                    CHECK_FOR_CUDA_ERROR();
                    
                    size_t bytes;
                    unsigned int* in;
                    hipGraphicsResourceGetMappedPointer((void**)&in, &bytes,
                                                         iResource);
                    CHECK_FOR_CUDA_ERROR();
                    float3* pos;
                    hipGraphicsResourceGetMappedPointer((void**)&pos, &bytes,
                                                         vResource);
                    CHECK_FOR_CUDA_ERROR();
                    float3* norms;
                    hipGraphicsResourceGetMappedPointer((void**)&norms, &bytes,
                                                         nResource);
                    CHECK_FOR_CUDA_ERROR();
                    float4* cols;
                    hipGraphicsResourceGetMappedPointer((void**)&cols, &bytes,
                                                         cResource);
                    CHECK_FOR_CUDA_ERROR();

                    unsigned int blocks, threads;
                    Calc1DKernelDimensions(indices->GetSize(), blocks, threads);
                    Math::CUDA::Matrix44f mat;
                    mat.Init(modelMat.GetTranspose());
                    Math::CUDA::Matrix33f normMat; // should be transposed and inverted, jada jada bla bla just don't do weird scaling
                    normMat.Init(mat);
                    CHECK_FOR_CUDA_ERROR();

                    AddMeshKernel<<<blocks, threads>>>(in, pos, norms, cols,
                                                       mat, normMat,
                                                       p0->GetDeviceData() + size, p1->GetDeviceData() + size, p2->GetDeviceData() + size,
                                                       n0->GetDeviceData() + size, n1->GetDeviceData() + size, n2->GetDeviceData() + size,
                                                       c0->GetDeviceData() + size, c1->GetDeviceData() + size, c2->GetDeviceData() + size,
                                                       triangles);
                    CHECK_FOR_CUDA_ERROR();

                    size += triangles;

                    hipGraphicsUnmapResources(1, &iResource, 0);
                    hipGraphicsUnmapResources(1, &vResource, 0);
                    hipGraphicsUnmapResources(1, &nResource, 0);
                    hipGraphicsUnmapResources(1, &cResource, 0);
                    CHECK_FOR_CUDA_ERROR();

                    hipGraphicsUnregisterResource(iResource);
                    hipGraphicsUnregisterResource(vResource);
                    hipGraphicsUnregisterResource(nResource);
                    hipGraphicsUnregisterResource(cResource);
                    CHECK_FOR_CUDA_ERROR();

                    PRINT_TIMER(timerID, "Geometry collection ");
                }else{
                    // Geometry is still on the CPU
                    throw Exception("Not implemented");
                }
            }

            void GeometryList::AddMesh(CUDAMeshNode* mesh, 
                                       Matrix<4, 4, float> modelMat){
                
                //START_TIMER(timerID);

                unsigned int triangles = mesh->GetSize() / 3;
                Extend(size + triangles);                

                unsigned int blocks, threads;
                Calc1DKernelDimensions(mesh->GetSize(), blocks, threads);
                Math::CUDA::Matrix44f mat;
                mat.Init(modelMat.GetTranspose());
                Math::CUDA::Matrix33f normMat; // should be transposed and inverted, jada jada bla bla just don't do weird scaling
                normMat.Init(mat);
                CHECK_FOR_CUDA_ERROR();
                
                AddMeshKernel<<<blocks, threads>>>(mesh->GetIndexData(), mesh->GetVertexData(), mesh->GetNormalData(), mesh->GetColorData(),
                                                   mat, normMat,
                                                   p0->GetDeviceData() + size, p1->GetDeviceData() + size, p2->GetDeviceData() + size,
                                                   n0->GetDeviceData() + size, n1->GetDeviceData() + size, n2->GetDeviceData() + size,
                                                   c0->GetDeviceData() + size, c1->GetDeviceData() + size, c2->GetDeviceData() + size,
                                                   triangles);
                CHECK_FOR_CUDA_ERROR();

                

                size += triangles;
                
                //PRINT_TIMER(timerID, "Geometry collection ");
            }
            
            void GeometryList::CollectGeometry(ISceneNode* node){
                currentModelMat = Matrix<4,4, float>();
                size = 0;
                node->Accept(*this);
            }

            void GeometryList::VisitRenderStateNode(RenderStateNode* node){
                node->VisitSubNodes(*this);
            }

            void GeometryList::VisitTransformationNode(TransformationNode* node){
                // push transformation matrix
                Matrix<4,4,float> m = node->GetTransformationMatrix();
                Matrix<4, 4, float> oldModelMat = currentModelMat;
                currentModelMat = m * currentModelMat;

                // traverse sub nodes
                node->VisitSubNodes(*this);

                // pop transformation matrix
                currentModelMat = oldModelMat;
            }
            
            void GeometryList::VisitMeshNode(MeshNode* node){
                if (node->GetMesh()->GetGeometrySet()->GetVertices()->GetID() != 0){
                    AddMesh(node->GetMesh(), currentModelMat);
                    
                    node->VisitSubNodes(*this);
                }else{
                    CUDAMeshNode* mesh = new CUDAMeshNode(node);

                    node->GetParent()->ReplaceNode(node, mesh);

                    std::list<ISceneNode*> subNodes = node->subNodes;
                    for (std::list<ISceneNode*>::iterator itr = subNodes.begin();
                         itr != subNodes.end(); ++itr){
                        node->RemoveNode(*itr);
                        mesh->AddNode(*itr);
                    }

                    mesh->Accept(*this);
                }
            }

            void GeometryList::VisitCUDAMeshNode(CUDAMeshNode* node){
                AddMesh(node, currentModelMat);

                node->VisitSubNodes(*this);
            }

        }
    }
}
