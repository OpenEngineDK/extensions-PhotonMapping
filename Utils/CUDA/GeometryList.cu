#include "hip/hip_runtime.h"
// Geometry List.
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/GeometryList.h>

#include <Geometry/Mesh.h>
#include <Geometry/GeometrySet.h>
#include <Math/CUDA/Matrix.h>
#include <Scene/ISceneNode.h>
#include <Scene/MeshNode.h>
#include <Scene/CUDAMeshNode.h>
#include <Scene/RenderStateNode.h>
#include <Scene/TransformationNode.h>
#include <Utils/CUDA/Utils.h>
#include <Utils/CUDA/IntersectionTests.h>
#include <Utils/CUDA/Convert.h>

#include <Utils/CUDA/LoggerExtensions.h>

#include <sstream>

using namespace OpenEngine::Geometry;
using namespace OpenEngine::Math;
using namespace OpenEngine::Math::CUDA;
using namespace OpenEngine::Scene;
using namespace OpenEngine::Resources::CUDA;

#define MAX_THREADS 128

namespace OpenEngine {
    namespace Utils {
        namespace CUDA {

            GeometryList::GeometryList()
                : maxSize(0), size(0) {}

            GeometryList::GeometryList(int size)
                : maxSize(size), size(0) {

                cutCreateTimer(&timerID);

                p0 = new CUDADataBlock<1, float4>(maxSize);
                p1 = new CUDADataBlock<1, float4>(maxSize);
                p2 = new CUDADataBlock<1, float4>(maxSize);

                n0 = new CUDADataBlock<1, float4>(maxSize);
                n1 = new CUDADataBlock<1, float4>(maxSize);
                n2 = new CUDADataBlock<1, float4>(maxSize);

                c0 = new CUDADataBlock<1, uchar4>(maxSize);
                c1 = new CUDADataBlock<1, uchar4>(maxSize);
                c2 = new CUDADataBlock<1, uchar4>(maxSize);

                woop0 = new CUDADataBlock<1, float4>(maxSize);
                woop1 = new CUDADataBlock<1, float4>(maxSize);
                woop2 = new CUDADataBlock<1, float4>(maxSize);
            }

            void GeometryList::Resize(int i){
                p0->Resize(i); p1->Resize(i); p2->Resize(i);
                n0->Resize(i); n1->Resize(i); n2->Resize(i);
                c0->Resize(i); c1->Resize(i); c2->Resize(i);
                woop0->Resize(i); woop1->Resize(i); woop2->Resize(i);

                maxSize = i;
                size = min(size, i);
            }

            void GeometryList::Extend(int i){
                if (maxSize < i)
                    Resize(i);
            }

            __global__ void 
            __launch_bounds__(MAX_THREADS) 
                CreateWoopValues(float4* p0s, float4* p1s, float4* p2s, 
                                 float4* m0s, float4* m1s, float4* m2s,
                                 int primitives){
                
                const int id = blockDim.x * blockIdx.x + threadIdx.x;
                
                if (id < primitives){

                    const float3 p0 = make_float3(p0s[id]);
                    const float3 p1 = make_float3(p1s[id]);
                    const float3 p2 = make_float3(p2s[id]);

                    float4 m0, m1, m2;
                    WoopTransformationMatrix(p1, p2, p0, m0, m1, m2);

                    m0s[id] = m0;
                    m1s[id] = m1;
                    m2s[id] = m2;
                }
            }

            void GeometryList::GetWoopValues(float4** m0, float4** m1, float4** m2){
                int i = p0->GetSize();
                woop0->Resize(i); woop1->Resize(i); woop2->Resize(i);
                
                KernelConf conf = KernelConf1D(i, MAX_THREADS);
                CreateWoopValues<<<conf.blocks, conf.threads>>>
                    (p0->GetDeviceData(), p1->GetDeviceData(), p2->GetDeviceData(),
                     woop0->GetDeviceData(), woop1->GetDeviceData(), woop2->GetDeviceData(),
                     i);
                CHECK_FOR_CUDA_ERROR();

                *m0 = woop0->GetDeviceData();
                *m1 = woop1->GetDeviceData();
                *m2 = woop2->GetDeviceData();
            }

            std::string GeometryList::ToString(unsigned int i) const {
                std::ostringstream out;

                out <<  "Triangle #" << i << "\n";

                out << "Points: " << FetchGlobalData(p0->GetDeviceData(), i) << ", " 
                    << FetchGlobalData(p1->GetDeviceData(), i) << " & " 
                    << FetchGlobalData(p2->GetDeviceData(), i) << "\n";

                out << "Normals: " << FetchGlobalData(n0->GetDeviceData(), i) << ", " 
                    << FetchGlobalData(n1->GetDeviceData(), i) << " & " 
                    << FetchGlobalData(n2->GetDeviceData(), i) << "\n";

                out << "Colors: " << FetchGlobalData(c0->GetDeviceData(), i) << ", " 
                    << FetchGlobalData(c1->GetDeviceData(), i) << " & " 
                    << FetchGlobalData(c2->GetDeviceData(), i) << "\n";
                
                return out.str();
            }

            __global__ void AddMeshKernel(unsigned int *indices,
                                          float3 *verticesIn,
                                          float3 *normalsIn,
                                          float4 *colorsIn,
                                          const Matrix44f modelMat, const Matrix33f normalMat,
                                          float4 *p0, float4 *p1, float4 *p2,
                                          float4 *n0, float4 *n1, float4 *n2,
                                          uchar4 *c0, uchar4 *c1, uchar4 *c2,
                                          int size){
                
                const int id = blockDim.x * blockIdx.x + threadIdx.x;

                if (id < size){
                    const int i = __mul24(id, 3);
                    const unsigned int i0 = indices[i];
                    const unsigned int i1 = indices[i+1];
                    const unsigned int i2 = indices[i+2];
                    const float3 v0 = verticesIn[i0];
                    const float3 v1 = verticesIn[i1];
                    const float3 v2 = verticesIn[i2];
                    
                    p0[id] = modelMat * make_float4(v0, 1.0f);
                    p1[id] = modelMat * make_float4(v1, 1.0f);
                    p2[id] = modelMat * make_float4(v2, 1.0f);
                    
                    n0[id] = make_float4(normalMat * normalsIn[i0], 0);
                    n1[id] = make_float4(normalMat * normalsIn[i1], 0);
                    n2[id] = make_float4(normalMat * normalsIn[i2], 0);

                    c0[id] = make_uchar4(colorsIn[i0].x * 255.0f, colorsIn[i0].y * 255.0f, colorsIn[i0].z * 255.0f, colorsIn[i0].w * 255.0f);
                    c1[id] = make_uchar4(colorsIn[i1].x * 255.0f, colorsIn[i1].y * 255.0f, colorsIn[i1].z * 255.0f, colorsIn[i1].w * 255.0f);
                    c2[id] = make_uchar4(colorsIn[i2].x * 255.0f, colorsIn[i2].y * 255.0f, colorsIn[i2].z * 255.0f, colorsIn[i2].w * 255.0f);
                }
            }

            __global__ void 
            __launch_bounds__(MAX_THREADS) 
                AddMeshKernel(unsigned int *indices,
                              float4 *verticesIn,
                              float4 *normalsIn,
                              uchar4 *colorsIn,
                              const Matrix44f modelMat, const Matrix33f normalMat,
                              float4 *p0, float4 *p1, float4 *p2,
                              float4 *n0, float4 *n1, float4 *n2,
                              uchar4 *c0, uchar4 *c1, uchar4 *c2,
                              int size){
                
                const int id = blockDim.x * blockIdx.x + threadIdx.x;

                if (id < size){
                    const int i = __mul24(id, 3);
                    const unsigned int i0 = indices[i];
                    const unsigned int i1 = indices[i+1];
                    const unsigned int i2 = indices[i+2];
                    const float4 v0 = verticesIn[i0];
                    const float4 v1 = verticesIn[i1];
                    const float4 v2 = verticesIn[i2];
                    
                    p0[id] = modelMat * v0;
                    p1[id] = modelMat * v1;
                    p2[id] = modelMat * v2;
                    
                    n0[id] = make_float4(normalMat * make_float3(normalsIn[i0]), 0);
                    n1[id] = make_float4(normalMat * make_float3(normalsIn[i1]), 0);
                    n2[id] = make_float4(normalMat * make_float3(normalsIn[i2]), 0);

                    c0[id] = colorsIn[i0];
                    c1[id] = colorsIn[i1];
                    c2[id] = colorsIn[i2];
                }
            }

            void GeometryList::AddMesh(MeshPtr mesh, Matrix<4,4,float> modelMat){

                GeometrySetPtr geom = mesh->GetGeometrySet();
                if (geom->GetDataBlock("vertex") && geom->GetDataBlock("vertex")->GetID() != 0){
                    // Geometry has been loaded to the graphics card
                    // and we can copy it from there.
                    IndicesPtr indices = mesh->GetIndices();
                    IDataBlockPtr vertices = geom->GetDataBlock("vertex");
                    IDataBlockPtr normals = geom->GetDataBlock("normal");
                    IDataBlockPtr colors = geom->GetDataBlock("color");

                    START_TIMER(timerID);
                    unsigned int triangles = indices->GetSize() / 3;
                    Extend(size + triangles);
                    
                    hipGraphicsResource *iResource, *vResource, *nResource, *cResource;
                    hipGraphicsGLRegisterBuffer(&iResource, indices->GetID(), cudaGraphicsMapFlagsReadOnly);
                    hipGraphicsMapResources(1, &iResource, 0);
                    CHECK_FOR_CUDA_ERROR();
                    hipGraphicsGLRegisterBuffer(&vResource, vertices->GetID(), cudaGraphicsMapFlagsReadOnly);
                    hipGraphicsMapResources(1, &vResource, 0);
                    CHECK_FOR_CUDA_ERROR();
                    hipGraphicsGLRegisterBuffer(&nResource, normals->GetID(), cudaGraphicsMapFlagsReadOnly);
                    hipGraphicsMapResources(1, &nResource, 0);
                    CHECK_FOR_CUDA_ERROR();
                    hipGraphicsGLRegisterBuffer(&cResource, colors->GetID(), cudaGraphicsMapFlagsReadOnly);
                    hipGraphicsMapResources(1, &cResource, 0);
                    CHECK_FOR_CUDA_ERROR();
                    
                    size_t bytes;
                    unsigned int* in;
                    hipGraphicsResourceGetMappedPointer((void**)&in, &bytes,
                                                         iResource);
                    CHECK_FOR_CUDA_ERROR();
                    float3* pos;
                    hipGraphicsResourceGetMappedPointer((void**)&pos, &bytes,
                                                         vResource);
                    CHECK_FOR_CUDA_ERROR();
                    float3* norms;
                    hipGraphicsResourceGetMappedPointer((void**)&norms, &bytes,
                                                         nResource);
                    CHECK_FOR_CUDA_ERROR();
                    float4* cols;
                    hipGraphicsResourceGetMappedPointer((void**)&cols, &bytes,
                                                         cResource);
                    CHECK_FOR_CUDA_ERROR();

                    unsigned int blocks, threads;
                    Calc1DKernelDimensions(indices->GetSize(), blocks, threads);
                    Math::CUDA::Matrix44f mat;
                    mat.Init(modelMat.GetTranspose());
                    Math::CUDA::Matrix33f normMat; // should be transposed and inverted, jada jada bla bla just don't do weird scaling
                    normMat.Init(mat);
                    CHECK_FOR_CUDA_ERROR();

                    AddMeshKernel<<<blocks, threads>>>(in, pos, norms, cols,
                                                       mat, normMat,
                                                       p0->GetDeviceData() + size, p1->GetDeviceData() + size, p2->GetDeviceData() + size,
                                                       n0->GetDeviceData() + size, n1->GetDeviceData() + size, n2->GetDeviceData() + size,
                                                       c0->GetDeviceData() + size, c1->GetDeviceData() + size, c2->GetDeviceData() + size,
                                                       triangles);
                    CHECK_FOR_CUDA_ERROR();

                    size += triangles;

                    hipGraphicsUnmapResources(1, &iResource, 0);
                    hipGraphicsUnmapResources(1, &vResource, 0);
                    hipGraphicsUnmapResources(1, &nResource, 0);
                    hipGraphicsUnmapResources(1, &cResource, 0);
                    CHECK_FOR_CUDA_ERROR();

                    hipGraphicsUnregisterResource(iResource);
                    hipGraphicsUnregisterResource(vResource);
                    hipGraphicsUnregisterResource(nResource);
                    hipGraphicsUnregisterResource(cResource);
                    CHECK_FOR_CUDA_ERROR();

                    PRINT_TIMER(timerID, "Geometry collection ");
                }else{
                    // Geometry is still on the CPU
                    throw Exception("Not implemented");
                }
            }

            void GeometryList::AddMesh(CUDAMeshNode* mesh, 
                                       Matrix<4, 4, float> modelMat){
                
                unsigned int triangles = mesh->GetSize() / 3;
                Extend(size + triangles);                

                Math::CUDA::Matrix44f mat;
                mat.Init(modelMat.GetTranspose());
                Math::CUDA::Matrix33f normMat; // should be transposed and inverted, jada jada bla bla just don't do weird scaling
                normMat.Init(mat);
                CHECK_FOR_CUDA_ERROR();
                
                unsigned int blocks, threads;
                Calc1DKernelDimensions(mesh->GetSize(), blocks, threads, MAX_THREADS);
                AddMeshKernel<<<blocks, threads>>>(mesh->GetIndexData(), mesh->GetVertexData(), mesh->GetNormalData(), mesh->GetColorData(),
                                                   mat, normMat,
                                                   p0->GetDeviceData() + size, p1->GetDeviceData() + size, p2->GetDeviceData() + size,
                                                   n0->GetDeviceData() + size, n1->GetDeviceData() + size, n2->GetDeviceData() + size,
                                                   c0->GetDeviceData() + size, c1->GetDeviceData() + size, c2->GetDeviceData() + size,
                                                   triangles);
                CHECK_FOR_CUDA_ERROR();

                size += triangles;
            }
            
            void GeometryList::CollectGeometry(ISceneNode* node){
                currentModelMat = Matrix<4,4, float>();
                size = 0;
                node->Accept(*this);
            }

            void GeometryList::VisitRenderStateNode(RenderStateNode* node){
                node->VisitSubNodes(*this);
            }

            void GeometryList::VisitTransformationNode(TransformationNode* node){
                // push transformation matrix
                Matrix<4,4,float> m = node->GetTransformationMatrix();
                Matrix<4, 4, float> oldModelMat = currentModelMat;
                currentModelMat = m * currentModelMat;

                // traverse sub nodes
                node->VisitSubNodes(*this);

                // pop transformation matrix
                currentModelMat = oldModelMat;
            }
            
            void GeometryList::VisitMeshNode(MeshNode* node){
                if (node->GetMesh()->GetGeometrySet()->GetVertices()->GetID() != 0){
                    AddMesh(node->GetMesh(), currentModelMat);
                    
                    node->VisitSubNodes(*this);
                }else{
                    CUDAMeshNode* mesh = new CUDAMeshNode(node);

                    node->GetParent()->ReplaceNode(node, mesh);

                    std::list<ISceneNode*> subNodes = node->subNodes;
                    for (std::list<ISceneNode*>::iterator itr = subNodes.begin();
                         itr != subNodes.end(); ++itr){
                        node->RemoveNode(*itr);
                        mesh->AddNode(*itr);
                    }

                    mesh->Accept(*this);
                }
            }

            void GeometryList::VisitCUDAMeshNode(CUDAMeshNode* node){
                AddMesh(node, currentModelMat);

                node->VisitSubNodes(*this);
            }

        }
    }
}
