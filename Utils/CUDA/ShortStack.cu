#include "hip/hip_runtime.h"
// Short stack raytracer for CUDA
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/ShortStack.h>

#include <Display/IViewingVolume.h>
#include <Display/IRenderCanvas.h>
#include <Scene/TriangleNode.h>
#include <Utils/CUDA/Convert.h>
#include <Utils/CUDA/SharedMemory.h>
#include <Utils/CUDA/TriangleMap.h>
#include <Utils/CUDA/Utils.h>

namespace OpenEngine {
    using namespace Display;
    using namespace Resources;
    using namespace Resources::CUDA;
    using namespace Scene;
    namespace Utils {
        namespace CUDA {

#include <Utils/CUDA/Kernels/ColorKernels.h>

            ShortStack::ShortStack(TriangleMap* map)
                : map(map) {
                
                cutCreateTimer(&timerID);

                float3 lightPosition = make_float3(0.0f, 4.0f, 0.0f);
                hipMemcpyToSymbol(HIP_SYMBOL(d_lightPosition), &lightPosition, sizeof(float3));
                float3 lightColor = make_float3(1.0f, 0.92f, 0.8f);
                float3 ambient = lightColor * 0.3f;
                hipMemcpyToSymbol(HIP_SYMBOL(d_lightAmbient), &ambient, sizeof(float3));
                float3 diffuse = lightColor * 0.7f;
                hipMemcpyToSymbol(HIP_SYMBOL(d_lightDiffuse), &diffuse, sizeof(float3));
                float3 specular = lightColor * 0.3f;
                hipMemcpyToSymbol(HIP_SYMBOL(d_lightSpecular), &specular, sizeof(float3));
                CHECK_FOR_CUDA_ERROR();
            }

            ShortStack::~ShortStack() {}

            __constant__ int d_rays;

            __global__ void ShortStackTrace(float4* origins, float4* directions,
                                            char* nodeInfo, float* splitPos,
                                            int* leftChild, int* rightChild,
                                            int2 *primitiveInfo, 
                                            int *primIndices, 
                                            float4 *v0, float4 *v1, float4 *v2,
                                            float4 *n0s, float4 *n1s, float4 *n2s,
                                            uchar4 *c0s,
                                            uchar4 *canvas){
                
                const int id = blockDim.x * blockIdx.x + threadIdx.x;
                
                if (id < d_rays){
                    
                    ShortStack::Stack<6> stack;

                    float3 origin = make_float3(origins[id]);
                    float3 direction = make_float3(directions[id]);

                    float3 tHit;
                    tHit.x = 0.0f;

                    float4 color = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

                    do {
                        int node; float tNext;
                        if (stack.IsEmpty()){
                            node = 0;
                            tNext = fInfinity;
                        }else{
                            ShortStack::Element e = stack.Pop();
                            node = e.node;
                            tNext = e.tMax;
                        }
                        
                        char info = nodeInfo[node];

                        while ((info & 3) != KDNode::LEAF){
                            float splitValue = splitPos[node];
                            int left = leftChild[node];
                            int right = rightChild[node];
                            
                            float ori, dir;
                            switch(info & 3){
                            case KDNode::X:
                                ori = origin.x; dir = direction.x;
                                break;
                            case KDNode::Y:
                                ori = origin.y; dir = direction.y;
                                break;
                            case KDNode::Z:
                                ori = origin.z; dir = direction.z;
                                break;
                            }
                        
                            float tSplit = (splitValue - ori) / dir;
                            int lowerChild = 0 < dir ? left : right;
                            int upperChild = 0 < dir ? right : left;
                        
                            if (tHit.x < tSplit){
                                node = lowerChild;
                                if (tSplit < tNext)
                                    stack.Push(ShortStack::Element(upperChild, tNext));
                                tNext = min(tSplit, tNext);
                            }else
                                node = upperChild;

                            info = nodeInfo[node];
                        }
                        
                        tHit.x = tNext;
                        
                        int2 primInfo = primitiveInfo[node];
                        int primHit = -1;
                        int triangles = primInfo.y;
                        while (triangles){
                            int i = __ffs(triangles) - 1;
                            
                            int prim = primIndices[primInfo.x + i];

                            float3 hitCoords;
                            bool hit = TriangleRayIntersection(make_float3(v0[prim]), make_float3(v1[prim]), make_float3(v2[prim]), 
                                                               origin, direction, hitCoords);

                            if (hit && hitCoords.x < tHit.x){
                                primHit = prim;
                                tHit = hitCoords;
                            }
                            
                            triangles -= 1<<i;
                        }

                        if (primHit != -1){
                            // Invalidate the short stack as a new ray has been spawned.
                            stack.Erase();
                            
                            float4 newColor = Lighting(tHit, origin, direction, 
                                                       n0s[primHit], n1s[primHit], n2s[primHit],
                                                       c0s[primHit]);
                            
                            color = BlendColor(color, newColor);
                        }
                        
                    } while(tHit.x < fInfinity && color.w < 0.97f);

                    canvas[id] = make_uchar4(color.x * 255, color.y * 255, color.z * 255, color.w * 255);
                }
            }

            void ShortStack::Trace(IRenderCanvas* canvas, uchar4* canvasData){
                CreateInitialRays(canvas);

                int height = canvas->GetHeight();
                int width = canvas->GetWidth();
                
                int rays = height * width;

                hipMemcpyToSymbol(HIP_SYMBOL(d_rays), &rays, sizeof(int));

                if (visualizeRays){
                    RenderRays(canvasData, rays);
                    return;
                }

                TriangleNode* nodes = map->GetNodes();
                GeometryList* geom = map->GetGeometry();

                unsigned int blocks, threads;
                Calc1DKernelDimensions(rays, blocks, threads, 128);
                START_TIMER(timerID); 
                ShortStackTrace<<<blocks, threads>>>(origin->GetDeviceData(), direction->GetDeviceData(),
                                                     nodes->GetInfoData(), nodes->GetSplitPositionData(),
                                                     nodes->GetLeftData(), nodes->GetRightData(),
                                                     nodes->GetPrimitiveInfoData(),
                                                     map->GetPrimitiveIndices()->GetDeviceData(),
                                                     geom->GetP0Data(), geom->GetP1Data(), geom->GetP2Data(),
                                                     geom->GetNormal0Data(), geom->GetNormal1Data(), geom->GetNormal2Data(),
                                                     geom->GetColor0Data(),
                                                     canvasData);
                PRINT_TIMER(timerID, "Short stack");
                CHECK_FOR_CUDA_ERROR();                                               
            }
            
            void ShortStack::HostTrace(float3 origin, float3 direction, TriangleNode* nodes){
                GeometryList* geom = map->GetGeometry();

                Stack<3> stack;
                
                float3 tHit;
                tHit.x = 0.0f;

                float4 color = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

                do {
                    logger.info << "=== Ray:  " << Convert::ToString(origin) << " -> " << Convert::ToString(direction) << " ===" << logger.end;
                    logger.info << stack.ToString() << logger.end;
                    logger.info << stack.count << logger.end;
                    
                    int node; float tNext;
                    if (stack.IsEmpty()){
                        node = 0;
                        tNext = fInfinity;
                    }else{
                        Element e = stack.Pop();
                        node = e.node;
                        tNext = e.tMax;
                    }

                    char info;
                    hipMemcpy(&info, nodes->GetInfoData() + node, sizeof(char), hipMemcpyDeviceToHost);
                    CHECK_FOR_CUDA_ERROR();

                    while ((info & 3) != KDNode::LEAF){
                        logger.info << "Tracing " << node << " with info " << (int)info << logger.end;
                        
                        float splitValue;
                        hipMemcpy(&splitValue, nodes->GetSplitPositionData() + node, sizeof(float), hipMemcpyDeviceToHost);
                        CHECK_FOR_CUDA_ERROR();
                        
                        int left, right;
                        hipMemcpy(&left, nodes->GetLeftData() + node, sizeof(int), hipMemcpyDeviceToHost);
                        hipMemcpy(&right, nodes->GetRightData() + node, sizeof(int), hipMemcpyDeviceToHost);
                        CHECK_FOR_CUDA_ERROR();

                        // Trace
                        float ori, dir;
                        switch(info & 3){
                        case KDNode::X:
                            ori = origin.x; dir = direction.x;
                            break;
                        case KDNode::Y:
                            ori = origin.y; dir = direction.y;
                            break;
                        case KDNode::Z:
                            ori = origin.z; dir = direction.z;
                            break;
                        }
                        
                        float tSplit = (splitValue - ori) / dir;
                        int lowerChild = 0 < dir ? left : right;
                        int upperChild = 0 < dir ? right : left;

                        if (tHit.x < tSplit){
                            node = lowerChild;
                            if (tSplit < tNext)
                                stack.Push(Element(upperChild, tNext));
                            tNext = min(tSplit, tNext);
                        }else
                            node = upperChild;
                        
                        // New nodes info
                        hipMemcpy(&info, nodes->GetInfoData() + node, sizeof(char), hipMemcpyDeviceToHost);
                    }

                    logger.info << "Found leaf: " << node << "\n" << logger.end;

                    tHit.x = tNext;

                    int2 primInfo;
                    hipMemcpy(&primInfo, nodes->GetPrimitiveInfoData() + node, sizeof(int2), hipMemcpyDeviceToHost);
                    CHECK_FOR_CUDA_ERROR();
                    int primHit = -1;
                    int triangles = primInfo.y;
                    while (triangles){
                        int i = ffs(triangles) - 1;

                        //logger.info << "Testing indice " << primInfo.x << " + " << i << " = " << primInfo.x + i << logger.end;

                        int prim;
                        hipMemcpy(&prim, map->GetPrimitiveIndices()->GetDeviceData() + primInfo.x + i, sizeof(int), hipMemcpyDeviceToHost);
                        CHECK_FOR_CUDA_ERROR();
                        
                        //logger.info << "Testing primitive " << prim << logger.end;

                        float3 v0, v1, v2;
                        hipMemcpy(&v0, geom->GetP0Data() + prim, sizeof(float3), hipMemcpyDeviceToHost);
                        hipMemcpy(&v1, geom->GetP1Data() + prim, sizeof(float3), hipMemcpyDeviceToHost);
                        hipMemcpy(&v2, geom->GetP2Data() + prim, sizeof(float3), hipMemcpyDeviceToHost);
                        CHECK_FOR_CUDA_ERROR();

                        float3 hitCoords;
                        bool hit = TriangleRayIntersection(v0, v1, v2, 
                                                           origin, direction, hitCoords);

                        if (hit && hitCoords.x < tHit.x){
                            primHit = prim;
                            tHit = hitCoords;
                        }
                        
                        triangles -= 1<<i;
                    }
                    
                    //logger.info << "\n" << logger.end;

                    if (primHit != -1){
                        // Invalidate the shortstack as we're now tracing a new ray.
                        stack.Erase();

                        float4 n0, n1, n2;
                        hipMemcpy(&n0, geom->GetNormal0Data() + primHit, sizeof(float4), hipMemcpyDeviceToHost);
                        hipMemcpy(&n1, geom->GetNormal1Data() + primHit, sizeof(float4), hipMemcpyDeviceToHost);
                        hipMemcpy(&n2, geom->GetNormal2Data() + primHit, sizeof(float4), hipMemcpyDeviceToHost);
                        CHECK_FOR_CUDA_ERROR();
                        
                        uchar4 c0;
                        hipMemcpy(&c0, geom->GetColor0Data() + primHit, sizeof(uchar4), hipMemcpyDeviceToHost);                        
                        
                        logger.info << "Prim color: " << Convert::ToString(c0) << logger.end;
                        
                        float4 newColor = Lighting(tHit, origin, direction, 
                                                   n0, n1, n2,
                                                   c0);

                        logger.info << "New color: " << Convert::ToString(newColor) << logger.end;
                        
                        color = BlendColor(color, newColor);

                        logger.info << "Color: " << Convert::ToString(color) << "\n" << logger.end;
                    }

                } while(tHit.x < fInfinity && color.w < 0.97f);

            }

        }
    }
}
