#include "hip/hip_runtime.h"
// Triangle map upper node creator interface for CUDA
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/TriangleMapUpperCreator.h>

#include <Scene/TriangleNode.h>
#include <Utils/CUDA/Convert.h>
#include <Utils/CUDA/GeometryList.h>
#include <Utils/CUDA/TriangleMap.h>
#include <Utils/CUDA/Utils.h>
#include <Utils/CUDA/IntersectionTests.h>

#include <Utils/CUDA/Kernels/PhotonMapDeviceVars.h>

namespace OpenEngine {    
    using namespace Scene;
    namespace Utils {
        namespace CUDA {

            using namespace Kernels;

            TriangleMapUpperCreator::TriangleMapUpperCreator()
                : ITriangleMapCreator(), emptySpaceSplitting(true),
                  emptySpaceThreshold(0.25f), splitAlg(DIVIDE) {
                
                cutCreateTimer(&timerID);

                aabbMin = new CUDADataBlock<1, float4>(1);
                aabbMax = new CUDADataBlock<1, float4>(1);
                tempAabbMin = new CUDADataBlock<1, float4>(1);
                tempAabbMax = new CUDADataBlock<1, float4>(1);

                segments = Segments(1);
                nodeSegments = new CUDADataBlock<1, int>(1);

                splitSide = new CUDADataBlock<1, int>(1);
                splitAddr = new CUDADataBlock<1, int>(1);
                leafSide = new CUDADataBlock<1, int>(1);
                leafAddr = new CUDADataBlock<1, int>(1);
                emptySpacePlanes = new CUDADataBlock<1, char>(1);
                emptySpaceNodes = new CUDADataBlock<1, int>(1);
                emptySpaceAddrs = new CUDADataBlock<1, int>(1);
                nodeIndices = new CUDADataBlock<1, int>(1);
                childSize = new CUDADataBlock<1, int2>(1);
                tempNodeAmount = new CUDADataBlock<1, KDNode::amount>(1);

                // CUDPP doesn't handle removing handles well, so we
                // define them to accept some arbitrary high number of
                // elements here.
                scanConfig.algorithm = CUDPP_SCAN;
                scanConfig.op = CUDPP_ADD;
                scanConfig.datatype = CUDPP_INT;
                scanConfig.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;
                scanSize = 262144;
                
                CUDPPResult res = cudppPlan(&scanHandle, scanConfig, scanSize, 1, 0);
                if (CUDPP_SUCCESS != res)
                    throw Core::Exception("Error creating CUDPP scanPlan for Triangle Map");

                scanInclConfig.algorithm = CUDPP_SCAN;
                scanInclConfig.op = CUDPP_ADD;
                scanInclConfig.datatype = CUDPP_INT;
                scanInclConfig.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;
                scanInclSize = 262144;

                res = cudppPlan(&scanInclHandle, scanInclConfig, scanInclSize, 1, 0);
                if (CUDPP_SUCCESS != res)
                    throw Core::Exception("Error creating CUDPP inclusive scanPlan for Triangle Map");
            }

            TriangleMapUpperCreator::~TriangleMapUpperCreator(){
                
                if (aabbMin) delete aabbMin;
                if (aabbMax) delete aabbMax;
                if (tempAabbMin) delete tempAabbMin;
                if (tempAabbMax) delete tempAabbMax;

                if (nodeSegments) delete nodeSegments;

                if (splitSide) delete splitSide;
                if (splitAddr) delete splitAddr;
                if (leafSide) delete leafSide;
                if (leafAddr) delete leafAddr;
                if (emptySpacePlanes) delete emptySpacePlanes;
                if (emptySpaceNodes) delete emptySpaceNodes;
                if (emptySpaceAddrs) delete emptySpaceAddrs;
                if (nodeIndices) delete nodeIndices;
                if (childSize) delete childSize;
                if (tempNodeAmount) delete tempNodeAmount;
            }

            namespace KernelsHat {
                #include <Utils/CUDA/Kernels/TriangleUpper.h>
                #include <Utils/CUDA/Kernels/TriangleUpperSegment.h>
                #include <Utils/CUDA/Kernels/ReduceSegments.h>
                #include <Utils/CUDA/Kernels/TriangleUpperChildren.h>
                #include <Utils/CUDA/Kernels/TriangleKernels.h>
                #include <Utils/CUDA/Kernels/EmptySpaceSplitting.h>
            }
            using namespace KernelsHat;

            void TriangleMapUpperCreator::Create(TriangleMap* map, 
                                                 CUDADataBlock<1, int>* upperLeafIDs){

                this->map = map;

                primMin = map->primMin;
                primMax = map->primMax;
                primIndices = map->primIndices;
                leafIDs = map->leafIDs;
                
                int activeIndex = 0, activeRange = 1;
                int childrenCreated;
                int triangles = map->GetGeometry()->GetSize();

                hipMemcpyToSymbol(HIP_SYMBOL(d_emptySpaceThreshold), &emptySpaceThreshold, sizeof(float));

                primMin->Extend(0);
                primMax->Extend(0);
                leafIDs->Extend(0);

                // Setup root node!
                int i = 0; KDNode::amount tris = triangles;
                hipMemcpy(map->nodes->GetPrimitiveIndexData(), &i, sizeof(int), hipMemcpyHostToDevice);
                hipMemcpy(map->nodes->GetPrimitiveAmountData(), &tris, sizeof(KDNode::amount), hipMemcpyHostToDevice);
                int parent = 0;
                hipMemcpy(map->nodes->GetParentData(), &parent, sizeof(int), hipMemcpyHostToDevice);
                float4 zero = make_float4(0.0f);
                hipMemcpy(map->nodes->GetAabbMinData(), &zero, sizeof(float4), hipMemcpyHostToDevice);
                hipMemcpy(map->nodes->GetAabbMaxData(), &zero, sizeof(float4), hipMemcpyHostToDevice);
                map->nodes->Resize(1);

                // Setup bounding box info
                aabbMin->Extend(triangles);
                aabbMax->Extend(triangles);
                
                unsigned int blocks, threads;
                Calc1DKernelDimensions(triangles, blocks, threads);
                CalcPrimitiveAabb<<<blocks, threads>>>(map->GetGeometry()->GetP0Data(),
                                                       map->GetGeometry()->GetP1Data(),
                                                       map->GetGeometry()->GetP2Data(),
                                                       aabbMin->GetDeviceData(),
                                                       aabbMax->GetDeviceData(),
                                                       triangles);
                CHECK_FOR_CUDA_ERROR();                

                START_TIMER(timerID);
                while (activeRange > 0){
                    ProcessNodes(activeIndex, activeRange, 
                                 childrenCreated);

                    activeIndex = map->nodes->GetSize() - childrenCreated;
                    activeRange = childrenCreated;

                    //logger.info << "activeIndex = " << map->nodes->GetSize() << " - " << childrenCreated << " = " << activeIndex << logger.end;
                }
                PRINT_TIMER(timerID, "triangle upper map");

                // Extract indices from primMin.
                primIndices->Extend(primMin->GetSize(), false);
                Calc1DKernelDimensions(primMin->GetSize(), blocks, threads, 128);
                ExtractIndexFromAabb<<<blocks, threads>>>(primMin->GetDeviceData(), 
                                                          primIndices->GetDeviceData(),
                                                          primMin->GetSize());
                CHECK_FOR_CUDA_ERROR();

                // If empty space splitting doesn't propagate the
                // aabbs, then we need to do it here.
                if (!emptySpaceSplitting){
                    KernelConf conf = KernelConf1D(leafIDs->GetSize(), 128);
                    PropagateParentAabb<true><<<conf.blocks, conf.threads>>>
                        (leafIDs->GetDeviceData(),
                         map->nodes->GetInfoData(), map->nodes->GetSplitPositionData(), 
                         map->nodes->GetAabbMinData(), map->nodes->GetAabbMaxData(), 
                         map->nodes->GetParentData(), map->nodes->GetChildrenData(),
                         leafIDs->GetSize());
                    CHECK_FOR_CUDA_ERROR();
                }
            }
            
            void TriangleMapUpperCreator::ProcessNodes(int activeIndex, int activeRange, 
                                                       int &childrenCreated){
                int triangles = aabbMin->GetSize();
                //logger.info << "=== Process " << activeRange << " Upper Nodes Starting at " << activeIndex << " === with " << triangles << " primitives" << logger.end;

                // Copy bookkeeping to symbols
                hipMemcpyToSymbol(HIP_SYMBOL(d_activeNodeIndex), &activeIndex, sizeof(int));
                hipMemcpyToSymbol(HIP_SYMBOL(d_activeNodeRange), &activeRange, sizeof(int));
                hipMemcpyToSymbol(HIP_SYMBOL(d_triangles), &triangles, sizeof(int));
                CHECK_FOR_CUDA_ERROR();

                Segment(activeIndex, activeRange);
                CHECK_FOR_CUDA_ERROR();

                // Calculate aabb
                ReduceAabb(activeIndex, activeRange);
                CHECK_FOR_CUDA_ERROR();

                // Calculate children placement
                CreateChildren(activeIndex, activeRange, childrenCreated);
            }

            void TriangleMapUpperCreator::Segment(int activeIndex, int activeRange){
                nodeSegments->Extend(activeRange+1);

                unsigned int blocks, threads;
                Calc1DKernelDimensions(activeRange, blocks, threads);
                NodeSegments<<<blocks, threads>>>(map->nodes->GetPrimitiveAmountData() + activeIndex,
                                                  nodeSegments->GetDeviceData());

                CHECK_FOR_CUDA_ERROR();
                cudppScan(scanHandle, nodeSegments->GetDeviceData(), nodeSegments->GetDeviceData(), activeRange+1);

                int amountOfSegments;
                hipMemcpy(&amountOfSegments, nodeSegments->GetDeviceData() + activeRange, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpyToSymbol(HIP_SYMBOL(d_segments), nodeSegments->GetDeviceData() + activeRange, sizeof(int), 0, hipMemcpyDeviceToDevice);
                CHECK_FOR_CUDA_ERROR();

                segments.Extend(amountOfSegments);

                hipMemset(segments.GetOwnerData(), 0, amountOfSegments * sizeof(int));
                MarkOwnerStart<<<blocks, threads>>>(segments.GetOwnerData(),
                                                    nodeSegments->GetDeviceData());
                CHECK_FOR_CUDA_ERROR();

                cudppScan(scanInclHandle, segments.GetOwnerData(), segments.GetOwnerData(), amountOfSegments);

                Calc1DKernelDimensions(amountOfSegments, blocks, threads);
                CalcSegmentPrimitives<<<blocks, threads>>>(segments.GetOwnerData(),
                                                           nodeSegments->GetDeviceData(),
                                                           map->nodes->GetPrimitiveIndexData(),
                                                           map->nodes->GetPrimitiveAmountData(),
                                                           segments.GetPrimitiveInfoData());
                CHECK_FOR_CUDA_ERROR();
            }
                
            void TriangleMapUpperCreator::ReduceAabb(int &activeIndex, int activeRange){
                // Reduce aabb pr segment
                unsigned int blocks = segments.size;
                unsigned int threads = Segments::SEGMENT_SIZE/2;
                unsigned int smemSize = 2 * 3 * sizeof(float) * segments.SEGMENT_SIZE/2;

                //logger.info << "ReduceSegmentsShared<<<" << blocks << ", " << threads << ", " << smemSize << ">>>" << logger.end;

                ReduceSegmentsShared<<<blocks, threads, smemSize>>>(segments.GetPrimitiveInfoData(),
                                                                   aabbMin->GetDeviceData(), aabbMax->GetDeviceData(),
                                                                   segments.GetAabbMinData(), segments.GetAabbMaxData());
                CHECK_FOR_CUDA_ERROR();

#if CPU_VERIFY
                float4 *finalMin, *finalMax;
                CheckSegmentReduction(activeIndex, activeRange,
                                      segments, &finalMin, &finalMax);
#endif

                if (emptySpaceSplitting){
                    tempAabbMin->Resize(activeRange, false);
                    tempAabbMax->Resize(activeRange, false);
                    
                    Calc1DKernelDimensions(activeRange, blocks, threads);
                    AabbMemset<<<blocks, threads>>>(tempAabbMin->GetDeviceData(),
                                                    tempAabbMax->GetDeviceData());
                    CHECK_FOR_CUDA_ERROR();
                    
                    Calc1DKernelDimensions(segments.GetSize(), blocks, threads);
                    for (int i = 0; i < blocks; ++i){
                        int segs = segments.GetSize() - i * threads;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_segments), &segs, sizeof(int));
                        FinalSegmentedReduce<<<1, threads>>>(segments.GetAabbMinData() + i * threads,
                                                             segments.GetAabbMaxData() + i * threads,
                                                             segments.GetOwnerData() + i * threads,
                                                             tempAabbMin->GetDeviceData(),
                                                             tempAabbMax->GetDeviceData());
                    }
                    int segs = segments.GetSize();
                    hipMemcpyToSymbol(HIP_SYMBOL(d_segments), &segs, sizeof(int));
                    CHECK_FOR_CUDA_ERROR();
                    
                    // Calculate empty space splitting planes before copying aabbs to nodes.
                    CreateEmptySplits(activeIndex, activeRange);
                    
                    hipMemcpy(map->nodes->GetAabbMinData() + activeIndex, 
                               tempAabbMin->GetDeviceData(), 
                               activeRange * sizeof(float4), hipMemcpyDeviceToDevice);
                    hipMemcpy(map->nodes->GetAabbMaxData() + activeIndex, 
                               tempAabbMax->GetDeviceData(), 
                               activeRange * sizeof(float4), hipMemcpyDeviceToDevice);

                }else{
                    Calc1DKernelDimensions(activeRange, blocks, threads);
                    AabbMemset<<<blocks, threads>>>(map->nodes->GetAabbMinData() + activeIndex,
                                                    map->nodes->GetAabbMaxData() + activeIndex);
                    CHECK_FOR_CUDA_ERROR();

                    Calc1DKernelDimensions(segments.GetSize(), blocks, threads);
                    for (int i = 0; i < blocks; ++i){
                        int segs = segments.GetSize() - i * threads;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_segments), &segs, sizeof(int));
                        FinalSegmentedReduce<<<1, threads>>>(segments.GetAabbMinData() + i * threads,
                                                             segments.GetAabbMaxData() + i * threads,
                                                             segments.GetOwnerData() + i * threads,
                                                             map->nodes->GetAabbMinData() + activeIndex,
                                                             map->nodes->GetAabbMaxData() + activeIndex);
                    }
                    int segs = segments.GetSize();
                    hipMemcpyToSymbol(HIP_SYMBOL(d_segments), &segs, sizeof(int));
                    CHECK_FOR_CUDA_ERROR();
                }

#if CPU_VERIFY
                CheckFinalReduction(activeIndex, activeRange, map->nodes, 
                                    finalMin, finalMax);
#endif

                // Calc splitting planes.
                Calc1DKernelDimensions(activeRange, blocks, threads);
                CalcUpperNodeSplitInfo<<<blocks, threads>>>(map->nodes->GetAabbMinData() + activeIndex,
                                                            map->nodes->GetAabbMaxData() + activeIndex,
                                                            map->nodes->GetSplitPositionData() + activeIndex,
                                                            map->nodes->GetInfoData() + activeIndex);
                CHECK_FOR_CUDA_ERROR();
            }

            void TriangleMapUpperCreator::CreateEmptySplits(int &activeIndex, int activeRange){
                bool createdEmptySplits = false;
                hipMemcpyToSymbol(HIP_SYMBOL(d_createdEmptySplits), &createdEmptySplits, sizeof(bool));
                emptySpacePlanes->Resize(activeRange, false);
                emptySpaceNodes->Resize(activeRange+1, false);
                
                unsigned int blocks, threads;
                Calc1DKernelDimensions(activeRange, blocks, threads);
                CalcEmptySpaceSplits<<<blocks, threads>>>(map->nodes->GetAabbMinData() + activeIndex,
                                                          map->nodes->GetAabbMaxData() + activeIndex,
                                                          tempAabbMin->GetDeviceData(), 
                                                          tempAabbMax->GetDeviceData(), 
                                                          emptySpacePlanes->GetDeviceData(),
                                                          emptySpaceNodes->GetDeviceData());
                CHECK_FOR_CUDA_ERROR();

                hipMemcpyFromSymbol(&createdEmptySplits, HIP_SYMBOL(d_createdEmptySplits), sizeof(bool));

                if (createdEmptySplits){
                    emptySpaceAddrs->Resize(activeRange+1, false);
                    cudppScan(scanHandle, emptySpaceAddrs->GetDeviceData(), emptySpaceNodes->GetDeviceData(), activeRange+1);

                    int emptyNodes;
                    hipMemcpy(&emptyNodes, emptySpaceAddrs->GetDeviceData() + emptySpaceAddrs->GetSize()-1, sizeof(int), hipMemcpyDeviceToHost);
                    CHECK_FOR_CUDA_ERROR();

                    map->nodes->Resize(map->nodes->GetSize() + emptyNodes);

                    //logger.info << "Empty space " << emptyNodes << " and nodesize " << map->nodes->GetSize() <<  logger.end;

                    EmptySpaceSplitting2<<<blocks, threads>>>(map->nodes->GetInfoData(), 
                                                             map->nodes->GetSplitPositionData(),
                                                             map->nodes->GetPrimitiveAmountData(), 
                                                             map->nodes->GetParentData(), 
                                                             map->nodes->GetChildrenData(),
                                                             emptySpacePlanes->GetDeviceData(),
                                                             emptySpaceAddrs->GetDeviceData(),
                                                             tempAabbMin->GetDeviceData(),
                                                             tempAabbMax->GetDeviceData(),
                                                             emptyNodes);
                    CHECK_FOR_CUDA_ERROR();

                    /*
                    for (int i = 0; i < emptyNodes; ++i)
                        logger.info << map->nodes->ToString(i + activeIndex + activeRange) << logger.end;
                    // Move nodes to make room for empty space nodes.
                    // That means moving primitiveInfo, using childSize as temp storage
                    // And moving parents, using splitSide as temp storage

                    splitSide->Resize(activeRange);
                    tempNodeAmount->Resize(activeRange);
                    
                    hipMemcpy(splitSide->GetDeviceData(), map->nodes->GetPrimitiveIndexData() + activeIndex, activeRange * sizeof(int), hipMemcpyDeviceToDevice);
                    hipMemcpy(tempNodeAmount->GetDeviceData(), map->nodes->GetPrimitiveAmountData() + activeIndex, activeRange * sizeof(KDNode::amount), hipMemcpyDeviceToDevice);

                    activeIndex += emptyNodes;
                    hipMemcpyToSymbol(HIP_SYMBOL(d_activeNodeIndex), &activeIndex, sizeof(int));

                    hipMemcpy(map->nodes->GetPrimitiveIndexData() + activeIndex, splitSide->GetDeviceData(), activeRange * sizeof(int), hipMemcpyDeviceToDevice);
                    hipMemcpy(map->nodes->GetPrimitiveAmountData() + activeIndex, tempNodeAmount->GetDeviceData(), activeRange * sizeof(KDNode::amount), hipMemcpyDeviceToDevice);
                    segments.IncreaseNodeIDs(emptyNodes);

                    // Create the empty space nodes
                    EmptySpaceSplitting<<<blocks, threads>>>(map->nodes->GetInfoData(), 
                                                             map->nodes->GetSplitPositionData(),
                                                             map->nodes->GetPrimitiveAmountData(), 
                                                             map->nodes->GetParentData() + activeIndex - emptyNodes, 
                                                             map->nodes->GetChildrenData(),
                                                             emptySpacePlanes->GetDeviceData(),
                                                             emptySpaceAddrs->GetDeviceData(),
                                                             tempAabbMin->GetDeviceData(),
                                                             tempAabbMax->GetDeviceData(),
                                                             emptyNodes);
                    CHECK_FOR_CUDA_ERROR();
                    */

                    /*
                    for (int i = 0; i < emptyNodes; ++i)
                        logger.info << map->nodes->ToString(i + activeIndex - emptyNodes) << logger.end;
                    */
                }
            }

            void TriangleMapUpperCreator::CheckSegmentReduction(int activeIndex, int activeRange,
                                                                Segments &segments, 
                                                                float4 **finalMin, 
                                                                float4 **finalMax){
                int2 info[segments.GetSize()];
                hipMemcpy(info, segments.GetPrimitiveInfoData(), 
                           segments.GetSize() * sizeof(int2), hipMemcpyDeviceToHost);

                float4 segMin[segments.GetSize()];
                hipMemcpy(segMin, segments.GetAabbMinData(), 
                           segments.GetSize() * sizeof(float4), hipMemcpyDeviceToHost);
                float4 segMax[segments.GetSize()];
                hipMemcpy(segMax, segments.GetAabbMaxData(), 
                           segments.GetSize() * sizeof(float4), hipMemcpyDeviceToHost);

                for (int i = 0; i < segments.GetSize(); ++i){
                    int index = info[i].x;
                    int range = info[i].y;

                    float4 cpuMin[range];
                    hipMemcpy(cpuMin, aabbMin->GetDeviceData() + index, 
                               range * sizeof(float4), hipMemcpyDeviceToHost);
                    float4 cpuMax[range];
                    hipMemcpy(cpuMax, aabbMax->GetDeviceData() + index, 
                               range * sizeof(float4), hipMemcpyDeviceToHost);

                    for (int j = 1; j < range; ++j){
                        cpuMin[0] = min(cpuMin[0], cpuMin[j]);
                        cpuMax[0] = max(cpuMax[0], cpuMax[j]);
                    }
                    
                    if (cpuMin[0].x != segMin[i].x || cpuMin[0].y != segMin[i].y || cpuMin[0].z != segMin[i].z)
                        throw Core::Exception("aabbMin error at segment " + Utils::Convert::ToString(i) +
                                              ": CPU min " + Utils::CUDA::Convert::ToString(cpuMin[0])
                                              + ", GPU min " + Utils::CUDA::Convert::ToString(segMin[i]));

                    if (cpuMax[0].x != segMax[i].x || cpuMax[0].y != segMax[i].y || cpuMax[0].z != segMax[i].z)
                        throw Core::Exception("aabbMax error at segment " + Utils::Convert::ToString(i) +
                                              ": CPU max " + Utils::CUDA::Convert::ToString(cpuMax[0])
                                              + ", GPU max " + Utils::CUDA::Convert::ToString(segMax[i]));
                }

                int segOwner[segments.GetSize()];
                hipMemcpy(segOwner, segments.GetOwnerData(), 
                           segments.GetSize() * sizeof(int), hipMemcpyDeviceToHost);

                (*finalMin) = new float4[activeRange];
                (*finalMax) = new float4[activeRange];

                int owner0 = segOwner[0];
                float4 localMin = segMin[0];
                float4 localMax = segMax[0];
                for (int i = 1; i < segments.GetSize(); ++i){
                    int owner1 = segOwner[i];
                    if (owner0 != owner1){
                        (*finalMin)[owner0 - activeIndex] = localMin;
                         (*finalMax)[owner0 - activeIndex] = localMax;
                        owner0 = segOwner[i];
                        localMin = segMin[i];
                        localMax = segMax[i];
                    }else{
                        localMin = min(localMin, segMin[i]);
                        localMax = max(localMax, segMax[i]);
                    }
                }
                (*finalMin)[owner0 - activeIndex] = localMin;
                (*finalMax)[owner0 - activeIndex] = localMax;
            }

            void TriangleMapUpperCreator::CheckFinalReduction(int activeIndex, int activeRange,
                                                              TriangleNode* nodes, 
                                                              float4 *finalMin, 
                                                              float4 *finalMax){
                
                float4 gpuMin[activeRange];
                hipMemcpy(gpuMin, nodes->GetAabbMinData() + activeIndex,
                           activeRange * sizeof(float4), hipMemcpyDeviceToHost);
                float4 gpuMax[activeRange];
                hipMemcpy(gpuMax, nodes->GetAabbMaxData() + activeIndex,
                           activeRange * sizeof(float4), hipMemcpyDeviceToHost);

                for (int i = 0; i < activeRange; ++i){
                    if (finalMin[i].x != gpuMin[i].x || finalMin[i].y != gpuMin[i].y || finalMin[i].z != gpuMin[i].z)
                        throw Core::Exception("Final aabbMin error at node " + Utils::Convert::ToString(i + activeIndex) +
                                              ": CPU min " + Utils::CUDA::Convert::ToString(finalMin[i])
                                              + ", GPU min " + Utils::CUDA::Convert::ToString(gpuMin[i]));

                    if (finalMax[i].x != gpuMax[i].x || finalMax[i].y != gpuMax[i].y || finalMax[i].z != gpuMax[i].z)
                        throw Core::Exception("Final aabbMax error at node " + Utils::Convert::ToString(i + activeIndex) +
                                              ": CPU max " + Utils::CUDA::Convert::ToString(finalMax[i])
                                              + ", GPU max " + Utils::CUDA::Convert::ToString(gpuMax[i]));
                }

                delete finalMin;
                delete finalMax;
            }

            void TriangleMapUpperCreator::CreateChildren(int activeIndex, int activeRange,
                                                         int &childrenCreated){

                TriangleNode* nodes = map->GetNodes();
                int triangles = aabbMin->GetSize();

                unsigned int blocks = segments.GetSize();
                unsigned int threads = Segments::SEGMENT_SIZE;

                splitSide->Extend(triangles * 2, false);
                splitAddr->Extend(triangles * 2 + 1, false);
                leafSide->Extend(triangles * 2, false);
                leafAddr->Extend(triangles * 2 + 1, false);
                childSize->Extend(activeRange, false);
                int childStartAddr = nodes->GetSize();
                nodes->Extend(nodes->GetSize() + activeRange * 2);
                
                switch(splitAlg){
                case BOX:
                    SetSplitSide<<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                      segments.GetOwnerData(),
                                                      nodes->GetInfoData(),
                                                      nodes->GetSplitPositionData(),
                                                      aabbMin->GetDeviceData(),
                                                      aabbMax->GetDeviceData(),
                                                      splitSide->GetDeviceData());
                    break;
                case DIVIDE:
                    SetDivideSide<false><<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                              segments.GetOwnerData(),
                                                              nodes->GetInfoData(),
                                                              nodes->GetSplitPositionData(),
                                                              aabbMin->GetDeviceData(), aabbMax->GetDeviceData(),
                                                              nodes->GetAabbMinData(), nodes->GetAabbMaxData(),
                                                              map->GetGeometry()->GetP0Data(), map->GetGeometry()->GetP1Data(), map->GetGeometry()->GetP2Data(), 
                                                              splitSide->GetDeviceData());
                    break;
                case SPLIT:
                    break;
                }
                CHECK_FOR_CUDA_ERROR();

                cudppScan(scanHandle, splitAddr->GetDeviceData(), splitSide->GetDeviceData(), triangles * 2 + 1);
                CHECK_FOR_CUDA_ERROR();

                /*
                if (activeRange == 86){
                    int node = 356;
                    logger.info << nodes->ToString(node) << logger.end;

                    logger.info << "primMin: " << Convert::ToString(aabbMin->GetDeviceData() + 7935, 1) << logger.end;
                    logger.info << "primMax: " << Convert::ToString(aabbMax->GetDeviceData() + 7935, 1) << logger.end;

                    logger.info << "v0: " << Convert::ToString(map->GetGeometry()->GetP0Data() + 2, 1) << logger.end;
                    logger.info << "v1: " << Convert::ToString(map->GetGeometry()->GetP1Data() + 2, 1) << logger.end;
                    logger.info << "v2: " << Convert::ToString(map->GetGeometry()->GetP2Data() + 2, 1) << logger.end;

                    float3 v0, v1, v2, nodeMin, nodeMax;
                    hipMemcpy(&v0, map->GetGeometry()->GetP0Data() + 2, sizeof(float3), hipMemcpyDeviceToHost);
                    hipMemcpy(&v1, map->GetGeometry()->GetP1Data() + 2, sizeof(float3), hipMemcpyDeviceToHost);
                    hipMemcpy(&v2, map->GetGeometry()->GetP2Data() + 2, sizeof(float3), hipMemcpyDeviceToHost);
                    hipMemcpy(&nodeMin, nodes->GetAabbMinData() + node, sizeof(float3), hipMemcpyDeviceToHost);
                    hipMemcpy(&nodeMax, nodes->GetAabbMaxData() + node, sizeof(float3), hipMemcpyDeviceToHost);
                    char axis;
                    hipMemcpy(&axis, nodes->GetInfoData() + node, sizeof(char), hipMemcpyDeviceToHost);
                    float splitPos;
                    hipMemcpy(&splitPos, nodes->GetSplitPositionData() + node, sizeof(float), hipMemcpyDeviceToHost);

                    if (TriangleAabbIntersectionStep3(v0, v1, v2, nodeMin, nodeMax))
                        logger.info << "included in parent\n" << logger.end;
                    else
                        logger.info << "WTF!!\n" << logger.end;

                    bool hit =  TriangleAabbIntersectionStep3(v0, v1, v2, nodeMin,
                                                               make_float3(axis == KDNode::X ? splitPos : nodeMax.x,
                                                                           axis == KDNode::Y ? splitPos : nodeMax.y,
                                                                           axis == KDNode::Z ? splitPos : nodeMax.z));
                    logger.info << "included in left: " << hit << "\n" << logger.end;
                }
                */

#ifdef CPU_VERIFY
                CheckSplits();
                CHECK_FOR_CUDA_ERROR();
#endif

                int newTriangles;
                hipMemcpy(&newTriangles, splitAddr->GetDeviceData() + triangles * 2, sizeof(int), hipMemcpyDeviceToHost);
                //logger.info << "new triangles " << newTriangles << logger.end;
                CHECK_FOR_CUDA_ERROR();
                
                if (newTriangles < triangles)
                    throw Exception("New triangles amount " + Utils::Convert::ToString(newTriangles) + " was below old. WTF");
                
                bool createdLeafs = false;
                hipMemcpyToSymbol(HIP_SYMBOL(d_createdLeafs), &createdLeafs, sizeof(bool));

                unsigned int hatte, traade;
                Calc1DKernelDimensions(activeRange, hatte, traade);
                CalcNodeChildSize<<<hatte, traade>>>(nodes->GetPrimitiveIndexData() + activeIndex,
                                                     nodes->GetPrimitiveAmountData() + activeIndex,
                                                     splitAddr->GetDeviceData(),
                                                     childSize->GetDeviceData());
                CHECK_FOR_CUDA_ERROR();
                hipMemcpyFromSymbol(&createdLeafs, HIP_SYMBOL(d_createdLeafs), sizeof(bool));

                if (createdLeafs){
                    //logger.info << "Created leafs. Split resulted in " << newTriangles << " triangles." << logger.end;

                    SetPrimitiveLeafSide<<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                              segments.GetOwnerData(),
                                                              childSize->GetDeviceData(),
                                                              splitSide->GetDeviceData(),
                                                              leafSide->GetDeviceData());
                    CHECK_FOR_CUDA_ERROR();
                    
                    cudppScan(scanHandle, leafAddr->GetDeviceData(), leafSide->GetDeviceData(), triangles * 2 + 1);

                    int leafTriangles;
                    hipMemcpy(&leafTriangles, leafAddr->GetDeviceData() + triangles * 2, sizeof(int), hipMemcpyDeviceToHost);
                    
                    newTriangles -= leafTriangles;

                    tempAabbMin->Extend(newTriangles);
                    tempAabbMax->Extend(newTriangles);
                    int upperLeafPrimitives = primMax->GetSize();
                    primMax->Extend(upperLeafPrimitives + leafTriangles);
                    primMin->Extend(upperLeafPrimitives + leafTriangles);

                    SplitTriangles<<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                        segments.GetOwnerData(),
                                                        nodes->GetInfoData(),
                                                        nodes->GetSplitPositionData(),
                                                        splitSide->GetDeviceData(),
                                                        splitAddr->GetDeviceData(),
                                                        leafSide->GetDeviceData(),
                                                        leafAddr->GetDeviceData(),
                                                        aabbMin->GetDeviceData(),
                                                        aabbMax->GetDeviceData(),
                                                        tempAabbMin->GetDeviceData(),
                                                        tempAabbMax->GetDeviceData(),
                                                        primMin->GetDeviceData() + upperLeafPrimitives,
                                                        primMax->GetDeviceData() + upperLeafPrimitives);
                    CHECK_FOR_CUDA_ERROR();
                    std::swap(aabbMin, tempAabbMin);
                    std::swap(aabbMax, tempAabbMax);
                    
                    MarkNodeLeafs<<<hatte, traade>>>(childSize->GetDeviceData(),
                                                     leafSide->GetDeviceData());
                    CHECK_FOR_CUDA_ERROR();
                    
                    cudppScan(scanHandle, splitSide->GetDeviceData(), leafSide->GetDeviceData(), activeRange * 2 + 1);
                    CHECK_FOR_CUDA_ERROR();
                    
                    int leafNodes;
                    hipMemcpy(&leafNodes, splitSide->GetDeviceData() + activeRange * 2, sizeof(int), hipMemcpyDeviceToHost);
                    hipMemcpyToSymbol(HIP_SYMBOL(d_leafNodes), splitSide->GetDeviceData() + activeRange * 2, sizeof(int), 0, hipMemcpyDeviceToDevice);

                    CreateUpperChildren<false>
                        <<<hatte, traade>>>(NULL, nodes->GetPrimitiveIndexData(),
                                            nodes->GetPrimitiveAmountData(),
                                            childSize->GetDeviceData(),
                                            splitAddr->GetDeviceData(),
                                            leafAddr->GetDeviceData(),
                                            splitSide->GetDeviceData(),
                                            nodes->GetChildrenData(),
                                            nodes->GetParentData(),
                                            upperLeafPrimitives,
                                            childStartAddr);
                    CHECK_FOR_CUDA_ERROR();

                    childrenCreated = activeRange * 2 - leafNodes;

                    int upperNodeLeafs = leafIDs->GetSize();
                    leafIDs->Extend(leafIDs->GetSize() + leafNodes);
                    Calc1DKernelDimensions(leafNodes, blocks, threads);
                    int leafIndex = nodes->GetSize() - activeRange * 2;
                    MarkLeafNodes
                        <<<blocks, threads>>>(leafIDs->GetDeviceData() + upperNodeLeafs, 
                                              nodes->GetInfoData() + leafIndex,
                                              leafIndex, leafNodes);

                }else{
                    //logger.info << "No leafs created. Split resulted in " << newTriangles << " triangles."  << logger.end;

                    tempAabbMin->Extend(newTriangles);
                    tempAabbMax->Extend(newTriangles);

                    CreateUpperChildren<false>
                        <<<hatte, traade>>>(NULL, nodes->GetPrimitiveIndexData(),
                                            nodes->GetPrimitiveAmountData(),
                                            childSize->GetDeviceData(),
                                            splitAddr->GetDeviceData(),
                                            nodes->GetChildrenData(),
                                            nodes->GetParentData(),
                                            childStartAddr);
                    CHECK_FOR_CUDA_ERROR();

                    SplitTriangles<<<blocks, threads>>>(segments.GetPrimitiveInfoData(),
                                                        segments.GetOwnerData(),
                                                        nodes->GetInfoData(),
                                                        nodes->GetSplitPositionData(),
                                                        splitSide->GetDeviceData(),
                                                        splitAddr->GetDeviceData(),
                                                        aabbMin->GetDeviceData(),
                                                        aabbMax->GetDeviceData(),
                                                        tempAabbMin->GetDeviceData(),
                                                        tempAabbMax->GetDeviceData());
                    CHECK_FOR_CUDA_ERROR();

                    std::swap(aabbMin, tempAabbMin);
                    std::swap(aabbMax, tempAabbMax);
                    
                    childrenCreated = activeRange * 2;
                }

                if (emptySpaceSplitting){
                    Calc1DKernelDimensions(activeRange, blocks, threads);
                    PropagateAabbToChildren<false><<<blocks, threads>>>(NULL, nodes->GetInfoData(),
                                                                        nodes->GetSplitPositionData(),
                                                                        nodes->GetAabbMinData(),
                                                                        nodes->GetAabbMaxData(),
                                                                        nodes->GetChildrenData());
                    CHECK_FOR_CUDA_ERROR();
                }
                
#if CPU_VERIFY
                /*
                for (int i = activeIndex; i < map->nodes->GetSize() - childrenCreated; ++i)
                    logger.info << map->nodes->ToString(i) << logger.end;
                */

                // Check that all primitive bounding boxes are tight or inside the primitive
                CheckPrimAabb(aabbMin, aabbMax);
                
                // Check that the nodes aabb cover all their respective primitives.
                for (int i = activeIndex; i < activeIndex + activeRange; ++i){
                    float4 parentAabbMin, parentAabbMax;
                    hipMemcpy(&parentAabbMin, nodes->GetAabbMinData() + i, sizeof(float4), hipMemcpyDeviceToHost);
                    hipMemcpy(&parentAabbMax, nodes->GetAabbMaxData() + i, sizeof(float4), hipMemcpyDeviceToHost);

                    CheckUpperNode(i, parentAabbMin, parentAabbMax, activeRange);
                }
#endif
            }
            
            void TriangleMapUpperCreator::CheckPrimAabb(CUDADataBlock<1, float4> *aabbMin, 
                                                        CUDADataBlock<1, float4> *aabbMax){
                int triangles = aabbMax->GetSize();
                GeometryList* geom = map->GetGeometry();

                float4 primAabbMin[triangles];
                hipMemcpy(primAabbMin, aabbMin->GetDeviceData(), triangles * sizeof(float4), hipMemcpyDeviceToHost);
                float4 primAabbMax[triangles];
                hipMemcpy(primAabbMax, aabbMax->GetDeviceData(), triangles * sizeof(float4), hipMemcpyDeviceToHost);
                CHECK_FOR_CUDA_ERROR();
                for (int i = 0; i < triangles; ++i){
                    int index = primAabbMin[i].w;
                    float4 p0, p1, p2;
                    hipMemcpy(&p0, geom->p0->GetDeviceData() + index, sizeof(float4), hipMemcpyDeviceToHost);
                    hipMemcpy(&p1, geom->p1->GetDeviceData() + index, sizeof(float4), hipMemcpyDeviceToHost);
                    hipMemcpy(&p2, geom->p2->GetDeviceData() + index, sizeof(float4), hipMemcpyDeviceToHost);

                    float4 aabbMin = min(p0, min(p1, p2));
                    float4 aabbMax = max(p0, max(p1, p2));

                    if (primAabbMin[i].x < aabbMin.x || 
                        primAabbMin[i].y < aabbMin.y || 
                        primAabbMin[i].z < aabbMin.z ||
                        aabbMax.x < primAabbMax[i].x ||
                        aabbMax.y < primAabbMax[i].y ||
                        aabbMax.z < primAabbMax[i].z)
                        throw Exception("Element " + Utils::Convert::ToString(i) + 
                                        " with cornors " + Convert::ToString(p0) +
                                        ", " + Convert::ToString(p1) + " and " + Convert::ToString(p2) +
                                        " is not strictly contained in aabb " + Convert::ToString(primAabbMin[i]) +
                                        " -> " + Convert::ToString(primAabbMax[i]));
                }
                CHECK_FOR_CUDA_ERROR();
            }
            
            void TriangleMapUpperCreator::CheckUpperNode(int index, float4 calcedAabbMin, 
                                                         float4 calcedAabbMax, int activeRange){
                //logger.info << "Checking node " << index << logger.end;
                char axis;
                hipMemcpy(&axis, map->nodes->GetInfoData() + index, sizeof(char), hipMemcpyDeviceToHost);
                
                if (axis == KDNode::LEAF){
                    CheckUpperLeaf(index, calcedAabbMin, calcedAabbMax);                    
                }else{
                    float splitPos;
                    hipMemcpy(&splitPos, map->nodes->GetSplitPositionData() + index, sizeof(float), hipMemcpyDeviceToHost);

                    int2 childrenIndex;
                    hipMemcpy(&childrenIndex, map->nodes->GetChildrenData() + index, sizeof(int2), hipMemcpyDeviceToHost);
                    
                    int leftIndex = childrenIndex.x;

                    float4 leftAabbMin = calcedAabbMin;
                    float4 leftAabbMax = make_float4(axis == KDNode::X ? splitPos : calcedAabbMax.x,
                                                     axis == KDNode::Y ? splitPos : calcedAabbMax.y,
                                                     axis == KDNode::Z ? splitPos : calcedAabbMax.z,
                                                     calcedAabbMax.w);

                    if (leftIndex < map->nodes->GetSize() - 2 * activeRange)
                        CheckUpperNode(leftIndex, leftAabbMin, leftAabbMax, activeRange);
                    else
                        CheckUpperLeaf(leftIndex, leftAabbMin, leftAabbMax);

                    int rightIndex = childrenIndex.y;
                        
                    float4 rightAabbMin = make_float4(axis == KDNode::X ? splitPos : calcedAabbMin.x,
                                                      axis == KDNode::Y ? splitPos : calcedAabbMin.y,
                                                      axis == KDNode::Z ? splitPos : calcedAabbMin.z,
                                                      calcedAabbMin.w);
                    float4 rightAabbMax = calcedAabbMax;

                    if (rightIndex < map->nodes->GetSize() - 2 * activeRange)
                        CheckUpperNode(rightIndex, rightAabbMin, rightAabbMax, activeRange);
                    else
                        CheckUpperLeaf(rightIndex, rightAabbMin, rightAabbMax);
                }                
            }

            void TriangleMapUpperCreator::CheckUpperLeaf(int index, float4 calcedAabbMin, float4 calcedAabbMax){
                //logger.info << "Node " << index << " is a leaf" << logger.end;
                int primIndex;
                hipMemcpy(&primIndex, map->nodes->GetPrimitiveIndexData() + index, sizeof(int), hipMemcpyDeviceToHost);
                KDNode::amount primAmount;
                hipMemcpy(&primAmount, map->nodes->GetPrimitiveAmountData() + index, sizeof(KDNode::amount), hipMemcpyDeviceToHost);
                CHECK_FOR_CUDA_ERROR();
                
                bool isLeaf = primAmount < TriangleNode::MAX_LOWER_SIZE;
                for (int j = primIndex; j < primIndex + primAmount; ++j){
                    float4 h_primMin, h_primMax;
                    if (isLeaf){
                        hipMemcpy(&h_primMin, primMin->GetDeviceData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                        hipMemcpy(&h_primMax, primMax->GetDeviceData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                    }else{
                        hipMemcpy(&h_primMin, aabbMin->GetDeviceData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                        hipMemcpy(&h_primMax, aabbMax->GetDeviceData() + j, sizeof(float4), hipMemcpyDeviceToHost);
                    }
                    CHECK_FOR_CUDA_ERROR();
                            
                    if (!aabbContains(calcedAabbMin, calcedAabbMax, h_primMin))
                        throw Core::Exception("primitive  " + Utils::Convert::ToString(j) + 
                                              "'s min " + Convert::ToString(h_primMin) +
                                              " not included in node " + Utils::Convert::ToString(index) +
                                              "'s aabb " + Convert::ToString(calcedAabbMin) +
                                              " -> " + Convert::ToString(calcedAabbMax));

                    if (!aabbContains(calcedAabbMin, calcedAabbMax, h_primMax))
                        throw Core::Exception("primitive  " + Utils::Convert::ToString(j) + 
                                              "'s max " + Convert::ToString(h_primMax) +
                                              " not included in node " + Utils::Convert::ToString(index) +
                                              "'s aabb " + Convert::ToString(calcedAabbMin) +
                                              " -> " + Convert::ToString(calcedAabbMax));
                }
            }

            void TriangleMapUpperCreator::CheckSplits() {
                int triangles = splitSide->GetSize() / 2;

                int sides[triangles * 2];
                hipMemcpy(sides, splitSide->GetDeviceData(), triangles * 2 * sizeof(int), hipMemcpyDeviceToHost);
                CHECK_FOR_CUDA_ERROR();

                int addrs[triangles * 2];
                hipMemcpy(addrs, splitAddr->GetDeviceData(), (triangles * 2 + 1) * sizeof(int), hipMemcpyDeviceToHost);
                CHECK_FOR_CUDA_ERROR();

                for (int i = 0; i < triangles; ++i){
                    // Check that a bounding box is at least assigned to one side.
                    if (sides[i] + sides[triangles + i] == 0){
                        throw Exception("Bounding box " + Utils::Convert::ToString(i) +
                                        " was neither left nor right.");
                    }
                }

                int prims = 0;
                for (int i = 1; i < triangles * 2 + 1; ++i){
                    prims += sides[i-1];
                    if (prims != addrs[i])
                        throw Exception("Stuff went wrong at bounding box " + Utils::Convert::ToString(i));
                }

                //logger.info << "New prims " << prims << logger.end;
            }

        }
    }
}
