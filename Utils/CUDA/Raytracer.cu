#include "hip/hip_runtime.h"
// Raytracer class for CUDA
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/Raytracer.h>

#include <Display/IViewingVolume.h>
#include <Display/IRenderCanvas.h>
#include <Scene/TriangleNode.h>
#include <Utils/CUDA/Convert.h>
#include <Utils/CUDA/TriangleMap.h>
#include <Utils/CUDA/Utils.h>

#define MAX_THREADS 64
#define MIN_BLOCKS 4

namespace OpenEngine {
    using namespace Display;
    using namespace Resources;
    using namespace Resources::CUDA;
    using namespace Scene;
    namespace Utils {
        namespace CUDA {

#include <Utils/CUDA/Kernels/ColorKernels.h>
            
            RayTracer::RayTracer(TriangleMap* map)
                : map(map) {
                
                cutCreateTimer(&timerID);

                float3 lightPosition = make_float3(0.0f, 4.0f, 0.0f);
                hipMemcpyToSymbol(HIP_SYMBOL(d_lightPosition), &lightPosition, sizeof(float3));
                float3 lightColor = make_float3(1.0f, 0.92f, 0.8f);
                float3 ambient = lightColor * 0.3f;
                hipMemcpyToSymbol(HIP_SYMBOL(d_lightAmbient), &ambient, sizeof(float3));
                float3 diffuse = lightColor * 0.7f;
                hipMemcpyToSymbol(HIP_SYMBOL(d_lightDiffuse), &diffuse, sizeof(float3));
                float3 specular = lightColor * 0.3f;
                hipMemcpyToSymbol(HIP_SYMBOL(d_lightSpecular), &specular, sizeof(float3));
                CHECK_FOR_CUDA_ERROR();
            }

            RayTracer::~RayTracer() {}

            __constant__ int d_rays;

            __device__ __host__ void TraceNode(float3 origin, float3 direction, 
                                               char axis, float splitPos,
                                               int left, int right, float tMin,
                                               int &node, float &tNext){
                float ori, dir;
                switch(axis){
                case KDNode::X:
                    ori = origin.x; dir = direction.x;
                    break;
                case KDNode::Y:
                    ori = origin.y; dir = direction.y;
                    break;
                case KDNode::Z:
                    ori = origin.z; dir = direction.z;
                    break;
                }
                
                float tSplit = (splitPos - ori) / dir;

                if (tMin < tSplit){
                    node = 0 < dir ? left : right;
                    tNext = min(tSplit, tNext);
                }else
                    node = 0 < dir ? right : left;
            }

            __global__ void
            __launch_bounds__(MAX_THREADS, MIN_BLOCKS)
            KDRestartWoop(float4* origins, float4* directions,
                          char* nodeInfo, float* splitPos,
                          int2* children,
                          int* nodePrimIndex, KDNode::bitmap* primBitmap,
                          int *primIndices, 
                          float4 *woop0, float4 *woop1, float4 *woop2,
                          float4 *n0s, float4 *n1s, float4 *n2s,
                          uchar4 *c0s,
                          uchar4 *canvas,
                          int screenWidth){

                int id = blockDim.x * blockIdx.x + threadIdx.x;
                
                if (id < d_rays){                

                    id = IRayTracer::PacketIndex(id, screenWidth);
    
                    float3 origin = make_float3(origins[id]);
                    float3 direction = make_float3(directions[id]);

                    float3 tHit;
                    tHit.x = 0.0f;

                    float4 color = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

                    do {
                        float tNext = fInfinity;
                        int node = 0;
                        char info = nodeInfo[node];
                        
                        while((info & 3) != KDNode::LEAF){
                            // Trace
                            float splitValue = splitPos[node];
                            int2 childPair = children[node];

                            TraceNode(origin, direction, info & 3, splitValue, childPair.x, childPair.y, tHit.x,
                                      node, tNext);
                                                        
                            info = nodeInfo[node];
                        }

                        tHit.x = tNext;

                        int primIndex = nodePrimIndex[node];
                        KDNode::bitmap triangles = primBitmap[node];
                        int primHit = -1;
                        while (triangles){
                            int i = firstBitSet(triangles) - 1;
                            int prim = primIndices[primIndex + i];

                            IRayTracer::Woop(woop0, woop1, woop2, prim,
                                             origin, direction, primHit, tHit);
                            
                            triangles -= KDNode::bitmap(1)<<i;
                        }

                        if (primHit != -1){
                            float4 newColor = Lighting(tHit, origin, direction, 
                                                       n0s[primHit], n1s[primHit], n2s[primHit],
                                                       c0s[primHit]);
                            color = BlendColor(color, newColor);

                            tHit.x = 0.0f;
                        }
                    } while(tHit.x < fInfinity && color.w < 0.97f);

                    canvas[id] = make_uchar4(color.x * 255, color.y * 255, color.z * 255, color.w * 255);
                }
            }

            __global__ void 
            __launch_bounds__(MAX_THREADS, MIN_BLOCKS) 
                KDRestart(float4* origins, float4* directions,
                          char* nodeInfo, float* splitPos,
                          int2* children,
                          int* nodePrimIndex, KDNode::bitmap* primBitmap,
                          int *primIndices, 
                          float4 *v0, float4 *v1, float4 *v2,
                          float4 *n0s, float4 *n1s, float4 *n2s,
                          uchar4 *c0s,
                          uchar4 *canvas,
                          int screenWidth){
                
                int id = blockDim.x * blockIdx.x + threadIdx.x;
                
                if (id < d_rays){
                
                    id = IRayTracer::PacketIndex(id, screenWidth);
    
                    float3 origin = make_float3(origins[id]);
                    float3 direction = make_float3(directions[id]);

                    float3 tHit;
                    tHit.x = 0.0f;

                    float4 color = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

                    do {
                        float tNext = fInfinity;
                        int node = 0;
                        char info = nodeInfo[node];
                        
                        while((info & 3) != KDNode::LEAF){
                            // Trace
                            float splitValue = splitPos[node];
                            int2 childPair = children[node];

                            TraceNode(origin, direction, info & 3, splitValue, childPair.x, childPair.y, tHit.x,
                                      node, tNext);
                                                        
                            info = nodeInfo[node];
                        }

                        tHit.x = tNext;

                        int primIndex = nodePrimIndex[node];
                        KDNode::bitmap triangles = primBitmap[node];
                        int primHit = -1;
                        while (triangles){
                            int i = firstBitSet(triangles) - 1;
                            int prim = primIndices[primIndex + i];

                            IRayTracer::MoellerTrumbore(v0, v1, v2, prim,
                                                        origin, direction, primHit, tHit);
                            
                            triangles -= KDNode::bitmap(1)<<i;
                        }

                        if (primHit != -1){
                            float4 newColor = Lighting(tHit, origin, direction, 
                                                       n0s[primHit], n1s[primHit], n2s[primHit],
                                                       c0s[primHit]);

                            color = BlendColor(color, newColor);

                            tHit.x = 0.0f;
                        }
                    } while(tHit.x < fInfinity && color.w < 0.97f);

                    canvas[id] = make_uchar4(color.x * 255, color.y * 255, color.z * 255, color.w * 255);
                }
            }

            void RayTracer::Trace(IRenderCanvas* canvas, uchar4* canvasData){
                //logger.info << "Trace!" << logger.end;

                CreateInitialRays(canvas);

                int height = canvas->GetHeight();
                int width = canvas->GetWidth();
                
                int rays = height * width;

                hipMemcpyToSymbol(HIP_SYMBOL(d_rays), &rays, sizeof(int));

                if (visualizeRays){
                    RenderRays(canvasData, rays);
                    return;
                }

                TriangleNode* nodes = map->GetNodes();
                GeometryList* geom = map->GetGeometry();

                if (this->intersectionAlgorithm == WOOP){
                    float4 *woop0, *woop1, *woop2;
                    geom->GetWoopValues(&woop0, &woop1, &woop2);

                    KernelConf conf = KernelConf1D(rays, 64);
                    START_TIMER(timerID);
                    KDRestartWoop<<<conf.blocks, conf.threads>>>
                        (origin->GetDeviceData(), direction->GetDeviceData(),
                         nodes->GetInfoData(), nodes->GetSplitPositionData(),
                         nodes->GetChildrenData(),
                         nodes->GetPrimitiveIndexData(),
                         nodes->GetPrimitiveBitmapData(),
                         map->GetPrimitiveIndices()->GetDeviceData(),
                         woop0, woop1, woop2,
                         geom->GetNormal0Data(), geom->GetNormal1Data(), geom->GetNormal2Data(),
                         geom->GetColor0Data(),
                         canvasData,
                         width);
                    PRINT_TIMER(timerID, "KDRestart with Woop intersection");
                }else{               
                    unsigned int blocks, threads;
                    Calc1DKernelDimensions(rays, blocks, threads, MAX_THREADS);
                    START_TIMER(timerID);
                    KDRestart<<<blocks, threads>>>(origin->GetDeviceData(), direction->GetDeviceData(),
                                                   nodes->GetInfoData(), nodes->GetSplitPositionData(),
                                                   nodes->GetChildrenData(),
                                                   nodes->GetPrimitiveIndexData(),
                                                   nodes->GetPrimitiveBitmapData(),
                                                   map->GetPrimitiveIndices()->GetDeviceData(),
                                                   geom->GetP0Data(), geom->GetP1Data(), geom->GetP2Data(),
                                                   geom->GetNormal0Data(), geom->GetNormal1Data(), geom->GetNormal2Data(),
                                                   geom->GetColor0Data(),
                                                   canvasData,
                                                   width);
                    PRINT_TIMER(timerID, "KDRestart with Möller-Trumbore");
                }                                
                CHECK_FOR_CUDA_ERROR();
            }

            void RayTracer::HostTrace(int x, int y, TriangleNode* nodes){

                int id = x + y * screenWidth;
                float3 ori, dir;
                hipMemcpy(&ori, origin->GetDeviceData() + id, sizeof(float3), hipMemcpyDeviceToHost);
                hipMemcpy(&dir, direction->GetDeviceData() + id, sizeof(float3), hipMemcpyDeviceToHost);

                GeometryList* geom = map->GetGeometry();

                float3 tHit;
                tHit.x = 0.0f;

                float4 color = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

                do {
                    logger.info << "=== Ray:  " << Convert::ToString(ori) << " -> " << Convert::ToString(dir) << " ===\n" << logger.end;
                
                    float tNext = fInfinity;
                    int node = 0;
                    char info;
                    hipMemcpy(&info, nodes->GetInfoData() + node, sizeof(char), hipMemcpyDeviceToHost);
                    CHECK_FOR_CUDA_ERROR();

                    while ((info & 3) != KDNode::LEAF){
                        logger.info << "Tracing " << node << " with info " << (int)info << logger.end;
                        
                        float splitValue;
                        hipMemcpy(&splitValue, nodes->GetSplitPositionData() + node, sizeof(float), hipMemcpyDeviceToHost);
                        CHECK_FOR_CUDA_ERROR();

                        int2 children;
                        hipMemcpy(&children, nodes->GetChildrenData() + node, sizeof(int2), hipMemcpyDeviceToHost);
                        CHECK_FOR_CUDA_ERROR();
                        
                        TraceNode(ori, dir, info & 3, splitValue, children.x, children.y, tHit.x,
                                  node, tNext);

                        //logger.info << "tNext " << tNext << logger.end;

                        hipMemcpy(&info, nodes->GetInfoData() + node, sizeof(char), hipMemcpyDeviceToHost);
                    }

                    logger.info << "Found leaf: " << node << "\n" << logger.end;
                    
                    tHit.x = tNext;
                    
                    int primIndex;
                    hipMemcpy(&primIndex, nodes->GetPrimitiveIndexData() + node, sizeof(int), hipMemcpyDeviceToHost);
                    CHECK_FOR_CUDA_ERROR();
                    int primHit = -1;
                    KDNode::bitmap triangles;
                    hipMemcpy(&triangles, nodes->GetPrimitiveBitmapData() + node, sizeof(KDNode::bitmap), hipMemcpyDeviceToHost);
                    while (triangles){
                        int i = ffs(triangles) - 1;

                        //logger.info << "Testing indice " << primInfo.x << " + " << i << " = " << primInfo.x + i << logger.end;

                        int prim;
                        hipMemcpy(&prim, map->GetPrimitiveIndices()->GetDeviceData() + primIndex + i, sizeof(int), hipMemcpyDeviceToHost);
                        CHECK_FOR_CUDA_ERROR();
                        
                        //logger.info << "Testing primitive " << prim << logger.end;

                        if (intersectionAlgorithm == WOOP){
                            float4 *woop0, *woop1, *woop2;
                            geom->GetWoopValues(&woop0, &woop1, &woop2);

                            IRayTracer::Woop(woop0, woop1, woop2, prim,
                                             ori, dir, primHit, tHit);

                        }else{
                            IRayTracer::MoellerTrumbore(geom->GetP0Data(), geom->GetP1Data(), geom->GetP2Data(), prim,
                                                        ori, dir, primHit, tHit);
                        }
                        
                        triangles -= KDNode::bitmap(1)<<i;
                    }
                    
                    //logger.info << "\n" << logger.end;
                    
                    if (primHit != -1){
                        float4 n0, n1, n2;
                        hipMemcpy(&n0, geom->GetNormal0Data() + primHit, sizeof(float4), hipMemcpyDeviceToHost);
                        hipMemcpy(&n1, geom->GetNormal1Data() + primHit, sizeof(float4), hipMemcpyDeviceToHost);
                        hipMemcpy(&n2, geom->GetNormal2Data() + primHit, sizeof(float4), hipMemcpyDeviceToHost);
                        CHECK_FOR_CUDA_ERROR();

                        uchar4 c0;
                        hipMemcpy(&c0, geom->GetColor0Data() + primHit, sizeof(uchar4), hipMemcpyDeviceToHost);                        

                        logger.info << "Prim color: " << Convert::ToString(c0) << logger.end;

                        float4 newColor = Lighting(tHit, ori, dir, 
                                                   n0, n1, n2,
                                                   c0);

                        logger.info << "New color: " << Convert::ToString(newColor) << logger.end;
                        
                        color = BlendColor(color, newColor);

                        logger.info << "Color: " << Convert::ToString(color) << "\n" << logger.end;

                        tHit.x = 0.0f;
                    }

                } while(tHit.x < fInfinity && color.w < 0.97f);

                logger.info << "Final color: " << Convert::ToString(color) << logger.end;
                //exit(0);
            }

        }
    }
}
