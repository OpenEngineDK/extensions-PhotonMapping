#include "hip/hip_runtime.h"
// Variables used when segmenting the upper nodes
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/Segments.h>

#include <Meta/CUDA.h>

#include <sstream>

namespace OpenEngine {
    namespace Utils {
        namespace CUDA {
            
            Segments::Segments()
                : maxSize(0){}

            Segments::Segments(int i)
                : maxSize(i){
                
                nodeIDs = new CUDADataBlock<1, int>(i);
                primitiveInfo = new CUDADataBlock<1, int2>(i);
                aabbMin = new CUDADataBlock<1, point>(i);
                aabbMax = new CUDADataBlock<1, point>(i);
                prefixSum = new CUDADataBlock<1, int>(i);
            }

            void Segments::Resize(int i){
                nodeIDs->Resize(i);
                primitiveInfo->Resize(i);
                aabbMin->Resize(i);
                aabbMax->Resize(i);
                prefixSum->Resize(i);
                
                maxSize = i;
                size = i;
            }

            std::string Segments::ToString(int i){
                std::ostringstream out;

                if (i >= size){
                    out << "No " << i << "'th segment\n";
                }else{
                
                    int h_nodeID;
                    hipMemcpy(&h_nodeID, nodeIDs->GetDeviceData() + i, sizeof(int), hipMemcpyDeviceToHost);
                    CHECK_FOR_CUDA_ERROR();
                    out << "Segment " << i << " belongs to node " << h_nodeID << "\n";

                    int2 primInfo;
                    hipMemcpy(&primInfo, primitiveInfo->GetDeviceData() + i, sizeof(int2), hipMemcpyDeviceToHost);
                    CHECK_FOR_CUDA_ERROR();
                    out << "Ranges over " << primInfo.y << " primitives from " << primInfo.x << "\n";
                }

                return out.str();
            }

        }
    }
}
