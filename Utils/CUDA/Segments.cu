#include "hip/hip_runtime.h"
// Variables used when segmenting the upper nodes
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/Segments.h>

#include <Meta/CUDA.h>

namespace OpenEngine {
    namespace Utils {
        namespace CUDA {
            
            Segments::Segments()
                : maxSize(0), size(0) {}

            Segments::Segments(int i)
                : maxSize(i), size(i){
                hipMalloc(&nodeIDs, maxSize * sizeof(int));
                hipMalloc(&photonIndices, maxSize * sizeof(int));
                hipMalloc(&photonRanges, maxSize * sizeof(int));

                hipMalloc(&aabbMin, maxSize * sizeof(point));
                hipMalloc(&aabbMax, maxSize * sizeof(point));
                
                hipMalloc(&prefixSum, maxSize * sizeof(int));
            }

            void Segments::Resize(int i){
                int copySize = min(i, size);
                
                int* tempInt;
                hipMalloc(&tempInt, i * sizeof(int));
                hipMemcpy(tempInt, nodeIDs, copySize * sizeof(int), hipMemcpyDeviceToDevice);
                hipFree(nodeIDs);
                nodeIDs = tempInt;
                CHECK_FOR_CUDA_ERROR();

                hipMalloc(&tempInt, i * sizeof(int));
                hipMemcpy(tempInt, photonIndices, copySize * sizeof(int), hipMemcpyDeviceToDevice);
                hipFree(photonIndices);
                photonIndices = tempInt;
                CHECK_FOR_CUDA_ERROR();

                hipMalloc(&tempInt, i * sizeof(int));
                hipMemcpy(tempInt, photonRanges, copySize * sizeof(int), hipMemcpyDeviceToDevice);
                hipFree(photonRanges);
                photonRanges = tempInt;
                CHECK_FOR_CUDA_ERROR();

                point *tempPoint;
                hipMalloc(&tempPoint, i * sizeof(point));
                hipMemcpy(tempPoint, aabbMin, copySize * sizeof(point), hipMemcpyDeviceToDevice);
                hipFree(aabbMin);
                aabbMin = tempPoint;
                CHECK_FOR_CUDA_ERROR();

                hipMalloc(&tempPoint, i * sizeof(point));
                hipMemcpy(tempPoint, aabbMax, copySize * sizeof(point), hipMemcpyDeviceToDevice);
                hipFree(aabbMax);
                aabbMax = tempPoint;
                CHECK_FOR_CUDA_ERROR();

                hipMalloc(&tempInt, i * sizeof(int));
                hipMemcpy(tempInt, prefixSum, copySize * sizeof(int), hipMemcpyDeviceToDevice);
                hipFree(prefixSum);
                prefixSum = tempInt;
                CHECK_FOR_CUDA_ERROR();
                
                maxSize = i;
                size = copySize;
            }

        }
    }
}
