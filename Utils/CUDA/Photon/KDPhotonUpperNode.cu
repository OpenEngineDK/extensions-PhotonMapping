#include "hip/hip_runtime.h"
// KD tree upper node for photons
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/Photon/KDPhotonUpperNode.h>
#include <sstream>
#include <Logging/Logger.h>

namespace OpenEngine {
    namespace Utils {
        namespace CUDA {
            namespace Photon {

                void KDPhotonUpperNode::Init(unsigned int size){
                    maxSize = size;
                    hipMalloc(&info, maxSize * sizeof(char));
                    hipMalloc(&splitPos, maxSize * sizeof(float));
                    hipMalloc(&aabbMin, maxSize * sizeof(float3));
                    hipMalloc(&aabbMax, maxSize * sizeof(float3));
                    hipMalloc(&startIndex, maxSize * sizeof(unsigned int));
                    hipMalloc(&range, maxSize * sizeof(unsigned int));
                    hipMalloc(&child, maxSize * sizeof(unsigned int));
                    CHECK_FOR_CUDA_ERROR();
                }
                
                std::string KDPhotonUpperNode::ToString(unsigned int i){
                    bool isLeaf = false;
                    std::ostringstream out;
                    
                    out << "Upper node " << i << ":\n";
                    char h_info;
                    hipMemcpy(&h_info, info + i, sizeof(char), hipMemcpyDeviceToHost);
                    
                    float h_pos;
                    hipMemcpy(&h_pos, splitPos + i, sizeof(float), hipMemcpyDeviceToHost);
                    switch(h_info){
                    case X:
                        out << "Splits along the X plane at pos " << h_pos << "\n";
                        break;
                    case Y:
                        out << "Splits along the Y plane at pos " << h_pos << "\n";
                        break;
                    case Z:
                        out << "Splits along the Z plane at pos " << h_pos << "\n";
                            break;
                    case LEAF:
                        isLeaf = true;
                        out << "Is a leaf\n";
                        break;
                    }
                    
                    float3 h_aabbmin, h_aabbmax;
                    hipMemcpy(&h_aabbmin, aabbMin + i, sizeof(float3), hipMemcpyDeviceToHost);
                    hipMemcpy(&h_aabbmax, aabbMax + i, sizeof(float3), hipMemcpyDeviceToHost);
                    out << "Axis aligned bounding box: " << Utils::CUDA::Convert::ToString(h_aabbmin);
                    out << " -> " << Utils::CUDA::Convert::ToString(h_aabbmax) << "\n";
                    
                    if (i != 0){
                        unsigned int p;
                        hipMemcpy(&p, parent + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
                            out << "Has parent " << p << " and ";
                    }
                    
                    unsigned int h_child;
                    hipMemcpy(&h_child, child + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
                    if (!isLeaf){
                        out << "has leftchild " << h_child << "\n";
                    }else{
                        out << "points to lowernode " << h_child << "\n";
                    }
                    
                    return out.str();
                }

                void KDPhotonUpperNode::CheckBoundingBox(unsigned int i, photon photons){
                    unsigned int photonIndex;
                    hipMemcpy(&photonIndex, startIndex + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
                    unsigned int photonRange;
                    hipMemcpy(&photonRange, range + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
                    
                    float3 positions[photonRange];
                    hipMemcpy(positions, photons.pos + photonIndex, photonRange * sizeof(float3), hipMemcpyDeviceToHost);
                    float3 hostMax = positions[0];
                    float3 hostMin = positions[0];
                    for (unsigned int p = 1; p < photonRange; ++p){
                        hostMax = make_float3(max(hostMax.x, positions[p].x),
                                              max(hostMax.y, positions[p].y),
                                              max(hostMax.z, positions[p].z));
                        hostMin = make_float3(min(hostMin.x, positions[p].x),
                                              min(hostMin.y, positions[p].y),
                                              min(hostMin.z, positions[p].z));
                    }
                    
                    float3 gpuMax;
                    float3 gpuMin;
                    hipMemcpy(&gpuMax, aabbMax + i, sizeof(float3), hipMemcpyDeviceToHost);
                    hipMemcpy(&gpuMin, aabbMin + i, sizeof(float3), hipMemcpyDeviceToHost);
    
                    if (hostMax.x != gpuMax.x || hostMax.y != gpuMax.y || hostMax.z != gpuMax.z){
                        logger.info << "CPU max " << Utils::CUDA::Convert::ToString(hostMax);
                        logger.info << " != GPU max " << Utils::CUDA::Convert::ToString(gpuMax) << logger.end;
                    }
                    
                    if (hostMin.x != gpuMin.x || hostMin.y != gpuMin.y || hostMin.z != gpuMin.z){
                        logger.info << "CPU min " << Utils::CUDA::Convert::ToString(hostMin);
                        logger.info << " != GPU min " << Utils::CUDA::Convert::ToString(gpuMin) << logger.end;
                    }

                }

            }
        }
    }
}
