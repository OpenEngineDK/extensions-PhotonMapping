#include "hip/hip_runtime.h"
// Temp variables for holding temp child node values.
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Utils/CUDA/NodeChildren.h>

namespace OpenEngine {
    namespace Utils {
        namespace CUDA {
            
            NodeChildren::NodeChildren(int size)
                : size(size) {
                hipMalloc(&photonInfo, size * sizeof(int2));
                hipMalloc(&parents, size * sizeof(int));
            }

            void NodeChildren::Resize(int size){
                hipFree(photonInfo);
                hipMalloc(&photonInfo, size * sizeof(int2));
                hipFree(parents);
                hipMalloc(&parents, size * sizeof(int));
            }

        }
    }
}
