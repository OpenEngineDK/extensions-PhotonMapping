#include "hip/hip_runtime.h"
// KD tree upper node for photons
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Scene/PhotonUpperNode.h>
#include <Resources/IDataBlock.h>
#include <Utils/CUDA/Kernels/UpperNodeMapToGL.hcu>
#include <Utils/CUDA/Utils.h>

#include <Logging/Logger.h>

#include <sstream>

using namespace OpenEngine::Resources;
using namespace OpenEngine::Utils::CUDA::Kernels;

namespace OpenEngine {
    namespace Scene {

        PhotonUpperNode::PhotonUpperNode()
            : KDNode() {}

        PhotonUpperNode::PhotonUpperNode(int size)
            : KDNode(size) {

            logger.info << "Photon upper node inital max: " << size<< logger.end;

            cudaSafeMalloc(&parents, maxSize * sizeof(int));

            CHECK_FOR_CUDA_ERROR();
        }

        void PhotonUpperNode::Resize(int i){
            KDNode::Resize(i);
            
            unsigned int copySize = this->size;
            
            int *tempInt;

            cudaSafeMalloc(&tempInt, i * sizeof(int));
            hipMemcpy(tempInt, parents, copySize * sizeof(int), hipMemcpyDeviceToDevice);
            hipFree(parents);
            parents = tempInt;
            CHECK_FOR_CUDA_ERROR();

            maxSize = i;
            size = copySize;
        }

        void PhotonUpperNode::MapToDataBlocks(Resources::IDataBlock* position,
                                                Resources::IDataBlock* colors){
#ifdef DEBUG
            if (position->GetID() == 0 && colors->GetID() == 0)
                return;
#endif
            
            hipGraphicsResource *pResource, *cResource;
            hipGraphicsGLRegisterBuffer(&pResource, position->GetID(), cudaGraphicsMapFlagsWriteDiscard);
            hipGraphicsGLRegisterBuffer(&cResource, colors->GetID(), cudaGraphicsMapFlagsWriteDiscard);
            CHECK_FOR_CUDA_ERROR();
            
            hipGraphicsMapResources(1, &pResource, 0);
            hipGraphicsMapResources(1, &cResource, 0);
            CHECK_FOR_CUDA_ERROR();

            float3* posv;
            size_t bytes;
            hipGraphicsResourceGetMappedPointer((void**)&posv, &bytes,
                                                 pResource);
            float3* colv;
            hipGraphicsResourceGetMappedPointer((void**)&colv, &bytes,
                                                 cResource);
            CHECK_FOR_CUDA_ERROR();

            unsigned int s = min(size, position->GetSize());

            unsigned int blocks, threads;
            Calc1DKernelDimensions(s, blocks, threads);
            //logger.info << "blocks " << blocks << ", threads " << threads << logger.end;
            UpperNodeMapToGL<<<blocks, threads/2>>>(aabbMin, aabbMax, splitPos, info, posv, colv, s);
            CHECK_FOR_CUDA_ERROR();

            hipGraphicsUnmapResources(1, &pResource, 0);
            hipGraphicsUnmapResources(1, &cResource, 0);
            CHECK_FOR_CUDA_ERROR();
            
            hipGraphicsUnregisterResource(pResource);
            hipGraphicsUnregisterResource(cResource);
            CHECK_FOR_CUDA_ERROR();
        }
                
        std::string PhotonUpperNode::ToString(unsigned int i){
            bool isLeaf = false;
            std::ostringstream out;
                    
            out << "Upper node " << i << ":\n";
            char h_info;
            hipMemcpy(&h_info, info + i, sizeof(char), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
                    
            float h_pos;
            hipMemcpy(&h_pos, splitPos + i, sizeof(float), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            switch(h_info){
            case X:
                out << "Splits along the X plane at pos " << h_pos << "\n";
                break;
            case Y:
                out << "Splits along the Y plane at pos " << h_pos << "\n";
                break;
            case Z:
                out << "Splits along the Z plane at pos " << h_pos << "\n";
                break;
            case LEAF:
                isLeaf = true;
                out << "Is a leaf\n";
                break;
            }

            int2 info;
            hipMemcpy(&info, photonInfo + i, sizeof(int2), hipMemcpyDeviceToHost);
            out << "Index " << info.x << " and range " << info.y << "\n";
                   
            if (!isLeaf){
                point h_aabbmin, h_aabbmax;
                hipMemcpy(&h_aabbmin, aabbMin + i, sizeof(point), hipMemcpyDeviceToHost);
                hipMemcpy(&h_aabbmax, aabbMax + i, sizeof(point), hipMemcpyDeviceToHost);
                CHECK_FOR_CUDA_ERROR();
                out << "Axis aligned bounding box: " << Utils::CUDA::Convert::ToString(h_aabbmin);
                out << " -> " << Utils::CUDA::Convert::ToString(h_aabbmax) << "\n";
            }

            /*                    
            if (i != 0){
                int p;
                hipMemcpy(&p, parents + i, sizeof(int), hipMemcpyDeviceToHost);
                out << "Has parent " << p << " and ";
                }*/
                    
            int h_left, h_right;
            hipMemcpy(&h_left, left + i, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&h_right, right + i, sizeof(int), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            if (!isLeaf){
                out << "Has children " << h_left << " and " << h_right << "\n";
            }else{
                out << "points to lowernode " << h_left << "\n";
            }
                    
            return out.str();
        }

        std::string PhotonUpperNode::PhotonsToString(unsigned int i, 
                                                     PhotonNode photons){
            std::ostringstream out;

            int2 info;
            hipMemcpy(&info, photonInfo+i, sizeof(int2), hipMemcpyDeviceToHost);

            //logger.info << "Index " << info.x << ", range " << info.y << logger.end;
            
            /*
            unsigned int index,size;
            hipMemcpy(&index, photonIndex+i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            hipMemcpy(&size, range+i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            */

            point pos[info.y];
            hipMemcpy(&pos, photons.pos+info.x, 
                       info.y*sizeof(point), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            
            out << Utils::CUDA::Convert::ToString(pos[0]);
            for (int i = 1; i < info.y; ++i){
                out << "\n" << Utils::CUDA::Convert::ToString(pos[i]);
            }

            return out.str();
        }

        /*
        void PhotonUpperNode::CheckBoundingBox(unsigned int i, PhotonNode photons){
            unsigned int photonStart;
            hipMemcpy(&photonStart, photonIndex + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            unsigned int photonRange;
            hipMemcpy(&photonRange, range + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
                    
            point positions[photonRange];
            hipMemcpy(positions, photons.pos + photonStart, photonRange * sizeof(point), hipMemcpyDeviceToHost);
            point hostMax = positions[0];
            point hostMin = positions[0];
            for (unsigned int p = 1; p < photonRange; ++p){
                hostMax = make_float3(max(hostMax.x, positions[p].x),
                                      max(hostMax.y, positions[p].y),
                                      max(hostMax.z, positions[p].z));
                hostMin = make_float3(min(hostMin.x, positions[p].x),
                                      min(hostMin.y, positions[p].y),
                                      min(hostMin.z, positions[p].z));
            }
                    
            point gpuMax;
            point gpuMin;
            hipMemcpy(&gpuMax, aabbMax + i, sizeof(point), hipMemcpyDeviceToHost);
            hipMemcpy(&gpuMin, aabbMin + i, sizeof(point), hipMemcpyDeviceToHost);
    
            if (hostMax.x != gpuMax.x || hostMax.y != gpuMax.y || hostMax.z != gpuMax.z){
                logger.info << "CPU max " << Utils::CUDA::Convert::ToString(hostMax);
                logger.info << " != GPU max " << Utils::CUDA::Convert::ToString(gpuMax) << logger.end;
            }
                    
            if (hostMin.x != gpuMin.x || hostMin.y != gpuMin.y || hostMin.z != gpuMin.z){
                logger.info << "CPU min " << Utils::CUDA::Convert::ToString(hostMin);
                logger.info << " != GPU min " << Utils::CUDA::Convert::ToString(gpuMin) << logger.end;
            }

        }
        */
    }
}
