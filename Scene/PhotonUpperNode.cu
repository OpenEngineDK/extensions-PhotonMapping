#include "hip/hip_runtime.h"
// KD tree upper node for photons
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Scene/PhotonUpperNode.h>
#include <Resources/IDataBlock.h>
#include <Utils/CUDA/Kernels/UpperNodeMapToGL.hcu>
#include <Utils/CUDA/Utils.h>

#include <Logging/Logger.h>

#include <sstream>

using namespace OpenEngine::Resources;
using namespace OpenEngine::Utils::CUDA::Kernels;

namespace OpenEngine {
    namespace Scene {

        PhotonUpperNode::PhotonUpperNode()
            : KDNode() {}

        PhotonUpperNode::PhotonUpperNode(int size)
            : KDNode(size) {

            hipMalloc(&range, maxSize * sizeof(int));
            hipMalloc(&parent, maxSize * sizeof(int));
            hipMalloc(&left, maxSize * sizeof(int));
            hipMalloc(&right, maxSize * sizeof(int));

            CHECK_FOR_CUDA_ERROR();
        }

        void PhotonUpperNode::Resize(int i){
            KDNode::Resize(i);
            
            unsigned int copySize = this->size;
            
            int *tempInt;

            hipMalloc(&tempInt, i * sizeof(int));
            hipMemcpy(tempInt, range, copySize * sizeof(int), hipMemcpyDeviceToDevice);
            hipFree(range);
            range = tempInt;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempInt, i * sizeof(int));
            hipMemcpy(tempInt, parent, copySize * sizeof(int), hipMemcpyDeviceToDevice);
            hipFree(parent);
            parent = tempInt;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempInt, i * sizeof(int));
            hipMemcpy(tempInt, left, copySize * sizeof(int), hipMemcpyDeviceToDevice);
            hipFree(left);
            left = tempInt;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempInt, i * sizeof(int));
            hipMemcpy(tempInt, right, copySize * sizeof(int), hipMemcpyDeviceToDevice);
            hipFree(right);
            right = tempInt;
            CHECK_FOR_CUDA_ERROR();

            maxSize = i;
            size = copySize;
        }

        /*
        void PhotonUpperNode::MapToDataBlocks(Resources::IDataBlock* position,
                                                Resources::IDataBlock* colors){
#ifdef DEBUG
            if (position->GetID() == 0 && colors->GetID() == 0)
                return;
#endif
            
            hipGraphicsResource *pResource, *cResource;
            hipGraphicsGLRegisterBuffer(&pResource, position->GetID(), cudaGraphicsMapFlagsWriteDiscard);
            hipGraphicsGLRegisterBuffer(&cResource, colors->GetID(), cudaGraphicsMapFlagsWriteDiscard);
            CHECK_FOR_CUDA_ERROR();
            
            hipGraphicsMapResources(1, &pResource, 0);
            hipGraphicsMapResources(1, &cResource, 0);
            CHECK_FOR_CUDA_ERROR();

            float3* posv;
            size_t bytes;
            hipGraphicsResourceGetMappedPointer((void**)&posv, &bytes,
                                                 pResource);
            float3* colv;
            hipGraphicsResourceGetMappedPointer((void**)&colv, &bytes,
                                                 cResource);
            CHECK_FOR_CUDA_ERROR();

            unsigned int s = min(size, position->GetSize());

            unsigned int blocks, threads;
            Calc1DKernelDimensions(s, blocks, threads);
            UpperNodeMapToGL<<<blocks, threads/2>>>(*this, posv, colv, s);
            CHECK_FOR_CUDA_ERROR();

            hipGraphicsUnmapResources(1, &pResource, 0);
            hipGraphicsUnmapResources(1, &cResource, 0);
            CHECK_FOR_CUDA_ERROR();
            
            hipGraphicsUnregisterResource(pResource);
            hipGraphicsUnregisterResource(cResource);
            CHECK_FOR_CUDA_ERROR();
        }
                
        std::string PhotonUpperNode::ToString(unsigned int i){
            bool isLeaf = false;
            std::ostringstream out;
                    
            out << "Upper node " << i << ":\n";
            char h_info;
            hipMemcpy(&h_info, info + i, sizeof(char), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
                    
            float h_pos;
            hipMemcpy(&h_pos, splitPos + i, sizeof(float), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            switch(h_info){
            case X:
                out << "Splits along the X plane at pos " << h_pos << "\n";
                break;
            case Y:
                out << "Splits along the Y plane at pos " << h_pos << "\n";
                break;
            case Z:
                out << "Splits along the Z plane at pos " << h_pos << "\n";
                break;
            case LEAF:
                isLeaf = true;
                out << "Is a leaf\n";
                break;
            }

            unsigned int h_index, h_range;
            hipMemcpy(&h_index, photonIndex + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            hipMemcpy(&h_range, range + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            out << "Index " << h_index << " and range " << h_range << "\n";
                    
            point h_aabbmin, h_aabbmax;
            hipMemcpy(&h_aabbmin, aabbMin + i, sizeof(point), hipMemcpyDeviceToHost);
            hipMemcpy(&h_aabbmax, aabbMax + i, sizeof(point), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            out << "Axis aligned bounding box: " << Utils::CUDA::Convert::ToString(h_aabbmin);
            out << " -> " << Utils::CUDA::Convert::ToString(h_aabbmax) << "\n";
                    
            if (i != 0){
                unsigned int p;
                hipMemcpy(&p, parent + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
                out << "Has parent " << p << " and ";
            }
                    
            unsigned int h_child;
            hipMemcpy(&h_child, child + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            if (!isLeaf){
                out << "has leftchild " << h_child << "\n";
            }else{
                out << "points to lowernode " << h_child << "\n";
            }
                    
            return out.str();
        }

        std::string PhotonUpperNode::PhotonsToString(unsigned int i, 
                                                       PhotonNode photons){
            std::ostringstream out;
            
            unsigned int index,size;
            hipMemcpy(&index, photonIndex+i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            hipMemcpy(&size, range+i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            
            point pos[size];
            hipMemcpy(&pos, photons.pos+index, 
                       size*sizeof(point), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            
            out << Utils::CUDA::Convert::ToString(pos[0]);
            for (unsigned int i = 1; i < size; ++i){
                out << "\n" << Utils::CUDA::Convert::ToString(pos[i]);
            }

            return out.str();
        }

        void PhotonUpperNode::CheckBoundingBox(unsigned int i, PhotonNode photons){
            unsigned int photonStart;
            hipMemcpy(&photonStart, photonIndex + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            unsigned int photonRange;
            hipMemcpy(&photonRange, range + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
                    
            point positions[photonRange];
            hipMemcpy(positions, photons.pos + photonStart, photonRange * sizeof(point), hipMemcpyDeviceToHost);
            point hostMax = positions[0];
            point hostMin = positions[0];
            for (unsigned int p = 1; p < photonRange; ++p){
                hostMax = make_float3(max(hostMax.x, positions[p].x),
                                      max(hostMax.y, positions[p].y),
                                      max(hostMax.z, positions[p].z));
                hostMin = make_float3(min(hostMin.x, positions[p].x),
                                      min(hostMin.y, positions[p].y),
                                      min(hostMin.z, positions[p].z));
            }
                    
            point gpuMax;
            point gpuMin;
            hipMemcpy(&gpuMax, aabbMax + i, sizeof(point), hipMemcpyDeviceToHost);
            hipMemcpy(&gpuMin, aabbMin + i, sizeof(point), hipMemcpyDeviceToHost);
    
            if (hostMax.x != gpuMax.x || hostMax.y != gpuMax.y || hostMax.z != gpuMax.z){
                logger.info << "CPU max " << Utils::CUDA::Convert::ToString(hostMax);
                logger.info << " != GPU max " << Utils::CUDA::Convert::ToString(gpuMax) << logger.end;
            }
                    
            if (hostMin.x != gpuMin.x || hostMin.y != gpuMin.y || hostMin.z != gpuMin.z){
                logger.info << "CPU min " << Utils::CUDA::Convert::ToString(hostMin);
                logger.info << " != GPU min " << Utils::CUDA::Convert::ToString(gpuMin) << logger.end;
            }

        }
        */
    }
}
