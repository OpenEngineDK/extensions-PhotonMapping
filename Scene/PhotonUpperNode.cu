#include "hip/hip_runtime.h"
// KD tree upper node for photons
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Scene/PhotonUpperNode.h>
#include <Resources/IDataBlock.h>
#include <Utils/CUDA/Kernels/UpperNodeMapToGL.hcu>
#include <Utils/CUDA/Utils.h>

#include <Logging/Logger.h>

using namespace OpenEngine::Resources;
using namespace OpenEngine::Utils::CUDA::Kernels;

namespace OpenEngine {
    namespace Scene {

        PhotonUpperNode::PhotonUpperNode()
            : KDNode() {}

        PhotonUpperNode::PhotonUpperNode(int size)
            : KDNode(size) {

            logger.info << "Photon upper node inital max: " << size<< logger.end;

            //cudaSafeMalloc(&parents, maxSize * sizeof(int));

            CHECK_FOR_CUDA_ERROR();
        }

        void PhotonUpperNode::Resize(int i){
            KDNode::Resize(i);
            /*            
            unsigned int copySize = this->size;
            
            int *tempInt;
            cudaSafeMalloc(&tempInt, i * sizeof(int));
            hipMemcpy(tempInt, parents, copySize * sizeof(int), hipMemcpyDeviceToDevice);
            hipFree(parents);
            parents = tempInt;
            CHECK_FOR_CUDA_ERROR();

            maxSize = i;
            size = copySize;
            */
        }

        void PhotonUpperNode::MapToDataBlocks(Resources::IDataBlock* position,
                                                Resources::IDataBlock* colors){
#ifdef DEBUG
            if (position->GetID() == 0 && colors->GetID() == 0)
                return;
#endif
            
            hipGraphicsResource *pResource, *cResource;
            hipGraphicsGLRegisterBuffer(&pResource, position->GetID(), cudaGraphicsMapFlagsWriteDiscard);
            hipGraphicsGLRegisterBuffer(&cResource, colors->GetID(), cudaGraphicsMapFlagsWriteDiscard);
            CHECK_FOR_CUDA_ERROR();
            
            hipGraphicsMapResources(1, &pResource, 0);
            hipGraphicsMapResources(1, &cResource, 0);
            CHECK_FOR_CUDA_ERROR();

            float3* posv;
            size_t bytes;
            hipGraphicsResourceGetMappedPointer((void**)&posv, &bytes,
                                                 pResource);
            float3* colv;
            hipGraphicsResourceGetMappedPointer((void**)&colv, &bytes,
                                                 cResource);
            CHECK_FOR_CUDA_ERROR();

            unsigned int s = min(size, position->GetSize());

            unsigned int blocks, threads;
            Calc1DKernelDimensions(s, blocks, threads);
            //logger.info << "blocks " << blocks << ", threads " << threads << logger.end;
            UpperNodeMapToGL<<<blocks, threads/2>>>(aabbMin->GetDeviceData(), aabbMax->GetDeviceData(), splitPos->GetDeviceData(), info->GetDeviceData(), posv, colv, s);
            CHECK_FOR_CUDA_ERROR();

            hipGraphicsUnmapResources(1, &pResource, 0);
            hipGraphicsUnmapResources(1, &cResource, 0);
            CHECK_FOR_CUDA_ERROR();
            
            hipGraphicsUnregisterResource(pResource);
            hipGraphicsUnregisterResource(cResource);
            CHECK_FOR_CUDA_ERROR();
        }
                
        std::string PhotonUpperNode::PhotonsToString(unsigned int i, 
                                                     PhotonNode photons){
            std::ostringstream out;

            int2 info;
            hipMemcpy(&info, photonInfo+i, sizeof(int2), hipMemcpyDeviceToHost);

            point pos[info.y];
            hipMemcpy(&pos, photons.pos+info.x, 
                       info.y*sizeof(point), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            
            out << Utils::CUDA::Convert::ToString(pos[0]);
            for (int i = 1; i < info.y; ++i){
                out << "\n" << Utils::CUDA::Convert::ToString(pos[i]);
            }

            return out.str();
        }

    }
}
