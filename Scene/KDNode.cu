#include "hip/hip_runtime.h"
// KD tree upper node for photons
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Scene/KDNode.h>

namespace OpenEngine {
    namespace Scene {
        
        KDNode::KDNode()
            : maxSize(0), size(0) {}

        KDNode::KDNode(int i)
            : maxSize(i), size(0) {
            
            hipMalloc(&info, maxSize * sizeof(char));
            hipMalloc(&splitPos, maxSize * sizeof(float));
            hipMalloc(&aabbMin, maxSize * sizeof(point));
            hipMalloc(&aabbMax, maxSize * sizeof(point));

            hipMalloc(&photonIndex, maxSize * sizeof(int));

            CHECK_FOR_CUDA_ERROR();
        }

        void KDNode::Resize(int i){
            int copySize = min(i, size);
            
            char *tempChar;
            float *tempFloat;
            point *tempPoint;
            int *tempInt;

            hipMalloc(&tempChar, i * sizeof(char));
            hipMemcpy(tempChar, info, copySize * sizeof(char), hipMemcpyDeviceToDevice);
            hipFree(info);
            info = tempChar;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempFloat, i * sizeof(float));
            hipMemcpy(tempFloat, splitPos, copySize * sizeof(float), hipMemcpyDeviceToDevice);
            hipFree(splitPos);
            splitPos = tempFloat;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempPoint, i * sizeof(point));
            hipMemcpy(tempPoint, aabbMin, copySize * sizeof(point), hipMemcpyDeviceToDevice);
            hipFree(aabbMin);
            aabbMin = tempPoint;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempPoint, i * sizeof(point));
            hipMemcpy(tempPoint, aabbMax, copySize * sizeof(point), hipMemcpyDeviceToDevice);
            hipFree(aabbMax);
            aabbMax = tempPoint;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempInt, i * sizeof(int));
            hipMemcpy(tempInt, photonIndex, copySize * sizeof(int), hipMemcpyDeviceToDevice);
            hipFree(photonIndex);
            photonIndex = tempInt;
            CHECK_FOR_CUDA_ERROR();

            maxSize = i;
            size = copySize;
        }
        
    }
}
