#include "hip/hip_runtime.h"
// KD tree upper node for photons
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Scene/KDNode.h>
#include <Utils/CUDA/Utils.h>

namespace OpenEngine {
    namespace Scene {
        
        KDNode::KDNode()
            : maxSize(0), size(0) {}

        KDNode::KDNode(int i)
            : maxSize(i), size(0) {
            
            cudaSafeMalloc(&info, maxSize * sizeof(char));
            cudaSafeMalloc(&splitPos, maxSize * sizeof(float));
            cudaSafeMalloc(&aabbMin, maxSize * sizeof(point));
            cudaSafeMalloc(&aabbMax, maxSize * sizeof(point));

            cudaSafeMalloc(&photonInfo, maxSize * sizeof(int2));

            CHECK_FOR_CUDA_ERROR();
        }

        void KDNode::Resize(int i){
            int copySize = min(i, size);
            
            char *tempChar;
            float *tempFloat;
            point *tempPoint;
            int2 *tempInt2;

            hipMalloc(&tempChar, i * sizeof(char));
            hipMemcpy(tempChar, info, copySize * sizeof(char), hipMemcpyDeviceToDevice);
            hipFree(info);
            info = tempChar;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempFloat, i * sizeof(float));
            hipMemcpy(tempFloat, splitPos, copySize * sizeof(float), hipMemcpyDeviceToDevice);
            hipFree(splitPos);
            splitPos = tempFloat;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempPoint, i * sizeof(point));
            hipMemcpy(tempPoint, aabbMin, copySize * sizeof(point), hipMemcpyDeviceToDevice);
            hipFree(aabbMin);
            aabbMin = tempPoint;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempPoint, i * sizeof(point));
            hipMemcpy(tempPoint, aabbMax, copySize * sizeof(point), hipMemcpyDeviceToDevice);
            hipFree(aabbMax);
            aabbMax = tempPoint;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempInt2, i * sizeof(int2));
            hipMemcpy(tempInt2, photonInfo, copySize * sizeof(int2), hipMemcpyDeviceToDevice);
            hipFree(photonInfo);
            photonInfo = tempInt2;
            CHECK_FOR_CUDA_ERROR();

            maxSize = i;
            size = copySize;
        }
        
    }
}
