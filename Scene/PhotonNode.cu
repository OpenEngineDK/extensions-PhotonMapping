#include "hip/hip_runtime.h"
// Photon class for CUDA
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Scene/PhotonNode.h>

#include <Math/RandomGenerator.h>
#include <Resources/IDataBlock.h>
#include <Utils/CUDA/Convert.h>

#include <sstream>

using namespace OpenEngine::Resources;

namespace OpenEngine {
    namespace Scene {

        void PhotonNode::CreateRandomData(){
            point hat[maxSize];
            Math::RandomGenerator rand;
            rand.SeedWithTime();
            for (unsigned int i = 0; i < size; ++i)
                hat[i] = make_float3(rand.UniformFloat(0.0f, 10.0f),
                                     rand.UniformFloat(0.0f, 10.0f),
                                     rand.UniformFloat(0.0f, 10.0f));
            
            hipMemcpy(pos, hat, size * sizeof(point), hipMemcpyHostToDevice);
            size = maxSize;
        }

        std::string PhotonNode::PositionToString(unsigned int begin, unsigned int range){
            std::ostringstream out;
            
            point position[range];
            hipMemcpy(position, pos + begin, range * sizeof(point), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            
            out << "[ 0: " << Utils::CUDA::Convert::ToString(position[0]);
            for (unsigned int i = 1; i < range; ++i){
                out << "\n " << i << ": " << Utils::CUDA::Convert::ToString(position[i]);
            }
            out << "]";
            return out.str();
        }

        void PhotonNode::MapToDataBlocks(IDataBlock* position){
            if (position->GetID() > 0){
                hipGraphicsResource* resource;
                hipGraphicsGLRegisterBuffer(&resource, position->GetID(), cudaGraphicsMapFlagsWriteDiscard);
                CHECK_FOR_CUDA_ERROR();
                
                hipGraphicsMapResources(1, &resource, 0);
                CHECK_FOR_CUDA_ERROR();
            
                point* verts;
                size_t bytes;
                hipGraphicsResourceGetMappedPointer((void**)&verts, &bytes,
                                                     resource);
                CHECK_FOR_CUDA_ERROR();
                
                hipMemcpy(verts, pos, bytes, hipMemcpyDeviceToDevice);
                CHECK_FOR_CUDA_ERROR();
                
                hipGraphicsUnmapResources(1, &resource, 0);
                CHECK_FOR_CUDA_ERROR();
                
                hipGraphicsUnregisterResource(resource);
                CHECK_FOR_CUDA_ERROR();
            }else if (position->GetVoidDataPtr() != NULL){
                hipMemcpy(position->GetVoidDataPtr(), pos, size * sizeof(point), hipMemcpyDeviceToHost);
            }
        }
        
    }
}
