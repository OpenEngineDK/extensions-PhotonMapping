#include "hip/hip_runtime.h"
// KD tree upper node for photons
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Scene/KDPhotonUpperNode.h>
#include <Resources/IDataBlock.h>
#include <Utils/CUDA/Kernels/UpperNodeMapToGL.hcu>

#include <Logging/Logger.h>

#include <sstream>

using namespace OpenEngine::Resources;
using namespace OpenEngine::Utils::CUDA::Kernels;

namespace OpenEngine {
    namespace Scene {

        KDPhotonUpperNode::KDPhotonUpperNode()
            : maxSize(0), size(0){}

        KDPhotonUpperNode::KDPhotonUpperNode(unsigned int size)
            : maxSize(size), size(0) {

            hipMalloc(&info, maxSize * sizeof(char));
            hipMalloc(&splitPos, maxSize * sizeof(float));
            hipMalloc(&aabbMin, maxSize * sizeof(point));
            hipMalloc(&aabbMax, maxSize * sizeof(point));

            hipMalloc(&photonIndex, maxSize * sizeof(unsigned int));
            hipMalloc(&range, maxSize * sizeof(unsigned int));
            hipMalloc(&parent, maxSize * sizeof(unsigned int));
            hipMalloc(&child, maxSize * sizeof(unsigned int));

            CHECK_FOR_CUDA_ERROR();
        }

        void KDPhotonUpperNode::Resize(unsigned int i){
            unsigned int copySize = i < size ? i : size;
            
            char *tempChar;
            float *tempFloat;
            point *tempPoint;
            unsigned int *tempUint;

            hipMalloc(&tempChar, i * sizeof(char));
            hipMemcpy(tempChar, info, copySize * sizeof(char), hipMemcpyDeviceToDevice);
            hipFree(info);
            info = tempChar;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempFloat, i * sizeof(float));
            hipMemcpy(tempFloat, splitPos, copySize * sizeof(float), hipMemcpyDeviceToDevice);
            hipFree(splitPos);
            splitPos = tempFloat;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempPoint, i * sizeof(point));
            hipMemcpy(tempPoint, aabbMin, copySize * sizeof(point), hipMemcpyDeviceToDevice);
            hipFree(aabbMin);
            aabbMin = tempPoint;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempPoint, i * sizeof(point));
            hipMemcpy(tempPoint, aabbMax, copySize * sizeof(point), hipMemcpyDeviceToDevice);
            hipFree(aabbMax);
            aabbMax = tempPoint;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempUint, i * sizeof(unsigned int));
            hipMemcpy(tempUint, photonIndex, copySize * sizeof(unsigned int), hipMemcpyDeviceToDevice);
            hipFree(photonIndex);
            photonIndex = tempUint;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempUint, i * sizeof(unsigned int));
            hipMemcpy(tempUint, range, copySize * sizeof(unsigned int), hipMemcpyDeviceToDevice);
            hipFree(range);
            range = tempUint;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempUint, i * sizeof(unsigned int));
            hipMemcpy(tempUint, parent, copySize * sizeof(unsigned int), hipMemcpyDeviceToDevice);
            hipFree(parent);
            parent = tempUint;
            CHECK_FOR_CUDA_ERROR();

            hipMalloc(&tempUint, i * sizeof(unsigned int));
            hipMemcpy(tempUint, child, copySize * sizeof(unsigned int), hipMemcpyDeviceToDevice);
            hipFree(child);
            child = tempUint;
            CHECK_FOR_CUDA_ERROR();

            maxSize = i;
        }

        void KDPhotonUpperNode::MapToDataBlocks(Resources::IDataBlock* position,
                                                Resources::IDataBlock* colors){
#ifdef DEBUG
            if (position->GetID() == 0 && colors->GetID() == 0)
                return;
#endif
            
            hipGraphicsResource *pResource, *cResource;
            hipGraphicsGLRegisterBuffer(&pResource, position->GetID(), cudaGraphicsMapFlagsWriteDiscard);
            hipGraphicsGLRegisterBuffer(&cResource, colors->GetID(), cudaGraphicsMapFlagsWriteDiscard);
            CHECK_FOR_CUDA_ERROR();
            
            hipGraphicsMapResources(1, &pResource, 0);
            hipGraphicsMapResources(1, &cResource, 0);
            CHECK_FOR_CUDA_ERROR();

            float3* posv;
            size_t bytes;
            hipGraphicsResourceGetMappedPointer((void**)&posv, &bytes,
                                                 pResource);
            float3* colv;
            hipGraphicsResourceGetMappedPointer((void**)&colv, &bytes,
                                                 cResource);
            CHECK_FOR_CUDA_ERROR();

            unsigned int s = min(size, position->GetSize());
            
            UpperNodeMapToGL<<<64, 128>>>(*this, posv, colv, s);
            CHECK_FOR_CUDA_ERROR();

            hipGraphicsUnmapResources(1, &pResource, 0);
            hipGraphicsUnmapResources(1, &cResource, 0);
            CHECK_FOR_CUDA_ERROR();
            
            hipGraphicsUnregisterResource(pResource);
            hipGraphicsUnregisterResource(cResource);
            CHECK_FOR_CUDA_ERROR();
        }
                
        std::string KDPhotonUpperNode::ToString(unsigned int i){
            bool isLeaf = false;
            std::ostringstream out;
                    
            out << "Upper node " << i << ":\n";
            char h_info;
            hipMemcpy(&h_info, info + i, sizeof(char), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
                    
            float h_pos;
            hipMemcpy(&h_pos, splitPos + i, sizeof(float), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            switch(h_info){
            case X:
                out << "Splits along the X plane at pos " << h_pos << "\n";
                break;
            case Y:
                out << "Splits along the Y plane at pos " << h_pos << "\n";
                break;
            case Z:
                out << "Splits along the Z plane at pos " << h_pos << "\n";
                break;
            case LEAF:
                isLeaf = true;
                out << "Is a leaf\n";
                break;
            }

            unsigned int h_index, h_range;
            hipMemcpy(&h_index, photonIndex + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            hipMemcpy(&h_range, range + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            out << "Index " << h_index << " and range " << h_range << "\n";
                    
            point h_aabbmin, h_aabbmax;
            hipMemcpy(&h_aabbmin, aabbMin + i, sizeof(point), hipMemcpyDeviceToHost);
            hipMemcpy(&h_aabbmax, aabbMax + i, sizeof(point), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            out << "Axis aligned bounding box: " << Utils::CUDA::Convert::ToString(h_aabbmin);
            out << " -> " << Utils::CUDA::Convert::ToString(h_aabbmax) << "\n";
                    
            if (i != 0){
                unsigned int p;
                hipMemcpy(&p, parent + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
                out << "Has parent " << p << " and ";
            }
                    
            unsigned int h_child;
            hipMemcpy(&h_child, child + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            if (!isLeaf){
                out << "has leftchild " << h_child << "\n";
            }else{
                out << "points to lowernode " << h_child << "\n";
            }
                    
            return out.str();
        }

        std::string KDPhotonUpperNode::PhotonsToString(unsigned int i, 
                                                       PhotonNode photons){
            std::ostringstream out;
            
            unsigned int index,size;
            hipMemcpy(&index, photonIndex+i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            hipMemcpy(&size, range+i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            
            point pos[size];
            hipMemcpy(&pos, photons.pos+index, 
                       size*sizeof(point), hipMemcpyDeviceToHost);
            CHECK_FOR_CUDA_ERROR();
            
            out << Utils::CUDA::Convert::ToString(pos[0]);
            for (unsigned int i = 1; i < size; ++i){
                out << "\n" << Utils::CUDA::Convert::ToString(pos[i]);
            }

            return out.str();
        }

        void KDPhotonUpperNode::CheckBoundingBox(unsigned int i, PhotonNode photons){
            unsigned int photonStart;
            hipMemcpy(&photonStart, photonIndex + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
            unsigned int photonRange;
            hipMemcpy(&photonRange, range + i, sizeof(unsigned int), hipMemcpyDeviceToHost);
                    
            point positions[photonRange];
            hipMemcpy(positions, photons.pos + photonStart, photonRange * sizeof(point), hipMemcpyDeviceToHost);
            point hostMax = positions[0];
            point hostMin = positions[0];
            for (unsigned int p = 1; p < photonRange; ++p){
                hostMax = make_float3(max(hostMax.x, positions[p].x),
                                      max(hostMax.y, positions[p].y),
                                      max(hostMax.z, positions[p].z));
                hostMin = make_float3(min(hostMin.x, positions[p].x),
                                      min(hostMin.y, positions[p].y),
                                      min(hostMin.z, positions[p].z));
            }
                    
            point gpuMax;
            point gpuMin;
            hipMemcpy(&gpuMax, aabbMax + i, sizeof(point), hipMemcpyDeviceToHost);
            hipMemcpy(&gpuMin, aabbMin + i, sizeof(point), hipMemcpyDeviceToHost);
    
            if (hostMax.x != gpuMax.x || hostMax.y != gpuMax.y || hostMax.z != gpuMax.z){
                logger.info << "CPU max " << Utils::CUDA::Convert::ToString(hostMax);
                logger.info << " != GPU max " << Utils::CUDA::Convert::ToString(gpuMax) << logger.end;
            }
                    
            if (hostMin.x != gpuMin.x || hostMin.y != gpuMin.y || hostMin.z != gpuMin.z){
                logger.info << "CPU min " << Utils::CUDA::Convert::ToString(hostMin);
                logger.info << " != GPU min " << Utils::CUDA::Convert::ToString(gpuMin) << logger.end;
            }

        }

    }
}
