#include "hip/hip_runtime.h"
// KD tree lower node for photons
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Scene/PhotonLowerNode.h>
#include <Utils/CUDA/Utils.h>

namespace OpenEngine {
    namespace Scene {
     
        PhotonLowerNode::PhotonLowerNode()
            : KDNode() {}

        PhotonLowerNode::PhotonLowerNode(int photons)
            : KDNode(0) {

            maxSize = photons / MAX_SIZE * (2 * MAX_SIZE - 1);
            KDNode::Resize(maxSize);
            
            logger.info << "LowerNode inital max: " << maxSize << logger.end;

            cudaSafeMalloc(&smallRoot, this->maxSize * sizeof(int));

            // Alloc split information
            cudaSafeMalloc(&splitTriangleSet, 3 * photons * sizeof(int2));
            splitTriangleSetX = splitTriangleSet;
            splitTriangleSetY = splitTriangleSetX + photons;
            splitTriangleSetZ = splitTriangleSetY + photons;
            //cudaSafeMalloc(&splitTriangleSetX, photons * sizeof(int2));
            //cudaSafeMalloc(&splitTriangleSetY, photons * sizeof(int2));
            //cudaSafeMalloc(&splitTriangleSetZ, photons * sizeof(int2));

            CHECK_FOR_CUDA_ERROR();
        }

        void PhotonLowerNode::Resize(int i){
            KDNode::Resize(i);

            unsigned int copySize = this->size;

            int *tempInt;
            cudaSafeMalloc(&tempInt, i * sizeof(int));
            hipMemcpy(tempInt, smallRoot, copySize * sizeof(int), hipMemcpyDeviceToDevice);
            hipFree(smallRoot);
            smallRoot = tempInt;
            CHECK_FOR_CUDA_ERROR();
        }

    }
}
