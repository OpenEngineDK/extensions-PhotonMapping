#include "hip/hip_runtime.h"
// KD tree lower node for photons
// -------------------------------------------------------------------
// Copyright (C) 2010 OpenEngine.dk (See AUTHORS) 
// 
// This program is free software; It is covered by the GNU General 
// Public License version 2 or any later version. 
// See the GNU General Public License for more details (see LICENSE). 
//--------------------------------------------------------------------

#include <Scene/PhotonLowerNode.h>
#include <Utils/CUDA/Utils.h>

namespace OpenEngine {
    namespace Scene {
     
        PhotonLowerNode::PhotonLowerNode()
            : KDNode() {}

        PhotonLowerNode::PhotonLowerNode(int photons)
            : KDNode(0) {

            maxSize = photons / MAX_SIZE * (2 * MAX_SIZE - 1);
            KDNode::Resize(maxSize);
            
            logger.info << "LowerNode inital max: " << maxSize << logger.end;

            //cudaSafeMalloc(&smallRoot, this->maxSize * sizeof(int));
            cudaSafeMalloc(&extendedVolume, this->maxSize * sizeof(float));
            cudaSafeMalloc(&splittingPlane, this->maxSize * sizeof(int));
            //cudaSafeMalloc(&spanMask, this->maxSize * sizeof(int));

            // Alloc split information
            cudaSafeMalloc(&splitTriangleSet, 3 * photons * sizeof(int2));
            splitTriangleSetX = splitTriangleSet;
            splitTriangleSetY = splitTriangleSetX + photons;
            splitTriangleSetZ = splitTriangleSetY + photons;

            CHECK_FOR_CUDA_ERROR();
        }

        void PhotonLowerNode::Resize(int i){
            KDNode::Resize(i);

            unsigned int copySize = this->size;

            int *tempInt;
            /*
            cudaSafeMalloc(&tempInt, i * sizeof(int));
            hipMemcpy(tempInt, smallRoot, copySize * sizeof(int), hipMemcpyDeviceToDevice);
            hipFree(smallRoot);
            smallRoot = tempInt;
            CHECK_FOR_CUDA_ERROR();
            */
            float *tempFloat;
            cudaSafeMalloc(&tempFloat, i * sizeof(float));
            hipMemcpy(tempFloat, extendedVolume, copySize * sizeof(float), hipMemcpyDeviceToDevice);
            hipFree(extendedVolume);
            extendedVolume = tempFloat;
            CHECK_FOR_CUDA_ERROR();

            cudaSafeMalloc(&tempInt, i * sizeof(int));
            hipMemcpy(tempInt, splittingPlane, copySize * sizeof(int), hipMemcpyDeviceToDevice);
            hipFree(splittingPlane);
            splittingPlane = tempInt;
            CHECK_FOR_CUDA_ERROR();

            /*
            cudaSafeMalloc(&tempInt, i * sizeof(int));
            hipMemcpy(tempInt, spanMask, copySize * sizeof(int), hipMemcpyDeviceToDevice);
            hipFree(spanMask);
            spanMask = tempInt;
            CHECK_FOR_CUDA_ERROR();
            */
        }

    }
}
